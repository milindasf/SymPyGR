#include "hip/hip_runtime.h"
#include "derivs_cuda.h"

__device__ void device_calc_ko_deriv_x(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){

    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_ko_deriv; id<(thread_id+1)*thread_load_ko_deriv; id++){
            
        int i = id%(sz_x-8) + 4;
        int j = ((id/(sz_x-8))%(sz_y-6)) + 3;
        int k = (id/(sz_y-6)/(sz_x-8)) + 3;
        
        if(i >= nx-4 || j >= ny-3 || k >= nz-3) return;

        if(i==4) {
            int ib=3;
            output[IDX(3, j, k)] = (-1.0 / 64.0 / hx) *
            (
                    -      dev_var_in[offset + IDX(ib+4,j,k)]
                    +  6.0*dev_var_in[offset + IDX(ib+3,j,k)]
                    - 15.0*dev_var_in[offset + IDX(ib+2,j,k)]
                    + 20.0*dev_var_in[offset + IDX(ib+1,j,k)]
                    - 15.0*dev_var_in[offset + IDX(ib,j,k)]
                    +  6.0*dev_var_in[offset + IDX(ib-1,j,k)]
                    -      dev_var_in[offset + IDX(ib-2,j,k)]
            );
        }

        int pp = IDX(i, j, k);

        output[pp] = (-1.0 / 64.0 / hx) *
        (
        -      dev_var_in[offset + pp - 3]
        +  6.0*dev_var_in[offset + pp - 2]
        - 15.0*dev_var_in[offset + pp - 1]
        + 20.0*dev_var_in[offset + pp ]
        - 15.0*dev_var_in[offset + pp + 1]
        +  6.0*dev_var_in[offset + pp + 2]
        -      dev_var_in[offset + pp + 3]
        );

        if(i==5) {
            int ie = nx-3;
            output[IDX(ie-1, j, k)] = (-1.0 / 64.0 / hx) *
                (
                        -      dev_var_in[offset + IDX(ie+1,j,k)]
                        +  6.0*dev_var_in[offset + IDX(ie,j,k)]
                        - 15.0*dev_var_in[offset + IDX(ie-1,j,k)]
                        + 20.0*dev_var_in[offset + IDX(ie-2,j,k)]
                        - 15.0*dev_var_in[offset + IDX(ie-3,j,k)]
                        +  6.0*dev_var_in[offset + IDX(ie-4,j,k)]
                        -      dev_var_in[offset + IDX(ie-5,j,k)]
                );
        }

            if ((bflag & (1u<<OCT_DIR_LEFT)) && (i == 4)) {

            output[IDX(3,j,k)] =  (      dev_var_in[offset + IDX(6,j,k)]
                - 3.0*dev_var_in[offset + IDX(5,j,k)]
                + 3.0*dev_var_in[offset + IDX(4,j,k)]
                -     dev_var_in[offset + IDX(3,j,k)]
            )/59.0/48.0*64*hx;
            output[IDX(4,j,k)] =  (     dev_var_in[offset + IDX(7,j,k)]
                -  6.0*dev_var_in[offset + IDX(6,j,k)]
                + 12.0*dev_var_in[offset + IDX(5,j,k)]
                - 10.0*dev_var_in[offset + IDX(4,j,k)]
                +  3.0*dev_var_in[offset + IDX(3,j,k)]
            )/43.0/48.0*64*hx;
            output[IDX(5,j,k)] =  (     dev_var_in[offset + IDX(8,j,k)]
                -  6.0*dev_var_in[offset + IDX(7,j,k)]
                + 15.0*dev_var_in[offset + IDX(6,j,k)]
                - 19.0*dev_var_in[offset + IDX(5,j,k)]
                + 12.0*dev_var_in[offset + IDX(4,j,k)]
                -  3.0*dev_var_in[offset + IDX(3,j,k)]
            )/49.0/48.0*64*hx;
        }

            if ((bflag & (1u<<OCT_DIR_RIGHT)) && (i == 5)) {

            const int ie = nx - 3;
            output[IDX(ie-3,j,k)] = ( dev_var_in[offset + IDX(ie-6,j,k)]
                - 6.0*dev_var_in[offset + IDX(ie-5,j,k)]
                + 15.0*dev_var_in[offset + IDX(ie-4,j,k)]
                - 19.0*dev_var_in[offset + IDX(ie-3,j,k)]
                + 12.0*dev_var_in[offset + IDX(ie-2,j,k)]
                -  3.0*dev_var_in[offset + IDX(ie-1,j,k)]
            )/49.0/48.0*64*hx;

            output[IDX(ie-2,j,k)] =  ( dev_var_in[offset + IDX(ie-5,j,k)]
                -  6.0*dev_var_in[offset + IDX(ie-4,j,k)]
                + 12.0*dev_var_in[offset + IDX(ie-3,j,k)]
                - 10.0*dev_var_in[offset + IDX(ie-2,j,k)]
                +  3.0*dev_var_in[offset + IDX(ie-1,j,k)]
            )/43.0/48.0*64*hx;


            output[IDX(ie-1,j,k)] = ( dev_var_in[offset + IDX(ie-4,j,k)]
                -  3.0*dev_var_in[offset + IDX(ie-3,j,k)]
                +  3.0*dev_var_in[offset + IDX(ie-2,j,k)]
                -      dev_var_in[offset + IDX(ie-1,j,k)]
            )/59.0/48.0*64*hx;
        }
    }
} 

 
__device__ void device_calc_ko_deriv_y(double * output, double * dev_var_in,
    const int offset, double hy, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){

    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_ko_deriv; id<(thread_id+1)*thread_load_ko_deriv; id++){
            
        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-8)) + 4;
        int k = (id/(sz_y-8)/(sz_x-6)) + 3;

        if(i >= nx-3 || j >= ny-4 || k >= nz-3) return;

        if(j==4) {
            int jb=3;
            output[IDX(i,jb,k)] = (-1.0 / 64.0 / hy) *
            (
                    -      dev_var_in[offset + IDX(i,jb+4,k)]
                    +  6.0*dev_var_in[offset + IDX(i,jb+3,k)]
                    - 15.0*dev_var_in[offset + IDX(i,jb+2,k)]
                    + 20.0*dev_var_in[offset + IDX(i,jb+1,k)]
                    - 15.0*dev_var_in[offset + IDX(i,jb,k)]
                    +  6.0*dev_var_in[offset + IDX(i,jb-1,k)]
                    -      dev_var_in[offset + IDX(i,jb-2,k)]
            );
        }

        int pp = IDX(i, j, k);

        output[pp] = (-1.0 / 64.0 / hy) *
        (
        -      dev_var_in[offset + pp-3*nx]
        +  6.0*dev_var_in[offset + pp-2*nx]
        - 15.0*dev_var_in[offset + pp-nx]
        + 20.0*dev_var_in[offset + pp]
        - 15.0*dev_var_in[offset + pp+nx]
        +  6.0*dev_var_in[offset + pp+2*nx]
        -      dev_var_in[offset + pp+3*nx]
        );

        if(j==5) {
            int je = ny - 3;
            output[IDX(i,je-1,k)] = (-1.0 / 64.0 / hy) *
            (
                    -      dev_var_in[offset + IDX(i,je+1,k)]
                    +  6.0*dev_var_in[offset + IDX(i,je,k)]
                    - 15.0*dev_var_in[offset + IDX(i,je-1,k)]
                    + 20.0*dev_var_in[offset + IDX(i,je-2,k)]
                    - 15.0*dev_var_in[offset + IDX(i,je-3,k)]
                    +  6.0*dev_var_in[offset + IDX(i,je-4,k)]
                    -      dev_var_in[offset + IDX(i,je-5,k)]
            );
        }
            if ((bflag & (1u<<OCT_DIR_DOWN)) && (j == 4)) {

            output[IDX(i,3,k)] =  (      dev_var_in[offset +IDX(i,6,k)]
                - 3.0*dev_var_in[offset +IDX(i,5,k)]
                + 3.0*dev_var_in[offset + IDX(i,4,k)]
                -     dev_var_in[offset + IDX(i,3,k)]
            )/59.0/48.0*64*hy;
            output[IDX(i,4,k)] =  (     dev_var_in[offset + IDX(i,7,k)]
                -  6.0*dev_var_in[offset + IDX(i,6,k)]
                + 12.0*dev_var_in[offset + IDX(i,5,k)]
                - 10.0*dev_var_in[offset + IDX(i,4,k)]
                +  3.0*dev_var_in[offset + IDX(i,3,k)]
            )/43.0/48.0*64*hy;
            output[IDX(i,5,k)] =  (     dev_var_in[offset + IDX(i,8,k)]
                -  6.0*dev_var_in[offset + IDX(i,7,k)]
                + 15.0*dev_var_in[offset + IDX(i,6,k)]
                - 19.0*dev_var_in[offset + IDX(i,5,k)]
                + 12.0*dev_var_in[offset + IDX(i,4,k)]
                -  3.0*dev_var_in[offset + IDX(i,3,k)]
            )/49.0/48.0*64*hy;
        }

            if ((bflag & (1u<<OCT_DIR_UP)) && (j == 5)) {

            const int je = ny - 3;
            output[IDX(i,je-3,k)] = (dev_var_in[offset + IDX(i,je-6,k)]
            -  6.0*dev_var_in[offset + IDX(i,je-5,k)]
            + 15.0*dev_var_in[offset + IDX(i,je-4,k)]
            - 19.0*dev_var_in[offset + IDX(i,je-3,k)]
            + 12.0*dev_var_in[offset + IDX(i,je-2,k)]
            -  3.0*dev_var_in[offset + IDX(i,je-1,k)]
            )/49.0/48.0*64*hy;

            output[IDX(i,je-2,k)] = (dev_var_in[offset + IDX(i,je-5,k)]
            -  6.0*dev_var_in[offset + IDX(i,je-4,k)]
            + 12.0*dev_var_in[offset + IDX(i,je-3,k)]
            - 10.0*dev_var_in[offset + IDX(i,je-2,k)]
            +  3.0*dev_var_in[offset + IDX(i,je-1,k)]
            )/43.0/48.0*64*hy;


            output[IDX(i,je-1,k)] = ( dev_var_in[offset + IDX(i,je-4,k)]
                -  3.0*dev_var_in[offset + IDX(i,je-3,k)]
                +  3.0*dev_var_in[offset + IDX(i,je-2,k)]
                -      dev_var_in[offset + IDX(i,je-1,k)]
            )/59.0/48.0*64*hy;
        }
    }
}



__device__ void device_calc_ko_deriv_z(double * output, double * dev_var_in,
    const int offset, double hz, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){

    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_ko_deriv; id<(thread_id+1)*thread_load_ko_deriv; id++){
            
        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
        int k = (id/(sz_y-6)/(sz_x-6)) + 4;

        if(i >= nx-3 || j >= ny-3 || k >= nz-4) return;

        if(k==4) {
            int kb=3;
            output[IDX(i,j,kb)] = (-1.0 / 64.0 / hz) *
            (
                    -      dev_var_in[offset + IDX(i,j,kb+4)]
                    +  6.0*dev_var_in[offset + IDX(i,j,kb+3)]
                    - 15.0*dev_var_in[offset + IDX(i,j,kb+2)]
                    + 20.0*dev_var_in[offset + IDX(i,j,kb+1)]
                    - 15.0*dev_var_in[offset + IDX(i,j,kb)]
                    +  6.0*dev_var_in[offset + IDX(i,j,kb-1)]
                    -      dev_var_in[offset + IDX(i,j,kb-2)]
            );
        }

        int pp = IDX(i, j, k);
        int n = nx * ny;
        output[pp] = (-1.0 / 64.0 / hz) *
        (
        -      dev_var_in[offset + pp-3*n]
        +  6.0*dev_var_in[offset + pp-2*n]
        - 15.0*dev_var_in[offset + pp-n]
        + 20.0*dev_var_in[offset + pp]
        - 15.0*dev_var_in[offset + pp+n]
        +  6.0*dev_var_in[offset + pp+2*n]
        -      dev_var_in[offset + pp+3*n]
        );

        if(k==5) {
            int ke = nz - 3;
            output[IDX(i,j,ke-1)] = (-1.0 / 64.0 / hz) *
            (
                    -      dev_var_in[offset + IDX(i,j,ke+1)]
                    +  6.0*dev_var_in[offset + IDX(i,j,ke)]
                    - 15.0*dev_var_in[offset + IDX(i,j,ke-1)]
                    + 20.0*dev_var_in[offset + IDX(i,j,ke-2)]
                    - 15.0*dev_var_in[offset + IDX(i,j,ke-3)]
                    +  6.0*dev_var_in[offset + IDX(i,j,ke-4)]
                    -      dev_var_in[offset + IDX(i,j,ke-5)]
            );
        }

        if ((bflag & (1u<<OCT_DIR_BACK)) && (k == 4)) {

            output[IDX(i,3,k)] =  (      dev_var_in[offset +IDX(i,k,6)]
                - 3.0*dev_var_in[offset +IDX(i,k,5)]
                + 3.0*dev_var_in[offset + IDX(i,k,4)]
                -     dev_var_in[offset + IDX(i,k,3)]
            )/59.0/48.0*64*hz;
            output[IDX(i,j,4)] =  (     dev_var_in[offset + IDX(i,j,7)]
                -  6.0*dev_var_in[offset + IDX(i,j,6)]
                + 12.0*dev_var_in[offset + IDX(i,j,5)]
                - 10.0*dev_var_in[offset + IDX(i,j,4)]
                +  3.0*dev_var_in[offset + IDX(i,j,3)]
            )/43.0/48.0*64*hz;
            output[IDX(i,j,5)] =  (     dev_var_in[offset + IDX(i,j,8)]
                -  6.0*dev_var_in[offset + IDX(i,j,7)]
                + 15.0*dev_var_in[offset + IDX(i,j,6)]
                - 19.0*dev_var_in[offset + IDX(i,j,5)]
                + 12.0*dev_var_in[offset + IDX(i,j,4)]
                -  3.0*dev_var_in[offset + IDX(i,j,3)]
            )/49.0/48.0*64*hz;
        }

        if ((bflag & (1u<<OCT_DIR_FRONT)) && (k == 5)) {

            const int ke = nz - 3;
            output[IDX(i,j,ke-3)] = (    dev_var_in[offset + IDX(i,j,ke-6)]
                -  6.0*dev_var_in[offset + IDX(i,j,ke-5)]
                + 15.0*dev_var_in[offset + IDX(i,j,ke-4)]
                - 19.0*dev_var_in[offset + IDX(i,j,ke-3)]
                + 12.0*dev_var_in[offset + IDX(i,j,ke-2)]
                -  3.0*dev_var_in[offset + IDX(i,j,ke-1)]
            )/49.0/48.0*64*hz;

            output[IDX(i,j,ke-2)] = (   dev_var_in[offset + IDX(i,j,ke-5)]
                -  6.0*dev_var_in[offset + IDX(i,j,ke-4)]
                + 12.0*dev_var_in[offset + IDX(i,j,ke-3)]
                - 10.0*dev_var_in[offset + IDX(i,j,ke-2)]
                +  3.0*dev_var_in[offset + IDX(i,j,ke-1)]
            )/43.0/48.0*64*hz;


            output[IDX(i,j,ke-1)] = (   dev_var_in[offset + IDX(i,j,ke-4)]
                -  3.0*dev_var_in[offset + IDX(i,j,ke-3)]
                +  3.0*dev_var_in[offset + IDX(i,j,ke-2)]
                -      dev_var_in[offset + IDX(i,j,ke-1)]
            )/59.0/48.0*64*hz;
        }
    }
}

 
 __global__ void cuda_calc_ko_deriv_all(double * dev_var_in, double hx, double hy, double hz, 
    int sz_x, int sz_y, int sz_z, int bflag,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
 ) {
    int nx = sz_x;
    int ny = sz_y;
    int nz = sz_z;

    #include "bssnrhs_cuda_ko_derivs.h"
 }

 void calc_ko_deriv_all( double * dev_var_in, double hx, double hy, double hz, int sz_x, 
    int sz_y, int sz_z, int bflag, hipStream_t stream,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
 )
 {
    const int ie = sz_x - 3;//x direction
    const int je = sz_y - 3;//y direction
    const int ke = sz_z - 3;//z direction
 
    int total_points = ceil(1.0*ie*je*ke/thread_load_ko_deriv);
    int blocks = ceil(1.0*total_points/threads_per_block);

    cuda_calc_ko_deriv_all <<< blocks, threads_per_block, 0, stream >>> (
                    dev_var_in, hx, hy, hz, sz_x, sz_y, sz_z, bflag,
                  #include "list_of_args.h"
                  ,
                  #include "list_of_offset_args.h"
                );
 }