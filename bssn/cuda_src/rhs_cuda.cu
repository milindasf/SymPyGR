#include "hip/hip_runtime.h"
#include "rhs_cuda.h"
#include "bssneqn_solve.h"

#include <iostream>
#include <stdio.h>


enum VAR_CU {U_ALPHA=0,U_CHI,U_K,U_GT0,U_GT1,U_GT2,U_BETA0,U_BETA1,U_BETA2,U_B0,U_B1,U_B2,U_SYMGT0,U_SYMGT1,U_SYMGT2,U_SYMGT3,U_SYMGT4,U_SYMGT5,U_SYMAT0,U_SYMAT1,U_SYMAT2,U_SYMAT3,U_SYMAT4,U_SYMAT5};

void cuda_bssnrhs(double * dev_var_out, double * dev_var_in, const unsigned int unzip_dof, 
const unsigned int& offset, const double *pmin, const double *pmax, const unsigned int *sz, 
const unsigned int& bflag)
{
    int alphaInt = (VAR_CU::U_ALPHA) * unzip_dof + offset;
    int chiInt = (VAR_CU::U_CHI) * unzip_dof + offset;
    int KInt = (VAR_CU::U_K) * unzip_dof + offset;
    int gt0Int = (VAR_CU::U_SYMGT0) * unzip_dof + offset;
    int gt1Int = (VAR_CU::U_SYMGT1) * unzip_dof + offset;
    int gt2Int =  (VAR_CU::U_SYMGT2) * unzip_dof + offset;
    int gt3Int =(VAR_CU::U_SYMGT3) * unzip_dof + offset;
    int gt4Int = (VAR_CU::U_SYMGT4) * unzip_dof + offset;
    int gt5Int = (VAR_CU::U_SYMGT5) * unzip_dof + offset;
    int beta0Int = (VAR_CU::U_BETA0) * unzip_dof + offset;
    int beta1Int = (VAR_CU::U_BETA1) * unzip_dof + offset;
    int beta2Int =(VAR_CU::U_BETA2) * unzip_dof + offset;
    int At0Int = (VAR_CU::U_SYMAT0) * unzip_dof + offset;
    int At1Int = (VAR_CU::U_SYMAT1) * unzip_dof + offset;
    int At2Int = (VAR_CU::U_SYMAT2) * unzip_dof + offset;
    int At3Int = (VAR_CU::U_SYMAT3) * unzip_dof + offset;
    int At4Int = (VAR_CU::U_SYMAT4) * unzip_dof + offset;
    int At5Int = (VAR_CU::U_SYMAT5) * unzip_dof + offset;
    int Gt0Int = (VAR_CU::U_GT0) * unzip_dof + offset;
    int Gt1Int = (VAR_CU::U_GT1) * unzip_dof + offset;
    int Gt2Int = (VAR_CU::U_GT2) * unzip_dof + offset;
    int B0Int = (VAR_CU::U_B0) * unzip_dof + offset;
    int B1Int = (VAR_CU::U_B1) * unzip_dof + offset;
    int B2Int = (VAR_CU::U_B2) * unzip_dof + offset;

    double hx = (pmax[0] - pmin[0]) / (sz[0] - 1);
    double hy = (pmax[1] - pmin[1]) / (sz[1] - 1);
    double hz = (pmax[2] - pmin[2]) / (sz[2] - 1);

    // Send above values to GPU memory
    hipError_t cudaStatus;
    #include "bssnrhs_cuda_offset_malloc.h"

    double * dev_dy_hx; //similar to hx in cpu code
    cudaStatus = hipMalloc((void **) &dev_dy_hx, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hx hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hx, &hx, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hx hipMemcpy failed!\n"); return;}

    double * dev_dy_hy;
    cudaStatus = hipMalloc((void **) &dev_dy_hy, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hy hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hy, &hy, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hy hipMemcpy failed!\n"); return;}

    double * dev_dy_hz;
    cudaStatus = hipMalloc((void **) &dev_dy_hz, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hz hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hz, &hz, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hz hipMemcpy failed!\n"); return;}

    int * dev_sz;
    cudaStatus = hipMalloc((void **) &dev_sz, 3*sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "sz hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_sz, sz, 3*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "sz hipMemcpy failed!\n"); return;}

    int * dev_zero;
    cudaStatus = hipMalloc((void **) &dev_zero, sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "0 hipMalloc failed!\n"); return;}

    double * dev_pmin;
    cudaStatus = hipMalloc((void **) &dev_pmin, 3*sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmin hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_pmin, pmin, 3*sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmin hipMemcpy failed!\n"); return;}

    double *dev_pmax;
    cudaStatus = hipMalloc((void **) &dev_pmax, sizeof(pmax)*sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmax hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_pmax, pmax, sizeof(pmax)*sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmax hipMemcpy failed!\n"); return;}

    // Allocate memory to store the output of derivs
    unsigned int n = sz[0]*sz[1]*sz[2];
    int size = n * sizeof(double);

    bssn::timer::t_deriv.start();

    #include "bssnrhs_cuda_malloc.h"
    #include "bssnrhs_cuda_malloc_adv.h"
   

    // Deriv calls are follows
    #include "bssnrhs_cuda_derivs.h"
    #include "bssnrhs_cuda_derivs_adv.h"

    bssn::timer::t_deriv.stop();


    bssn::timer::t_rhs.start();
    calc_bssn_eqns(sz, dev_sz, dev_pmin, dev_dy_hz, dev_dy_hy, dev_dy_hx, dev_var_in, dev_var_out,
        #include "list_of_args.h"
    );
    bssn::timer::t_rhs.stop();

    #if test
    // Copying specified array to CPU for testing purpose
    double * host_array_cpu = (double *) malloc(size);
    cudaStatus = hipMemcpy(host_array_cpu, grad_1_alpha, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "TEST: host_array_cpu hipMemcpy from GPU to CPU failed!\n"); return;}
    test_file_write::writeToFile("output_cuda.txt", host_array_cpu, n);
    free(host_array_cpu);

    // double * host_array_cpu = (double *) malloc(unzip_dof*24);
    // cudaStatus = hipMemcpy(host_array_cpu, dev_var_out, unzip_dof*24, hipMemcpyDeviceToHost);
    // if (cudaStatus != hipSuccess) {fprintf(stderr, "TEST: host_array_cpu hipMemcpy from GPU to CPU failed!\n"); return;}
    // test_file_write::writeToFile("output_cuda.txt", host_array_cpu, unzip_dof*24);
    // free(host_array_cpu);
    #endif

    // Free up GPU memory
    #include "bssnrhs_cuda_offset_demalloc.h"
    #include "bssnrhs_cuda_mdealloc.h"
    #include "bssnrhs_cuda_mdealloc_adv.h"
    hipFree(dev_dy_hx);
    hipFree(dev_dy_hy);
    hipFree(dev_dy_hz);
    hipFree(dev_sz);
    hipFree(dev_zero);
    hipFree(dev_pmin);
}

__global__ void cacl_bssn_bcs_x(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    const double *pmin, const double f_falloff, const double f_asymptotic,
    const unsigned int *dev_sz, int* dev_bflag) {

        int x;
        int y;
        int z;

        if (*dev_bflag & (1u<<OCT_DIR_LEFT)) {
            x = pmin[0] + 3*((pmax[0] - pmin[0]) / (nx - 1));
            for (unsigned int k = kb; k < ke; k++) {
               z = pmin[2] + k*hz;
              for (unsigned int j = jb; j < je; j++) {
                 y = pmin[1] + j*hy;
                 pp = IDX(ib,j,k);
                 inv_r = 1.0 / sqrt(x*x + y*y + z*z);
        
                f_rhs[pp] = -  inv_r * (
                                 x * dxf[pp]
                               + y * dyf[pp]
                               + z * dzf[pp]
                               + f_falloff * (   f[pp] - f_asymptotic ) );
        
              }
            }
          }
        
          if (*dev_bflag & (1u<<OCT_DIR_RIGHT)) {
             x = pmin[0] + ie*hx;
            for (unsigned int k = kb; k < ke; k++) {
               z = pmin[2] + k*hz;
              for (unsigned int j = jb; j < je; j++) {
                 y = pmin[1] + j*hy;
                 pp = IDX(ie,j,k);
                 inv_r = 1.0 / sqrt(x*x + y*y + z*z);
        
                f_rhs[pp] = -  inv_r * (
                                 x * dxf[pp]
                               + y * dyf[pp]
                               + z * dzf[pp]
                               + f_falloff * (   f[pp] - f_asymptotic ) );
        
              }
            }
          }

}

__global__ void cacl_bssn_bcs_y(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    const double *pmin, const double f_falloff, const double f_asymptotic,
    const unsigned int *host_sz, int* dev_bflag) {



}

__global__ void cacl_bssn_bcs_z(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    const double *pmin, const double f_falloff, const double f_asymptotic,
    const unsigned int *host_sz, int* dev_bflag) {



}
void bssn_bcs(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    const double *pmin,const double *pmax, const double f_falloff, const double f_asymptotic,
    const unsigned int *host_sz, int* dev_bflag, int* dev_sz) {

        const unsigned int nx = host_sz[0];
        const unsigned int ny = host_sz[1];
        const unsigned int nz = host_sz[2];

        double hx = (pmax[0] - pmin[0]) / (nx - 1);
        double hy = (pmax[1] - pmin[1]) / (ny - 1);
        double hz = (pmax[2] - pmin[2]) / (nz - 1);

        const int ie = nx - 3;//x direction
        const int je = ny - 3;//y direction
        const int ke = nz - 3;//z direction

        int temp_max = (ie>je)? ie : je;
        int maximumIterations = (temp_max>ke) ? temp_max: ke;
        
        int requiredBlocks = maximumIterations / 10;
        if (ie % 10 != 0 || je % 10 != 0 || ke % 10 != 0) {
            requiredBlocks++;
        }
        
        int threads_x = ie / requiredBlocks;
        int threads_y = je / requiredBlocks;
        int threads_z = ke / requiredBlocks;
        
        calc_co_deriv42_x <<< dim2(threads_x,threads_y), dim2(threads_x,threads_y) >>> (output, dev_var_in,
            dev_dx, dev_bflag, dev_sz, dev_u_offset);


    }
