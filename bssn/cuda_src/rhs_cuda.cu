#include "hip/hip_runtime.h"
#include "rhs_cuda.h"
#include "bssneqn_solve.h"

#include <iostream>
#include <stdio.h>


enum VAR_CU {U_ALPHA=0,U_CHI,U_K,U_GT0,U_GT1,U_GT2,U_BETA0,U_BETA1,U_BETA2,U_B0,U_B1,U_B2,U_SYMGT0,U_SYMGT1,U_SYMGT2,U_SYMGT3,U_SYMGT4,U_SYMGT5,U_SYMAT0,U_SYMAT1,U_SYMAT2,U_SYMAT3,U_SYMAT4,U_SYMAT5};

void cuda_bssnrhs(double * dev_var_out, double * dev_var_in, const unsigned int unzip_dof, 
const double * pmin, const double * pmax, const unsigned int * sz, 
const unsigned int& bflag, hipStream_t stream,
#include "list_of_para.h"
)
{ 
    int alphaInt = (VAR_CU::U_ALPHA) * unzip_dof;
    int chiInt = (VAR_CU::U_CHI) * unzip_dof;
    int KInt = (VAR_CU::U_K) * unzip_dof;
    int gt0Int = (VAR_CU::U_SYMGT0) * unzip_dof;
    int gt1Int = (VAR_CU::U_SYMGT1) * unzip_dof;
    int gt2Int =  (VAR_CU::U_SYMGT2) * unzip_dof;
    int gt3Int = (VAR_CU::U_SYMGT3) * unzip_dof;
    int gt4Int = (VAR_CU::U_SYMGT4) * unzip_dof;
    int gt5Int = (VAR_CU::U_SYMGT5) * unzip_dof;
    int beta0Int = (VAR_CU::U_BETA0) * unzip_dof;
    int beta1Int = (VAR_CU::U_BETA1) * unzip_dof;
    int beta2Int = (VAR_CU::U_BETA2) * unzip_dof;
    int At0Int = (VAR_CU::U_SYMAT0) * unzip_dof;
    int At1Int = (VAR_CU::U_SYMAT1) * unzip_dof;
    int At2Int = (VAR_CU::U_SYMAT2) * unzip_dof;
    int At3Int = (VAR_CU::U_SYMAT3) * unzip_dof;
    int At4Int = (VAR_CU::U_SYMAT4) * unzip_dof;
    int At5Int = (VAR_CU::U_SYMAT5) * unzip_dof;
    int Gt0Int = (VAR_CU::U_GT0) * unzip_dof;
    int Gt1Int = (VAR_CU::U_GT1) * unzip_dof;
    int Gt2Int = (VAR_CU::U_GT2) * unzip_dof;
    int B0Int = (VAR_CU::U_B0) * unzip_dof;
    int B1Int = (VAR_CU::U_B1) * unzip_dof;
    int B2Int = (VAR_CU::U_B2) * unzip_dof;

    double hx = (pmax[0] - pmin[0]) / (sz[0] - 1);
    double hy = (pmax[1] - pmin[1]) / (sz[1] - 1);
    double hz = (pmax[2] - pmin[2]) / (sz[2] - 1);

    // Deriv calls are follows
    cuda_calc_all(dev_var_in, hx, hy, hz, sz[0], sz[1], sz[2], bflag, stream, 
        #include "list_of_args.h"
        ,
        #include "list_of_offset_args.h"
            );
        
    cuda_deriv_calc_all_adv(dev_var_in, hx, hy, hz, sz[0], sz[1], sz[2], bflag, stream,
        #include "list_of_args.h"
        ,
        #include "list_of_offset_args.h"
            );
    
    calc_bssn_eqns(dev_var_in, dev_var_out, sz, pmin, hz, hy, hx, stream,
    #include "list_of_offset_args.h"
    ,
    #include "list_of_args.h"
    );

    if (bflag != 0) {

        bssn_bcs(dev_var_out, dev_var_in, alphaInt, grad_0_alpha, grad_1_alpha, grad_2_alpha,
            pmin, pmax, 1.0, 1.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, chiInt, grad_0_chi, grad_1_chi, grad_2_chi,
            pmin, pmax, 1.0, 1.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, KInt, grad_0_K, grad_1_K, grad_2_K,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);

        bssn_bcs(dev_var_out, dev_var_in, beta0Int, grad_0_beta0, grad_1_beta0, grad_2_beta0,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, beta1Int, grad_0_beta1, grad_1_beta1, grad_2_beta1,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, beta2Int, grad_0_beta2, grad_1_beta2, grad_2_beta2,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);

        bssn_bcs(dev_var_out, dev_var_in, Gt0Int, grad_0_Gt0, grad_1_Gt0, grad_2_Gt0,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, Gt1Int, grad_0_Gt1, grad_1_Gt1, grad_2_Gt1,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, Gt2Int, grad_0_Gt2, grad_1_Gt2, grad_2_Gt2,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);

        bssn_bcs(dev_var_out, dev_var_in, B0Int, grad_0_B0, grad_1_B0, grad_2_B0,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, B1Int, grad_0_B1, grad_1_B1, grad_2_B1,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, B2Int, grad_0_B2, grad_1_B2, grad_2_B2,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);

        bssn_bcs(dev_var_out, dev_var_in, At0Int, grad_0_At0, grad_1_At0, grad_2_At0,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, At1Int, grad_0_At1, grad_1_At1, grad_2_At1,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, At2Int, grad_0_At2, grad_1_At2, grad_2_At2,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, At3Int, grad_0_At3, grad_1_At3, grad_2_At3,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, At4Int, grad_0_At4, grad_1_At4, grad_2_At4,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, At5Int, grad_0_At5, grad_1_At5, grad_2_At5,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream); 

        bssn_bcs(dev_var_out, dev_var_in, gt0Int, grad_0_gt0, grad_1_gt0, grad_2_gt0,
            pmin, pmax, 1.0, 1.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, gt1Int, grad_0_gt1, grad_1_gt1, grad_2_gt1,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, gt2Int, grad_0_gt2, grad_1_gt2, grad_2_gt2,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, gt3Int, grad_0_gt3, grad_1_gt3, grad_2_gt3,
            pmin, pmax, 1.0, 1.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, gt4Int, grad_0_gt4, grad_1_gt4, grad_2_gt4,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, gt5Int, grad_0_gt5, grad_1_gt5, grad_2_gt5,
            pmin, pmax, 1.0, 1.0, sz, bflag, stream);
        
    }

    calc_ko_deriv_all(dev_var_in, hx, hy, hz, sz[0], sz[1], sz[2], bflag, stream,
        #include "list_of_args.h"
        ,
        #include "list_of_offset_args.h"
        );

    get_output(dev_var_out, sz, stream,
        #include "list_of_offset_args.h"
        ,
        #include "list_of_args.h"
    );

}

__global__ void cacl_bssn_bcs_x(double * dev_var_out, double * dev_var_in, 
    int u_offset,
    double * dxf, double * dyf, double * dzf,
    double pmin_x, double pmin_y, double pmin_z, double pmax_x, double pmax_y, double pmax_z,
    const double f_falloff, const double f_asymptotic,
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z,
    int bflag) {

        int j = 3 + threadIdx.x + blockIdx.x * blockDim.x;
        int k = 3 + threadIdx.y + blockIdx.y * blockDim.y;
        int nx = host_sz_x;
        int ny = host_sz_y;
        int nz = host_sz_z;

        if(j >= ny-3 || k >= nz-3) return;

        double inv_r;
        double hx = (pmax_x - pmin_x) / (nx - 1);
        double hy = (pmax_y - pmin_y) / (ny - 1);
        double hz = (pmax_z - pmin_z) / (nz - 1);
        double x, y, z;
        int pp;

        if (bflag & (1u<<OCT_DIR_LEFT)) {
            
            x = pmin_x + 3*hx;
            z = pmin_z + k*hz;
            y = pmin_y + j*hy;
            pp = IDX(3,j,k);
            inv_r = 1.0 / sqrt(x*x + y*y + z*z);
   
            dev_var_out[u_offset + pp] = -  inv_r * (
                            x * dxf[pp]
                          + y * dyf[pp]
                          + z * dzf[pp]
                          + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
          }
        
          if (bflag & (1u<<OCT_DIR_RIGHT)) {
             x = pmin_x + (nx - 3)*hx;
             z = pmin_z + k*hz;
             y = pmin_y + j*hy;
             pp = IDX((nx - 3),j,k);
             inv_r = 1.0 / sqrt(x*x + y*y + z*z);
    
             dev_var_out[u_offset + pp] = -  inv_r * (
                             x * dxf[pp]
                           + y * dyf[pp]
                           + z * dzf[pp]
                           + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
          }

}

__global__ void cacl_bssn_bcs_y(double * dev_var_out, double * dev_var_in, 
    int u_offset,
    double * dxf, double * dyf, double * dzf,
    double pmin_x, double pmin_y, double pmin_z, double pmax_x, double pmax_y, double pmax_z,
    const double f_falloff, const double f_asymptotic,
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z,
    int bflag) {

        int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
        int k = 3 + threadIdx.y + blockIdx.y * blockDim.y;
        int nx = host_sz_x;
        int ny = host_sz_y;
        int nz = host_sz_z;

        if(i >= nx-3 || k >= nz-3) return;

        double inv_r;
        double hx = (pmax_x - pmin_x) / (nx - 1);
        double hy = (pmax_y - pmin_y) / (ny - 1);
        double hz = (pmax_z - pmin_z) / (nz - 1);
        double x, y, z;
        int pp;

        if (bflag & (1u<<OCT_DIR_DOWN)) {
            
            y = pmin_y + 3*hy;
            z = pmin_z + k*hz;
            x = pmin_x + i*hx;
            pp = IDX(i,3,k);
            inv_r = 1.0 / sqrt(x*x + y*y + z*z);
   
            dev_var_out[u_offset + pp] = -  inv_r * (
                            x * dxf[pp]
                          + y * dyf[pp]
                          + z * dzf[pp]
                          + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
            
          }
        
          if (bflag & (1u<<OCT_DIR_UP)) {
             x = pmin_x + i*hx;
             z = pmin_z + k*hz;
             y = pmin_y + (ny-3)*hy;
             pp = IDX(i,(ny - 3),k);
             inv_r = 1.0 / sqrt(x*x + y*y + z*z);
    
             dev_var_out[u_offset + pp] = -  inv_r * (
                             x * dxf[pp]
                           + y * dyf[pp]
                           + z * dzf[pp]
                           + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
               
          }
}

__global__ void cacl_bssn_bcs_z(double * dev_var_out, double * dev_var_in, 
    int u_offset,
    double * dxf, double * dyf, double * dzf,
    double pmin_x, double pmin_y, double pmin_z, double pmax_x, double pmax_y, double pmax_z,
    const double f_falloff, const double f_asymptotic,
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z,
    int bflag) {
        
            int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
            int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
            int nx = host_sz_x;
            int ny = host_sz_y;
            int nz = host_sz_z;

            if(i >= nx-3 || j >= ny-3) return;

            double inv_r;
            double hx = (pmax_x - pmin_x) / (nx - 1);
            double hy = (pmax_y - pmin_y) / (ny - 1);
            double hz = (pmax_z - pmin_z) / (nz - 1);
            double x, y, z;
            int pp;

            if (bflag & (1u<<OCT_DIR_BACK)) {
            
            y = pmin_y + j*hy;
            z = pmin_z + 3*hz;
            x = pmin_x + i*hx;
            pp = IDX(i,j,3);
            inv_r = 1.0 / sqrt(x*x + y*y + z*z);
   
            dev_var_out[u_offset + pp] = -  inv_r * (
                            x * dxf[pp]
                          + y * dyf[pp]
                          + z * dzf[pp]
                          + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
            
          }
        
          if (bflag & (1u<<OCT_DIR_FRONT)) {
            x = pmin_x + i*hx;
            z = pmin_z + (nz-3)*hz;
            y = pmin_y + j*hy;
            pp = IDX(i,j,3);
            inv_r = 1.0 / sqrt(x*x + y*y + z*z);
    
            dev_var_out[u_offset + pp] = -  inv_r * (
                             x * dxf[pp]
                           + y * dyf[pp]
                           + z * dzf[pp]
                           + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
               
          }
}


// dev_var_out, dev_var_in, dev_At5Int, grad_0_At5, grad_1_At5, grad_2_At5,
//             dev_pmin, dev_pmax, 2.0, 0.0, sz, dev_bflag, dev_sz, stream


void bssn_bcs(double * dev_var_out, double * dev_var_in, 
    int u_offset, double * dxf, double * dyf, double * dzf,
    const double * pmin, const double * pmax, const double f_falloff, const double f_asymptotic,
    const unsigned int * host_sz, int bflag, hipStream_t stream) {
        
        const unsigned int nx = host_sz[0];
        const unsigned int ny = host_sz[1];
        const unsigned int nz = host_sz[2];

        const int ie = nx - 3;//x direction
        const int je = ny - 3;//y direction
        const int ke = nz - 3;//z direction

        double pmin_x = pmin[0];
        double pmin_y = pmin[1];
        double pmin_z = pmin[2];

        double pmax_x = pmax[0];
        double pmax_y = pmax[1];
        double pmax_z = pmax[2];

        const unsigned int host_sz_x = host_sz[0];
        const unsigned int host_sz_y = host_sz[1];
        const unsigned int host_sz_z = host_sz[2];

        int maximumIterations = (je>ke) ? je: ke;
        
        int requiredBlocks = (9 + maximumIterations) / 10;
        
        int threads_y = (requiredBlocks-1+je) / requiredBlocks;
        int threads_z = (requiredBlocks-1+ke) / requiredBlocks;
        
        cacl_bssn_bcs_x <<< dim3(threads_y,threads_z), dim3(threads_y,threads_z), 0, stream >>> (
            dev_var_out, dev_var_in, 
            u_offset, dxf, dyf, dzf, 
            pmin_x, pmin_y, pmin_z, pmax_x, pmax_y, pmax_z, 
            f_falloff, f_asymptotic, 
            host_sz_x, host_sz_y, host_sz_z, 
            bflag );
        
        CHECK_ERROR(hipGetLastError(), "cacl_bssn_bcs_x Kernel launch failed");
           
        maximumIterations = (ke>ie) ? ke : ie ;
        requiredBlocks = (9 + maximumIterations)/10;
        int threads_x = (requiredBlocks-1+ie) / requiredBlocks;
        threads_z = (requiredBlocks-1+ke) / requiredBlocks;
        cacl_bssn_bcs_y <<< dim3(threads_x,threads_z), dim3(threads_x,threads_z), 0, stream >>> (
            dev_var_out, dev_var_in, 
            u_offset, dxf, dyf, dzf, 
            pmin_x, pmin_y, pmin_z, pmax_x, pmax_y, pmax_z, 
            f_falloff, f_asymptotic, 
            host_sz_x, host_sz_y, host_sz_z, 
            bflag );
 
        CHECK_ERROR(hipGetLastError(), "cacl_bssn_bcs_y Kernel launch failed");

        maximumIterations = (je>ie) ? je : ie ;
        requiredBlocks = (9 + maximumIterations)/10;
        threads_x = (requiredBlocks-1+ie) / requiredBlocks;
        threads_y = (requiredBlocks-1+je) / requiredBlocks;
        cacl_bssn_bcs_z <<< dim3(threads_x,threads_y), dim3(threads_x,threads_y), 0, stream >>> (
            dev_var_out, dev_var_in, 
            u_offset, dxf, dyf, dzf, 
            pmin_x, pmin_y, pmin_z, pmax_x, pmax_y, pmax_z, 
            f_falloff, f_asymptotic, 
            host_sz_x, host_sz_y, host_sz_z, 
            bflag );

        CHECK_ERROR(hipGetLastError(), "cacl_bssn_bcs_z Kernel launch failed");
    }

__global__ void kernal_get_output (double * dev_var_out, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
) 
{
    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_output; id<(thread_id+1)*thread_load_output; id++){
                

        int i = id%(host_sz_x-6) + 3;
        int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
        int k = (id/(host_sz_y-6)/(host_sz_x-6)) + 3;

        int nx = host_sz_x;
        int ny = host_sz_y;

        if(i >= nx-3 || j >= ny-3 || k >= host_sz_z-3) return;

        const  double sigma = 1e-4;
        int pp = i + nx*(j + ny*k);

        dev_var_out[alphaInt + pp] += sigma * (grad_0_alpha[pp] + grad_1_alpha[pp] + grad_2_alpha[pp]);
        dev_var_out[beta0Int + pp] += sigma * (grad_0_beta0[pp] + grad_1_beta0[pp] + grad_2_beta0[pp]);
        dev_var_out[beta1Int + pp] += sigma * (grad_0_beta1[pp] + grad_1_beta1[pp] + grad_2_beta1[pp]);
        dev_var_out[beta2Int + pp] += sigma * (grad_0_beta2[pp] + grad_1_beta2[pp] + grad_2_beta2[pp]);

        dev_var_out[gt0Int + pp] += sigma * (grad_0_gt0[pp] + grad_1_gt0[pp] + grad_2_gt0[pp]);
        dev_var_out[gt1Int + pp] += sigma * (grad_0_gt1[pp] + grad_1_gt1[pp] + grad_2_gt1[pp]);
        dev_var_out[gt2Int + pp] += sigma * (grad_0_gt2[pp] + grad_1_gt2[pp] + grad_2_gt2[pp]);
        dev_var_out[gt3Int + pp] += sigma * (grad_0_gt3[pp] + grad_1_gt3[pp] + grad_2_gt3[pp]);
        dev_var_out[gt4Int + pp] += sigma * (grad_0_gt4[pp] + grad_1_gt4[pp] + grad_2_gt4[pp]);
        dev_var_out[gt5Int + pp] += sigma * (grad_0_gt5[pp] + grad_1_gt5[pp] + grad_2_gt5[pp]);

        dev_var_out[chiInt + pp]  += sigma * (grad_0_chi[pp] + grad_1_chi[pp] + grad_2_chi[pp]);

        dev_var_out[At0Int + pp] += sigma * (grad_0_At0[pp] + grad_1_At0[pp] + grad_2_At0[pp]);
        dev_var_out[At1Int + pp] += sigma * (grad_0_At1[pp] + grad_1_At1[pp] + grad_2_At1[pp]);
        dev_var_out[At2Int + pp] += sigma * (grad_0_At2[pp] + grad_1_At2[pp] + grad_2_At2[pp]);
        dev_var_out[At3Int + pp] += sigma * (grad_0_At3[pp] + grad_1_At3[pp] + grad_2_At3[pp]);
        dev_var_out[At4Int + pp] += sigma * (grad_0_At4[pp] + grad_1_At4[pp] + grad_2_At4[pp]);
        dev_var_out[At5Int + pp] += sigma * (grad_0_At5[pp] + grad_1_At5[pp] + grad_2_At5[pp]);

        dev_var_out[KInt + pp] += sigma * (grad_0_K[pp] + grad_1_K[pp] + grad_2_K[pp]);
        
        dev_var_out[Gt0Int + pp] += sigma * (grad_0_Gt0[pp] + grad_1_Gt0[pp] + grad_2_Gt0[pp]);
        dev_var_out[Gt1Int + pp] += sigma * (grad_0_Gt1[pp] + grad_1_Gt1[pp] + grad_2_Gt1[pp]);
        dev_var_out[Gt2Int + pp] += sigma * (grad_0_Gt2[pp] + grad_1_Gt2[pp] + grad_2_Gt2[pp]);

        dev_var_out[B0Int + pp] += sigma * (grad_0_B0[pp] + grad_1_B0[pp] + grad_2_B0[pp]);
        dev_var_out[B1Int + pp] += sigma * (grad_0_B1[pp] + grad_1_B1[pp] + grad_2_B1[pp]);
        dev_var_out[B2Int + pp] += sigma * (grad_0_B2[pp] + grad_1_B2[pp] + grad_2_B2[pp]);
    }
}

void get_output (double * dev_var_out, const unsigned int * host_sz, hipStream_t stream,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
) 
{
        const int ie = host_sz[0] - 3;//x direction
        const int je = host_sz[1] - 3;//y direction
        const int ke = host_sz[2] - 3;//z direction

        const unsigned int host_sz_x = host_sz[0];
        const unsigned int host_sz_y = host_sz[1];
        const unsigned int host_sz_z = host_sz[2];

        int total_points = ceil(1.0*ie*je*ke/thread_load_output);
        int blocks = ceil(1.0*total_points/threads_per_block);

        kernal_get_output <<< blocks, threads_per_block, 0, stream >>> (dev_var_out, 
                    host_sz_x, host_sz_y, host_sz_z,
                    #include "list_of_offset_args.h"
                    ,
                    #include "list_of_args.h"
                    );
        
        CHECK_ERROR(hipGetLastError(), "kernal_get_output Kernel launch failed");
}
