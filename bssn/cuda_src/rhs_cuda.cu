#include "hip/hip_runtime.h"
#include "rhs_cuda.h"

#include <iostream>
#include <stdio.h>

enum VAR_CU {U_ALPHA=0,U_CHI,U_K,U_GT0,U_GT1,U_GT2,U_BETA0,U_BETA1,U_BETA2,U_B0,U_B1,U_B2,U_SYMGT0,U_SYMGT1,U_SYMGT2,U_SYMGT3,U_SYMGT4,U_SYMGT5,U_SYMAT0,U_SYMAT1,U_SYMAT2,U_SYMAT3,U_SYMAT4,U_SYMAT5};

__global__ void example_kernal(double * val){
    // Eminda you can use this if it is required
    //test GPU mem values
}

void cuda_bssnrhs(double * dev_var_out, double * dev_var_in, const unsigned int unzip_dof, 
const unsigned int& offset, const double *pmin, const double *pmax, const unsigned int *sz, 
const unsigned int& bflag)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
     cudaStatus = hipSetDevice(0);
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
         return;
     }

    int alphaInt = (VAR_CU::U_ALPHA) * unzip_dof + offset;
    int chiInt = (VAR_CU::U_CHI) * unzip_dof + offset;
    int KInt = (VAR_CU::U_K) * unzip_dof + offset;
    int gt0Int = (VAR_CU::U_SYMGT0) * unzip_dof + offset;
    int gt1Int = (VAR_CU::U_SYMGT1) * unzip_dof + offset;
    int gt2Int =  (VAR_CU::U_SYMGT2) * unzip_dof + offset;
    int gt3Int =(VAR_CU::U_SYMGT3) * unzip_dof + offset;
    int gt4Int = (VAR_CU::U_SYMGT4) * unzip_dof + offset;
    int gt5Int = (VAR_CU::U_SYMGT5) * unzip_dof + offset;
    int beta0Int = (VAR_CU::U_BETA0) * unzip_dof + offset;
    int beta1Int = (VAR_CU::U_BETA1) * unzip_dof + offset;
    int beta2Int =(VAR_CU::U_BETA2) * unzip_dof + offset;
    int At0Int = (VAR_CU::U_SYMAT0) * unzip_dof + offset;
    int At1Int = (VAR_CU::U_SYMAT1) * unzip_dof + offset;
    int At2Int = (VAR_CU::U_SYMAT2) * unzip_dof + offset;
    int At3Int = (VAR_CU::U_SYMAT3) * unzip_dof + offset;
    int At4Int = (VAR_CU::U_SYMAT4) * unzip_dof + offset;
    int At5Int = (VAR_CU::U_SYMAT5) * unzip_dof + offset;
    int Gt0Int = (VAR_CU::U_GT0) * unzip_dof + offset;
    int Gt1Int = (VAR_CU::U_GT1) * unzip_dof + offset;
    int Gt2Int = (VAR_CU::U_GT2) * unzip_dof + offset;
    int B0Int = (VAR_CU::U_B0) * unzip_dof + offset;
    int B1Int = (VAR_CU::U_B1) * unzip_dof + offset;
    int B2Int = (VAR_CU::U_B2) * unzip_dof + offset;

    double hx = (pmax[0] - pmin[0]) / (sz[0] - 1);
    double hy = (pmax[1] - pmin[1]) / (sz[1] - 1);
    double hz = (pmax[2] - pmin[2]) / (sz[2] - 1);

    // Send above values to GPU memory
    #include "bssnrhs_cuda_offset_malloc.h"

    double * dev_dy_hx;
    cudaStatus = hipMalloc((void **) &dev_dy_hx, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hx hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hx, &hx, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hx hipMemcpy failed!\n"); return;}

    double * dev_dy_hy;
    cudaStatus = hipMalloc((void **) &dev_dy_hy, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hy hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hy, &hy, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hy hipMemcpy failed!\n"); return;}

    double * dev_dy_hz;
    cudaStatus = hipMalloc((void **) &dev_dy_hz, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hz hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hz, &hz, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hz hipMemcpy failed!\n"); return;}

    int * dev_sz;
    cudaStatus = hipMalloc((void **) &dev_sz, 3*sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "sz hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_sz, sz, 3*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "sz hipMemcpy failed!\n"); return;}

    int * dev_zero;
    cudaStatus = hipMalloc((void **) &dev_zero, sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "0 hipMalloc failed!\n"); return;}

    // Allocate memory to store the output of derivs
    unsigned int n = sz[0]*sz[1]*sz[2];
    #include "bssnrhs_cuda_malloc.h"

    bssn::timer::t_deriv.start();

    // Deriv calls are follows
    #include "bssnrhs_cuda_derivs.h"

    bssn::timer::t_deriv.stop();


    // Free up GPU memory
    // #include "bssnrhs_cuda_offset_demalloc.h"
    // #include "bssnrhs_cuda_mdealloc.h"
    // hipFree(&dev_dy_hx);
    // hipFree(&dev_dy_hy);
    // hipFree(&dev_dy_hz);
    // hipFree(&dev_sz);
}