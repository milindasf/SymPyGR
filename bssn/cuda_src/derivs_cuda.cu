#include "hip/hip_runtime.h"
/**
 * kernal.cu
 * 
 * Created on: Feb 12, 2018
 * 		Author: Akila
 **/

 #include "derivs_cuda.h"
 #include "rhs.h"
 #include "hip/hip_runtime.h"
 #include ""
 #include <stdio.h>
 
 __global__ void firstThreeFor(double * dev_Dyu, double * dev_u, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*10;
    int y = threadIdx.y + blockIdx.x*10;
    int z = threadIdx.z + blockIdx.x*10;

    int i;
    int j;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[1]-3-3)<=y ){ return; } else { j = x+3; } //j handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = x+3; } //k handler

    int nx = dev_sz[0];
    int ny = dev_sz[1];
    int pp = IDX(i, j, k);//i + dev_sz[0]*(j + dev_sz[1]*k ); //IDX operation (i) + nx * ( (j) + ny * (k) )

    dev_Dyu[pp] = (dev_u[pp-2*dev_sz[0]] - 8.0*dev_u[pp-dev_sz[0]] + 8.0*dev_u[pp+dev_sz[0]] - dev_u[pp+2*dev_sz[0]]);//*((1.0/dev_dy[0])/12.0);

    // printf("%f\n", dev_u[pp-2*dev_sz[0]]);
    
    // const double idy = 1.0/dev_dy[0];
    // const double idy_by_2 = 0.50 * idy;
    // const double idy_by_12 = idy / 12.0;

    // const int nx = sz[0];
    // const int ny = sz[1];
    // const int nz = sz[2];

    
    // const int ib = 3;
    // const int jb = 3;
    // const int kb = 1;

    
    // const int ie = sz[0]-3;
    // const int je = sz[1]-3;
    // const int ke = sz[2]-1;

    // printf("%d\n", y);
    // const double idy = 1.0/dev_dy[0];
    // const double idy_by_2 = 0.50 * idy;
    // const double idy_by_12 = idy / 12.0;
  
    // const int nx = dev_sz[0];
    // const int ny = dev_sz[1];
    // const int nz = dev_sz[2];
    // const int ib = 3;
    // const int jb = 3;
    // const int kb = 1;
    // const int ie = dev_sz[0]-3;
    // const int je = dev_sz[1]-3;
    // const int ke = dev_sz[2]-1;
  
    // const int n=nx; 

 }
 
 
 void deriv42_yWithCuda(double * const  Dyu, const double * const u,
    const double dy, const unsigned int *sz, unsigned bflag)
 {
    double * dev_Dyu;
    double * dev_u;
    int * dev_sz;
    double * dev_dy;

    hipMalloc((void **) &dev_Dyu, sizeof(Dyu)*sizeof(double));
    hipMalloc((void **) &dev_u, sizeof(u)*sizeof(double));
    hipMalloc((void **) &dev_dy, sizeof(double));
    hipMalloc((void **) &dev_sz, 3*sizeof(int));

    // printf("%f\n", *u);
    // printf("%d\n", (int)sizeof(u));

    hipMemcpy(dev_Dyu, Dyu, sizeof(Dyu)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_u, &u, sizeof(u)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_dy, &dy, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_sz, sz, 3*sizeof(int), hipMemcpyHostToDevice);
  
    // std::cout << "real val " << dy <<std::endl;
    // dim3(ke-kb, ie-ib, je-jb)
    const double idy = 1.0/dy;
    const double idy_by_2 = 0.50 * idy;
    const double idy_by_12 = idy / 12.0;

    const int nx = sz[0];
    const int ny = sz[1];
    const int nz = sz[2];
    const int ib = 3;
    const int jb = 3;
    const int kb = 1;
    const int ie = sz[0]-3;
    const int je = sz[1]-3;
    const int ke = sz[2]-1;

    const int n=nx;


    int zblocks = ((sz[2]-1)/10)+1;
    int yblocks = ((sz[0]-3)/10)+1;
    int xblocks = ((sz[1]-3)/10)+1;
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    firstThreeFor<<< max, dim3(10, 10, 10) >>>(dev_Dyu, dev_u, dev_dy, dev_sz);

    // for (int k = kb; k < ke; k++) {
    //     for (int i = ib; i < ie; i++) {
    //       for (int j = jb; j < je; j++) {
    //         int pp = IDX(i,j,k); //(i) + nx * ( (j) + ny * (k) )
    //         Dyu[pp] = (u[pp-2*nx] - 8.0*u[pp-nx] + 8.0*u[pp+nx] - u[pp+2*nx])*idy_by_12;
    //         // printf("%f\n", u[0]);
    //       }
    //     }
    //   }
    
    //   if (bflag & (1u<<OCT_DIR_DOWN)) {
    //     for (int k = kb; k < ke; k++) {
    //       for (int i = ib; i < ie; i++) {
    //         Dyu[IDX(i, 3,k)] = ( - 3.0 * u[IDX(i,3,k)]
    //                             +  4.0 * u[IDX(i,4,k)]
    //                             -        u[IDX(i,5,k)]
    //                           ) * idy_by_2;
    
    //         Dyu[IDX(i,4,k)] = ( - u[IDX(i,3,k)]
    //                             + u[IDX(i,5,k)]
    //                           ) * idy_by_2;
    //       }
    //     }
    //   }
    
    //   if (bflag & (1u<<OCT_DIR_UP)) {
    //     for (int k = kb; k < ke; k++) {
    //       for (int i = ib; i < ie; i++) {
    //         Dyu[IDX(i,je-2,k)] = ( - u[IDX(i,je-3,k)]
    //                                + u[IDX(i,je-1,k)]
    //                              ) * idy_by_2;
    
    //         Dyu[IDX(i,je-1,k)] = (        u[IDX(i,je-3,k)]
    //                               - 4.0 * u[IDX(i,je-2,k)]
    //                               + 3.0 * u[IDX(i,je-1,k)]
    //                           ) * idy_by_2;
    //       }
    //     }
    //   } 
    hipFree(&dev_Dyu);
    hipFree(&dev_u);
    hipFree(&dev_sz);
    hipFree(&dev_dy);
    
    // #ifdef DEBUG_DERIVS_COMP
    //   for (int k = 3; k < sz[2]-3; k++) {
    //     for (int j = 3; j < sz[1]-3; j++) {
    //       for (int i = 3; i < sz[0]-3; i++) {
    //         int pp = IDX(i,j,k);
    //         if(std::isnan(Dyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //       }
    //     }
    //   }
    // #endif
    
    
 
//      int *dev_a = 0;
//      int *dev_b = 0;
//      int *dev_c = 0;
//      hipError_t cudaStatus;
 
//      // Choose which GPU to run on, change this on a multi-GPU system.
//      cudaStatus = hipSetDevice(0);
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//          goto Error;
//      }
 
//      // Allocate GPU buffers for three vectors (two input, one output)    .
//      cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMalloc failed!");
//          goto Error;
//      }
 
//      cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMalloc failed!");
//          goto Error;
//      }
 
//      cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMalloc failed!");
//          goto Error;
//      }
 
//      // Copy input vectors from host memory to GPU buffers.
//      cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMemcpy failed!");
//          goto Error;
//      }
 
//      cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMemcpy failed!");
//          goto Error;
//      }
 
//      // Launch a kernel on the GPU with one thread for each element.
//      addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
 
//      // Check for any errors launching the kernel
//      cudaStatus = hipGetLastError();
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//          goto Error;
//      }
 
//      // hipDeviceSynchronize waits for the kernel to finish, and returns
//      // any errors encountered during the launch.
//      cudaStatus = hipDeviceSynchronize();
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//          goto Error;
//      }
 
//      // Copy output vector from GPU buffer to host memory.
//      cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMemcpy failed!");
//          goto Error;
//      }
 
//  Error:
//      hipFree(dev_c);
//      hipFree(dev_a);
//      hipFree(dev_b);
 
//      //return 0;
 }
 
 