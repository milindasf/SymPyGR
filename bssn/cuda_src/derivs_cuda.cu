#include "hip/hip_runtime.h"
/**
 * kernal.cu
 * 
 * Created on: Feb 12, 2018
 * 		Author: Eminda, Akila, Eranga, Ruwan
 **/

 #include "derivs_cuda.h"
 #include "rhs.h"
 #include "hip/hip_runtime.h"
 #include ""
 #include <stdio.h>
 
 __global__ void firstThreeFor(int *c, const int *a, const int *b)
 {
     int i = threadIdx.x;
     printf ("dff %d",i);
     c[i] = a[i] + b[i];
 }
 
 
 void deriv42_yWithCuda(double * const  Dyu, const double * const  u,
    const double dy, const unsigned int *sz, unsigned bflag)
 {
    const double idy = 1.0/dy;
    const double idy_by_2 = 0.50 * idy;
    const double idy_by_12 = idy / 12.0;
  
    const int nx = sz[0];
    const int ny = sz[1];
    const int nz = sz[2];
    const int ib = 3;
    const int jb = 3;
    const int kb = 1;
    const int ie = sz[0]-3;
    const int je = sz[1]-3;
    const int ke = sz[2]-1;
  
    const int n=nx;
    std::cout << ie <<std::endl;
    for (int k = kb; k < ke; k++) {
        for (int i = ib; i < ie; i++) {
          for (int j = jb; j < je; j++) {
            int pp = IDX(i,j,k); //(i) + nx * ( (j) + ny * (k) )
            Dyu[pp] = (u[pp-2*nx] - 8.0*u[pp-nx] + 8.0*u[pp+nx] - u[pp+2*nx])*idy_by_12;
          }
        }
      }
    
      if (bflag & (1u<<OCT_DIR_DOWN)) {
        for (int k = kb; k < ke; k++) {
          for (int i = ib; i < ie; i++) {
            Dyu[IDX(i, 3,k)] = ( - 3.0 * u[IDX(i,3,k)]
                                +  4.0 * u[IDX(i,4,k)]
                                -        u[IDX(i,5,k)]
                              ) * idy_by_2;
    
            Dyu[IDX(i,4,k)] = ( - u[IDX(i,3,k)]
                                + u[IDX(i,5,k)]
                              ) * idy_by_2;
          }
        }
      }
    
      if (bflag & (1u<<OCT_DIR_UP)) {
        for (int k = kb; k < ke; k++) {
          for (int i = ib; i < ie; i++) {
            Dyu[IDX(i,je-2,k)] = ( - u[IDX(i,je-3,k)]
                                   + u[IDX(i,je-1,k)]
                                 ) * idy_by_2;
    
            Dyu[IDX(i,je-1,k)] = (        u[IDX(i,je-3,k)]
                                  - 4.0 * u[IDX(i,je-2,k)]
                                  + 3.0 * u[IDX(i,je-1,k)]
                              ) * idy_by_2;
          }
        }
      }
    
    // #ifdef DEBUG_DERIVS_COMP
    //   for (int k = 3; k < sz[2]-3; k++) {
    //     for (int j = 3; j < sz[1]-3; j++) {
    //       for (int i = 3; i < sz[0]-3; i++) {
    //         int pp = IDX(i,j,k);
    //         if(std::isnan(Dyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //       }
    //     }
    //   }
    // #endif
    
    
 
//      int *dev_a = 0;
//      int *dev_b = 0;
//      int *dev_c = 0;
//      hipError_t cudaStatus;
 
//      // Choose which GPU to run on, change this on a multi-GPU system.
//      cudaStatus = hipSetDevice(0);
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//          goto Error;
//      }
 
//      // Allocate GPU buffers for three vectors (two input, one output)    .
//      cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMalloc failed!");
//          goto Error;
//      }
 
//      cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMalloc failed!");
//          goto Error;
//      }
 
//      cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMalloc failed!");
//          goto Error;
//      }
 
//      // Copy input vectors from host memory to GPU buffers.
//      cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMemcpy failed!");
//          goto Error;
//      }
 
//      cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMemcpy failed!");
//          goto Error;
//      }
 
//      // Launch a kernel on the GPU with one thread for each element.
//      addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
 
//      // Check for any errors launching the kernel
//      cudaStatus = hipGetLastError();
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//          goto Error;
//      }
 
//      // hipDeviceSynchronize waits for the kernel to finish, and returns
//      // any errors encountered during the launch.
//      cudaStatus = hipDeviceSynchronize();
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//          goto Error;
//      }
 
//      // Copy output vector from GPU buffer to host memory.
//      cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//      if (cudaStatus != hipSuccess) {
//          fprintf(stderr, "hipMemcpy failed!");
//          goto Error;
//      }
 
//  Error:
//      hipFree(dev_c);
//      hipFree(dev_a);
//      hipFree(dev_b);
 
//      //return 0;
 }
 
 