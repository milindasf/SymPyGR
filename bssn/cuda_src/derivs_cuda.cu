
#include <hip/hip_runtime.h>
// /**
//  * Created on: March 15, 2018
//  * 		Author: Akila
//  **/

// //  #include "derivs_cuda.h"
 
//  __global__ void calc_deriv42_x(double * output, double * dev_var_in, 
//         const int u_offset, double dx, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
//  {
//     int thread_id = blockIdx.x*threads_per_block_deriv + threadIdx.x;

//     for (int id = thread_id*thread_load_deriv; id<(thread_id+1)*thread_load_deriv; id++){
            
//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-2)) + 1;
//         int k = (id/(host_sz_z-2)/(host_sz_x-6)) + 1; 

//         if (k>=host_sz_z-1) return;

//         int nx = host_sz_x; 
//         int ny = host_sz_y; 

//         int pp = IDX(i, j, k);

//         const double idx = 1.0/dx;
//         const double idx_by_2 = 0.50 * idx;
//         const double idx_by_12 = idx / 12.0;

//         output[pp] = (dev_var_in[(u_offset) + pp - 2] - 8.0*dev_var_in[(u_offset)
//                         + pp - 1] + 8.0*dev_var_in[(u_offset) + pp + 1] 
//                         - dev_var_in[(u_offset) + pp + 2] )*idx_by_12;

//         if ((bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
//             int pp3 = IDX(3, j, k);
//             int pp4 = IDX(4, j, k);
//             int pp5 = IDX(5, j, k);
//             output[pp3] = ((-3)*dev_var_in[(u_offset) + pp3] + 4*dev_var_in[(u_offset) 
//                         + pp4] - dev_var_in[(u_offset) + pp5]) * idx_by_2;
//             output[pp4] = (dev_var_in[(u_offset) + pp5] - dev_var_in[(u_offset) 
//                         + pp3]) * idx_by_2;
//         }

//         if ((bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
//             int pp2 = IDX(nx-5, j, k); // IDX(ie-2,j,k)
//             int pp3 = IDX(nx-6, j, k); // IDX(ie-3,j,k)
//             int pp1 = IDX(nx-4,j,k); // IDX(ie-1,j,k)
//             output[pp2] = (dev_var_in[(u_offset) + pp1] - dev_var_in[(u_offset) + pp3]) 
//                         * idx_by_2;

//             output[pp1] = (dev_var_in[(u_offset) + pp3]- 4.0 * dev_var_in[(u_offset) + pp2]
//                         + 3.0 * dev_var_in[(u_offset) + pp1]) * idx_by_2;

//         }
//     }
    
//  }

//  void cuda_deriv42_x(double * output, double * dev_var_in, int u_offset, double dx, 
//     int bflag, const unsigned int * host_sz, cudaStream_t stream)
//  {
//     const int ib = 3;
//     const int jb = 1;
//     const int kb = 1;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 1;
//     const int ke = host_sz[2] - 1;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];
 
//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_deriv);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_deriv);
  
//     calc_deriv42_x <<< number_of_blocks, threads_per_block_deriv, 0, stream>>> (output, dev_var_in, u_offset, dx, host_sz_x, host_sz_y, host_sz_z, bflag);
     
//     CHECK_ERROR(cudaGetLastError(), "calc_deriv42_x Kernel launch failed");
//  }

// __global__ void calc_deriv42_y(double* output, double * dev_var_in, 
//     const int u_offset, double dy, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
// {
//     int thread_id = blockIdx.x*threads_per_block_deriv + threadIdx.x;

//     for (int id = thread_id*thread_load_deriv; id<(thread_id+1)*thread_load_deriv; id++){
    
//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 1;

//         if (k>=host_sz_z-1) return;

//         int nx = host_sz_x; 
//         int ny = host_sz_y; 
        
//         int pp = IDX(i, j, k);

//         const double idy = 1.0/dy;
//         const double idy_by_2 = 0.50 * idy;
//         const double idy_by_12 = idy / 12.0;

//         output[pp] = (dev_var_in[u_offset + pp - 2*nx] 
//                     - 8.0*dev_var_in[u_offset + pp - nx] 
//                     + 8.0*dev_var_in[u_offset + pp + nx] 
//                     - dev_var_in[u_offset + pp + 2*nx] )*idy_by_12;
        
                
//         if ((bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
//             int pp3 = IDX(i, 3, k);
//             int pp4 = IDX(i, 4, k);
//             int pp5 = IDX(i, 5, k);

//             output[pp3] = ((-3)*dev_var_in[(u_offset) + pp3] +  4*dev_var_in[(u_offset) + pp4] 
//                         - dev_var_in[(u_offset) + pp5]) * idy_by_2;
//             output[pp4] = (dev_var_in[(u_offset) + pp5] - dev_var_in[(u_offset) + pp3]) 
//                         * idy_by_2;
            
//         }

//         if ((bflag & (1u<<OCT_DIR_UP)) && j==4)  {
//             int pp2 = IDX(i, ny-5, k); // IDX(i,je-2,k)
//             int pp3 = IDX(i, ny-6, k); // IDX(i,je-3,k)
//             int pp1 = IDX(i, ny-4, k); // IDX(i,je-1,k)
        
//             output[pp2] = (dev_var_in[(u_offset) + pp1] - dev_var_in[(u_offset) + pp3]) 
//                         * idy_by_2;
//             output[pp1] = (dev_var_in[(u_offset) + pp3]- 4.0 * dev_var_in[(u_offset) + pp2]
//                         + 3.0 * dev_var_in[(u_offset) + pp1]) * idy_by_2;
            
//         }
//     }
// }

// void cuda_deriv42_y(double * output, double * dev_var_in, int u_offset, double dy, 
//                 int bflag, const unsigned int * host_sz, cudaStream_t stream)
//  {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 1;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 1;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];
  
//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_deriv);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_deriv);
      
//     calc_deriv42_y <<< number_of_blocks, threads_per_block_deriv, 0, stream >>> (output, dev_var_in, u_offset, dy, host_sz_x, host_sz_y, host_sz_z, bflag);

//     CHECK_ERROR(cudaGetLastError(), "calc_deriv42_y Kernel launch failed");
//  }

// __global__ void calc_deriv42_z(double* output, double * dev_var_in, 
//     const int u_offset, double dz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
// {
//     int thread_id = blockIdx.x*threads_per_block_deriv + threadIdx.x;

//     for (int id = thread_id*thread_load_deriv; id<(thread_id+1)*thread_load_deriv; id++){
            
//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;

//         if (k>=host_sz_z-3) return;

//         int nx = host_sz_x; 
//         int ny = host_sz_y; 

//         int n = nx * ny;
//         int pp = IDX(i, j, k);

//         const double idz = 1.0/dz;
//         const double idz_by_2 = 0.50 * idz;
//         const double idz_by_12 = idz / 12.0;
    
//     output[pp] = (dev_var_in[(u_offset) + pp - 2*n] - 8.0*dev_var_in[(u_offset) + pp - n] 
//                     + 8.0*dev_var_in[(u_offset) + pp + n] - dev_var_in[(u_offset) + pp + 2*n]) 
//                     * idz_by_12;
        
//         if ((bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
//             int pp3 = IDX(i, j, 3); // IDX(i, j, 3)
//             int pp4 = IDX(i, j, 4); // IDX(i,j,4)
//             int pp5 = IDX(i, j, 5); // IDX(i,j,5)

//             output[pp3] = ((-3)*dev_var_in[(u_offset) + pp3] + 4*dev_var_in[(u_offset) + pp4] 
//                         - dev_var_in[(u_offset) + pp5]) * idz_by_2;
//             output[pp4] = (dev_var_in[(u_offset) + pp5] - dev_var_in[(u_offset) + pp3])
//                         * idz_by_2;
//         }
                
//         if ((bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
//             int pp2 = IDX(i, j, host_sz_z-5); // IDX(i,j,ke-2)
//             int pp3 = IDX(i, j, host_sz_z-6); // IDX(i,j,ke-3)
//             int pp1 = IDX(i, j, host_sz_z-4); // IDX(i,j,ke-1)

//             output[pp2] = (dev_var_in[(u_offset) + pp1] - dev_var_in[(u_offset) + pp3]) 
//                         * idz_by_2;
//             output[pp1] = (dev_var_in[(u_offset) + pp3]- 4.0 * dev_var_in[(u_offset) + pp2]
//                         + 3.0 * dev_var_in[(u_offset) + pp1]) * idz_by_2;
//         }
//     }
// }

// void cuda_deriv42_z(double * output, double * dev_var_in, int u_offset, double dz, 
//     int bflag, const unsigned int * host_sz, cudaStream_t stream)
// {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 3;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 3;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];
  
//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_deriv);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_deriv);
      
//     calc_deriv42_z <<< number_of_blocks, threads_per_block_deriv, 0, stream >>> (output, dev_var_in, u_offset, dz, host_sz_x, host_sz_y, host_sz_z, bflag);

//     CHECK_ERROR(cudaGetLastError(), "calc_deriv42_z Kernel launch failed");
// }

// __global__ void calc_deriv42_xx(double* output, double * dev_var_in, 
//     const int u_offset, double dx, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
//  {
//     int thread_id = blockIdx.x*threads_per_block_deriv_sec_order + threadIdx.x;

//     for (int id = thread_id*thread_load_deriv_sec_order; id<(thread_id+1)*thread_load_deriv_sec_order; id++){
    
//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;

//         if (k>=host_sz_z-3) return;

//         int nx = host_sz_x; 
//         int ny = host_sz_y; 

//         int pp = IDX(i, j, k);

//         const double idx_sqrd = 1.0/(dx*dx);
//         const double idx_sqrd_by_12 = idx_sqrd / 12.0;

//         output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2] 
//                     + 16.0*dev_var_in[(u_offset) + pp - 1] 
//                     - 30.0*dev_var_in[(u_offset) + pp] 
//                     + 16.0*dev_var_in[(u_offset) + pp + 1] 
//                     - dev_var_in[(u_offset) + pp + 2] 
//                 )*idx_sqrd_by_12;
                
//         if ((bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
//             int pp3 = IDX(3, j, k); 
//             int pp4 = IDX(4, j, k); 
//             int pp5 = IDX(5, j, k); 
//             int pp6 = IDX(6, j, k); 
        
//             output[pp3] = (
//                     2.0 *   dev_var_in[(u_offset) + pp3] 
//                 -   5.0 *   dev_var_in[(u_offset) + pp4] 
//                 +   4.0 *   dev_var_in[(u_offset) + pp5] 
//                 -           dev_var_in[(u_offset) + pp6]
//                 ) * idx_sqrd;
        
//             output[pp4] = (
//                             dev_var_in[(u_offset) + pp3]
//                 -   2.0 *   dev_var_in[(u_offset) + pp4]
//                 +           dev_var_in[(u_offset) + pp5]
//             ) * idx_sqrd;
        
//         }
                        
//         if ((bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
//             int pp1 = IDX(host_sz_x - 4, j, k); // IDX(ie-1,j,k)
//             int pp2 = IDX(host_sz_x - 5, j, k); // IDX(ie-2,j,k)
//             int pp3 = IDX(host_sz_x - 6, j, k); // IDX(ie-3,j,k)
//             int pp4 = IDX(host_sz_x - 7, j, k); // IDX(ie-4,j,k)

//             output[pp2] = (
//                                 dev_var_in[(u_offset) + pp3] 
//                     -   2.0 *   dev_var_in[(u_offset) + pp2] 
//                     +           dev_var_in[(u_offset) + pp1] 
//                     ) * idx_sqrd;


//                 output[pp1] = (
//                     -   1.0 *   dev_var_in[(u_offset) + pp4] 
//                     +   4.0 *   dev_var_in[(u_offset) + pp3] 
//                     -   5.0 *   dev_var_in[(u_offset) + pp2] 
//                     +   2.0 *   dev_var_in[(u_offset) + pp1]
//                     ) * idx_sqrd;
//         }
//     }
// }

// void cuda_deriv42_xx(double * output, double * dev_var_in, int u_offset, double dx, 
//     int bflag, const unsigned int * host_sz, cudaStream_t stream)
// {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 3;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 3;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];
  
//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_deriv_sec_order);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_deriv_sec_order);
     
//     calc_deriv42_xx <<< number_of_blocks, threads_per_block_deriv_sec_order, 0, stream >>> (output, dev_var_in, u_offset, dx, host_sz_x, host_sz_y, host_sz_z, bflag);

//     CHECK_ERROR(cudaGetLastError(), "calc_deriv42_xx Kernel launch failed");
// }

// __global__ void calc_deriv42_yy(double* output, double * dev_var_in, 
//     const int u_offset, double dy, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
//  {
//     int thread_id = blockIdx.x*threads_per_block_deriv_sec_order + threadIdx.x;

//     for (int id = thread_id*thread_load_deriv_sec_order; id<(thread_id+1)*thread_load_deriv_sec_order; id++){
        
//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;

//         if (k>=host_sz_z-3) return;

//         int nx = host_sz_x; 
//         int ny = host_sz_y; 

//         int pp = IDX(i, j, k);

//         const double idy_sqrd = 1.0/(dy*dy);
//         const double idy_sqrd_by_12 = idy_sqrd / 12.0;

//         output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2*nx] 
//                     + 16.0*dev_var_in[(u_offset) + pp - nx] 
//                     - 30.0*dev_var_in[(u_offset) + pp] 
//                     + 16.0*dev_var_in[(u_offset) + pp + nx] 
//                     - dev_var_in[(u_offset) + pp + 2*nx] 
//                 )*idy_sqrd_by_12;
                
//         if ((bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
//             int pp3 = IDX(i, 3, k); 
//             int pp4 = IDX(i, 4, k); 
//             int pp5 = IDX(i, 5, k); 
//             int pp6 = IDX(i, 6, k); 
        
//             output[pp3] = (
//                     2.0 *   dev_var_in[(u_offset) + pp3] 
//                 -   5.0 *   dev_var_in[(u_offset) + pp4] 
//                 +   4.0 *   dev_var_in[(u_offset) + pp5] 
//                 -           dev_var_in[(u_offset) + pp6]
//                 ) * idy_sqrd;
        
//             output[pp4] = (
//                             dev_var_in[(u_offset) + pp3]
//                 -   2.0 *   dev_var_in[(u_offset) + pp4]
//                 +           dev_var_in[(u_offset) + pp5]
//             ) * idy_sqrd;
//         }
                                
//         if ((bflag & (1u<<OCT_DIR_UP)) && j==4)  {
//             int pp1 = IDX(i, host_sz_y - 4, k); 
//             int pp2 = IDX(i, host_sz_y - 5, k); 
//             int pp3 = IDX(i, host_sz_y - 6, k); 
//             int pp4 = IDX(i, host_sz_y - 7, k); 
        
//             output[pp2] = (
//                             dev_var_in[(u_offset) + pp3] 
//                 -   2.0 *   dev_var_in[(u_offset) + pp2] 
//                 +           dev_var_in[(u_offset) + pp1] 
//                 ) * idy_sqrd;
        
        
//             output[pp1] = (
//                 -   1.0 *   dev_var_in[(u_offset) + pp4] 
//                 +   4.0 *   dev_var_in[(u_offset) + pp3] 
//                 -   5.0 *   dev_var_in[(u_offset) + pp2] 
//                 +   2.0 *   dev_var_in[(u_offset) + pp1]
//                 ) * idy_sqrd;
        
//         }
//     }
    
// }

// void cuda_deriv42_yy(double * output, double * dev_var_in, int u_offset, double dy, 
//     int bflag, const unsigned int * host_sz, cudaStream_t stream)
// {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 3;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 3;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];
  
//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_deriv_sec_order);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_deriv_sec_order);
     
//     calc_deriv42_yy <<< number_of_blocks, threads_per_block_deriv_sec_order, 0, stream >>> (output, dev_var_in, u_offset, dy, host_sz_x, host_sz_y, host_sz_z, bflag);           

//     CHECK_ERROR(cudaGetLastError(), "calc_deriv42_yy Kernel launch failed");
// }

// __global__ void calc_deriv42_zz(double* output, double * dev_var_in, 
//     const int u_offset, double dz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
//  {
//     int thread_id = blockIdx.x*threads_per_block_deriv_sec_order + threadIdx.x;

//     for (int id = thread_id*thread_load_deriv_sec_order; id<(thread_id+1)*thread_load_deriv_sec_order; id++){
//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;

//         if (k>=host_sz_z-3) return;

//         int nx = host_sz_x; 
//         int ny = host_sz_y; 

//         int pp = IDX(i, j, k);
//         int n = nx * ny;

//         const double idz_sqrd = 1.0/(dz*dz);
//         const double idz_sqrd_by_12 = idz_sqrd / 12.0;

//         output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2*n] 
//                     + 16.0*dev_var_in[(u_offset) + pp - n] 
//                     - 30.0*dev_var_in[(u_offset) + pp] 
//                     + 16.0*dev_var_in[(u_offset) + pp + n] 
//                     - dev_var_in[(u_offset) + pp + 2*n] 
//                 )*idz_sqrd_by_12;

//         if ((bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
//             int pp3 = IDX(i, j, 3); 
//             int pp4 = IDX(i, j, 4); 
//             int pp5 = IDX(i, j, 5); 
//             int pp6 = IDX(i, j, 6); 
        
//             output[pp3] = (
//                     2.0 *   dev_var_in[(u_offset) + pp3] 
//                 -   5.0 *   dev_var_in[(u_offset) + pp4] 
//                 +   4.0 *   dev_var_in[(u_offset) + pp5] 
//                 -           dev_var_in[(u_offset) + pp6]
//                 ) * idz_sqrd;
        
//             output[pp4] = (
//                             dev_var_in[(u_offset) + pp3]
//                 -   2.0 *   dev_var_in[(u_offset) + pp4]
//                 +           dev_var_in[(u_offset) + pp5]
//             ) * idz_sqrd;
//         }
                                        
//         if ((bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
//             int pp1 = IDX(i, j, host_sz_z - 4); 
//             int pp2 = IDX(i, j, host_sz_z - 5); 
//             int pp3 = IDX(i, j, host_sz_z - 6); 
//             int pp4 = IDX(i, j, host_sz_z - 7); 

//             output[pp2] = (
//                                 dev_var_in[(u_offset) + pp3] 
//                     -   2.0 *   dev_var_in[(u_offset) + pp2] 
//                     +           dev_var_in[(u_offset) + pp1] 
//                     ) * idz_sqrd;


//                 output[pp1] = (
//                     -   1.0 *   dev_var_in[(u_offset) + pp4] 
//                     +   4.0 *   dev_var_in[(u_offset) + pp3] 
//                     -   5.0 *   dev_var_in[(u_offset) + pp2] 
//                     +   2.0 *   dev_var_in[(u_offset) + pp1]
//                     ) * idz_sqrd;
//         }
//     }
// }


// void cuda_deriv42_zz(double * output, double * dev_var_in, int u_offset, double dz, 
//     int bflag, const unsigned int * host_sz, cudaStream_t stream)
// {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 3;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 3;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];
  
//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_deriv_sec_order);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_deriv_sec_order);
     
//     calc_deriv42_zz <<< number_of_blocks, threads_per_block_deriv_sec_order, 0, stream >>> (output, dev_var_in, u_offset, dz, host_sz_x, host_sz_y, host_sz_z, bflag);           

//     CHECK_ERROR(cudaGetLastError(), "calc_deriv42_zz Kernel launch failed");
// }

// __global__ void calc_deriv42_adv_x(double * output, double * dev_var_in, 
// int betax, double dx, int bflag, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int u_offset) 
// {
//     int thread_id = blockIdx.x*threads_per_block_adv_deriv + threadIdx.x;
    
//     for (int id = thread_id*thread_load_adv_deriv; id<(thread_id+1)*thread_load_adv_deriv; id++){

//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;

//         if (k>=host_sz_z-3) return;

//         int nx = host_sz_x;
//         int ny = host_sz_y;

//         int pp = IDX(i, j, k);

//         const double idx = 1.0/dx;
//         const double idx_by_2 = 0.50 * idx;
//         const double idx_by_12 = idx / 12.0;

//         if (dev_var_in[betax + pp] > 0.0 ) {
//             output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - 1]
//                         - 10.0 * dev_var_in[u_offset + pp]
//                         + 18.0 * dev_var_in[u_offset + pp + 1]
//                         -  6.0 * dev_var_in[u_offset + pp + 2]
//                         +        dev_var_in[u_offset + pp + 3]
//                     ) * idx_by_12;
//         }
//         else {
//             output[pp] = ( -        dev_var_in[u_offset + pp - 3]
//                         +  6.0 * dev_var_in[u_offset + pp - 2]
//                         - 18.0 * dev_var_in[u_offset + pp - 1]
//                         + 10.0 * dev_var_in[u_offset + pp]
//                         +  3.0 * dev_var_in[u_offset + pp +1]
//                     ) * idx_by_12;
//         }
        
//         if ((bflag & (1u<<OCT_DIR_LEFT)) && (i == 3)) {
            
//             output[IDX(3,j,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(3,j,k)]
//                     +  4.0 * dev_var_in[u_offset + IDX(4,j,k)]
//                     -        dev_var_in[u_offset + IDX(5,j,k)]
//                     ) * idx_by_2;

//             if (dev_var_in[betax + IDX(4,j,k)] > 0.0) {
//                 output[IDX(4,j,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(4,j,k)]
//                                 +  4.0 * dev_var_in[u_offset + IDX(5,j,k)]
//                                 -        dev_var_in[u_offset + IDX(6,j,k)]
//                             ) * idx_by_2;
//             }
//             else {
//                 output[IDX(4,j,k)] = ( -         dev_var_in[u_offset + IDX(3,j,k)]
//                                 +        dev_var_in[u_offset + IDX(5,j,k)]
//                             ) * idx_by_2;
//             }

//             if (dev_var_in[betax + IDX(5,j,k)] > 0.0 ) {
//                 output[IDX(5,j,k)] = (-  3.0 * dev_var_in[u_offset + IDX(4,j,k)]
//                             - 10.0 * dev_var_in[u_offset + IDX(5,j,k)]
//                             + 18.0 * dev_var_in[u_offset + IDX(6,j,k)]
//                             -  6.0 * dev_var_in[u_offset + IDX(7,j,k)]
//                             +        dev_var_in[u_offset + IDX(8,j,k)]
//                             ) * idx_by_12;
//             }
//             else {
//                 output[IDX(5,j,k)] = (           dev_var_in[u_offset + IDX(3,j,k)]
//                                 -  4.0 * dev_var_in[u_offset + IDX(4,j,k)]
//                                 +  3.0 * dev_var_in[u_offset + IDX(5,j,k)]
//                             ) * idx_by_2;
//             }
//         }

//         if ((bflag & (1u<<OCT_DIR_RIGHT)) && (i == 4)) {
            
//             const int ie = nx - 3;
            
//             if ( dev_var_in[betax + IDX(ie-3,j,k)] < 0.0 ) {
//                 output[IDX(ie-3,j,k)] = (  - 3.0 * dev_var_in[u_offset + IDX(ie-3,j,k)]
//                                         + 4.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
//                                         -       dev_var_in[u_offset + IDX(ie-1,j,k)]
//                                     ) * idx_by_2;
//             }
//             else {
//                 output[IDX(ie-3,j,k)] = ( -   dev_var_in[u_offset + IDX(ie-6,j,k)]
//                                 +  6.0 * dev_var_in[u_offset + IDX(ie-5,j,k)]
//                                 - 18.0 * dev_var_in[u_offset + IDX(ie-4,j,k)]
//                                 + 10.0 * dev_var_in[u_offset + IDX(ie-3  ,j,k)]
//                                 +  3.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
//                                 ) * idx_by_12;
//             }
    
//             if (dev_var_in[betax + IDX(ie-2,j,k)] > 0.0 ) {
//                 output[IDX(ie-2,j,k)] = (  -  dev_var_in[u_offset + IDX(ie-3,j,k)]
//                                         +  dev_var_in[u_offset + IDX(ie-1,j,k)]
//                                     ) * idx_by_2;
//             }
//             else {
//                 output[IDX(ie-2,j,k)] = (     dev_var_in[u_offset + IDX(ie-4,j,k)]
//                                 - 4.0 * dev_var_in[u_offset + IDX(ie-3,j,k)]
//                                 + 3.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
//                                     ) * idx_by_2;
//             }
    
//             output[IDX(ie-1,j,k)] = (          dev_var_in[u_offset + IDX(ie-3,j,k)]
//                                     - 4.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
//                                     + 3.0 * dev_var_in[u_offset + IDX(ie-1,j,k)]
//                                 ) * idx_by_2;
//         }
//     }
// }

// void cuda_deriv42_adv_x(double * output, double * dev_var_in, 
//     int u_offset, double dx, int betax, int bflag, const unsigned int * host_sz, cudaStream_t stream)
// {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 3;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 3;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];
    
//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_adv_deriv);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_adv_deriv);
  
//     calc_deriv42_adv_x <<< number_of_blocks, threads_per_block_adv_deriv, 0, stream >>> (output, dev_var_in, betax, dx, bflag, host_sz_x, host_sz_y, host_sz_z, u_offset);

//     CHECK_ERROR(cudaGetLastError(), "calc_deriv42_adv_x Kernel launch failed");
// }

// __global__ void calc_deriv42_adv_y(double * output, double * dev_var_in, 
//     int betay, double dy, int bflag, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int u_offset) 
// {
//     int thread_id = blockIdx.x*threads_per_block_adv_deriv + threadIdx.x;

//     for (int id = thread_id*thread_load_adv_deriv; id<(thread_id+1)*thread_load_adv_deriv; id++){
            
//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    
//         if (k>=host_sz_z-3) return;

//         int nx = host_sz_x;
//         int ny = host_sz_y;
        
//         const double idy = 1.0/dy;
//         const double idy_by_2 = 0.50 * idy;
//         const double idy_by_12 = idy / 12.0;
        
//         int pp = IDX(i, j, k);

//         if (dev_var_in[betay + pp] > 0.0 ) {
//                 output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - nx]
//                             - 10.0 * dev_var_in[u_offset + pp]
//                             + 18.0 * dev_var_in[u_offset + pp + nx]
//                             -  6.0 * dev_var_in[u_offset + pp + 2*nx]
//                             +        dev_var_in[u_offset + pp + 3*nx]
//                         ) * idy_by_12;
//         }
//         else {
//             output[pp] = ( -        dev_var_in[u_offset + pp - 3*nx]
//                         +  6.0 * dev_var_in[u_offset + pp - 2*nx]
//                         - 18.0 * dev_var_in[u_offset + pp - nx]
//                         + 10.0 * dev_var_in[u_offset + pp]
//                         +  3.0 * dev_var_in[u_offset + pp +nx]
//                         ) * idy_by_12;
                    
//         }
        
//         if ((bflag & (1u<<OCT_DIR_DOWN)) && (j == 3)) {
            
//             output[IDX(i,3,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,3,k)]
//                     +  4.0 * dev_var_in[u_offset + IDX(i,4,k)]
//                     -        dev_var_in[u_offset + IDX(i,5,k)]
//                     ) * idy_by_2;
                    
//             if (dev_var_in[betay + IDX(i,4,k)] > 0.0) {
//                 output[IDX(i,4,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,4,k)]
//                                 +  4.0 * dev_var_in[u_offset + IDX(i,5,k)]
//                                 -        dev_var_in[u_offset + IDX(i,6,k)]
//                             ) * idy_by_2;

//             }
//             else {
//                 output[IDX(i,4,k)] = ( -         dev_var_in[u_offset + IDX(i,3,k)]
//                                 +        dev_var_in[u_offset + IDX(i,5,k)]
//                             ) * idy_by_2;
                            
//             }

//             if (dev_var_in[betay + IDX(i,5,k)] > 0.0 ) {
//                 output[IDX(i,5,k)] = (-  3.0 * dev_var_in[u_offset + IDX(i,4,k)]
//                             - 10.0 * dev_var_in[u_offset + IDX(i,5,k)]
//                             + 18.0 * dev_var_in[u_offset + IDX(i,6,k)]
//                             -  6.0 * dev_var_in[u_offset + IDX(i,7,k)]
//                             +        dev_var_in[u_offset + IDX(i,8,k)]
//                             ) * idy_by_12;
//             }
//             else {
//                 output[IDX(i,5,k)] = (           dev_var_in[u_offset + IDX(i,3,k)]
//                                 -  4.0 * dev_var_in[u_offset + IDX(i,4,k)]
//                                 +  3.0 * dev_var_in[u_offset + IDX(i,5,k)]
//                             ) * idy_by_2;
//             }
//         }

//         if ((bflag & (1u<<OCT_DIR_UP)) && (j == 4)) {
            
//             const int je = host_sz_y - 3;
            
//             if ( dev_var_in[betay + IDX(i,je-3,k)] < 0.0 ) {
//                 output[IDX(i,je-3,k)] = (  - 3.0 * dev_var_in[u_offset + IDX(i,je-3,k)]
//                                         + 4.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
//                                         -       dev_var_in[u_offset + IDX(i,je-1,k)]
//                                         ) * idy_by_2;
//             }
//             else {
//                 output[IDX(i,je-3,k)] = ( -   dev_var_in[u_offset + IDX(i,je-6,k)]
//                                     +  6.0 * dev_var_in[u_offset + IDX(i,je-5,k)]
//                                     - 18.0 * dev_var_in[u_offset + IDX(i,je-4,k)]
//                                     + 10.0 * dev_var_in[u_offset + IDX(i,je-3,k)]
//                                     +  3.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
//                                 ) * idy_by_12;
//             }
        
//                 if (dev_var_in[betay + IDX(i,je-2,k)] > 0.0 ) {
//                 output[IDX(i,je-2,k)] = (  -  dev_var_in[u_offset + IDX(i,je-3,k)]
//                                         +  dev_var_in[u_offset + IDX(i,je-1,k)]
//                                         ) * idy_by_2;
//                 }
//                 else {
//                 output[IDX(i,je-2,k)] = (     dev_var_in[u_offset + IDX(i,je-4,k)]
//                                     - 4.0 * dev_var_in[u_offset + IDX(i,je-3,k)]
//                                     + 3.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
//                                         ) * idy_by_2;
//                 }
        
//                 output[IDX(i,je-1,k)]  = (          dev_var_in[u_offset + IDX(i,je-3,k)]
//                                         - 4.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
//                                         + 3.0 * dev_var_in[u_offset + IDX(i,je-1,k)]
//                                     ) * idy_by_2;
//         }
//     }
// }

// void cuda_deriv42_adv_y(double * output, double * dev_var_in, 
//     int u_offset, double dy, int betay, int bflag, const unsigned int * host_sz, cudaStream_t stream)
// {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 3;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 3;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];
    
//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_adv_deriv);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_adv_deriv);
  
//     calc_deriv42_adv_y <<< number_of_blocks, threads_per_block_adv_deriv, 0, stream >>> (output, dev_var_in, betay, dy, bflag, host_sz_x, host_sz_y, host_sz_z, u_offset);
        
//     CHECK_ERROR(cudaGetLastError(), "calc_deriv42_adv_y Kernel launch failed");
// }

// __global__ void calc_deriv42_adv_z(double * output, double * dev_var_in, 
//     int betaz, double dz, int bflag, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int u_offset) 
// {
//     int thread_id = blockIdx.x*threads_per_block_adv_deriv + threadIdx.x;

//     for (int id = thread_id*thread_load_adv_deriv; id<(thread_id+1)*thread_load_adv_deriv; id++){
            
//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    
//         if (k>=host_sz_z-3) return;

//         int nx = host_sz_x;
//         int ny = host_sz_y;
        
//         const double idz = 1.0/dz;
//         const double idz_by_2 = 0.50 * idz;
//         const double idz_by_12 = idz / 12.0;

//         int n = nx * ny;
//         int pp = IDX(i, j, k);
        
//         if (dev_var_in[betaz + pp] > 0.0 ) {
//                 output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - n]
//                             - 10.0 * dev_var_in[u_offset + pp]
//                             + 18.0 * dev_var_in[u_offset + pp + n]
//                             -  6.0 * dev_var_in[u_offset + pp + 2*n]
//                             +        dev_var_in[u_offset + pp + 3*n]
//                         ) * idz_by_12;
//         }
//         else {
//             output[pp] = ( -        dev_var_in[u_offset + pp - 3*n]
//                         +  6.0 * dev_var_in[u_offset + pp - 2*n]
//                         - 18.0 * dev_var_in[u_offset + pp - n]
//                         + 10.0 * dev_var_in[u_offset + pp]
//                         +  3.0 * dev_var_in[u_offset + pp +n]
//                         ) * idz_by_12;
                    
//         }
        
//         if ((bflag & (1u<<OCT_DIR_BACK)) && (k == 3)) {
            
//             output[IDX(i,j,3)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,j,3)]
//                     +  4.0 * dev_var_in[u_offset + IDX(i,j,4)]
//                     -        dev_var_in[u_offset + IDX(i,j,5)]
//                     ) * idz_by_2;
                    
//             if (dev_var_in[betaz + IDX(i,j,4)] > 0.0) {
//                 output[IDX(i,j,4)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,j,4)]
//                                 +  4.0 * dev_var_in[u_offset + IDX(i,j,5)]
//                                 -        dev_var_in[u_offset + IDX(i,j,6)]
//                             ) * idz_by_2;

//             }
//             else {
//                 output[IDX(i,j,4)] = ( -         dev_var_in[u_offset + IDX(i,j,3)]
//                                 +        dev_var_in[u_offset + IDX(i,j,5)]
//                             ) * idz_by_2;
                            
//             }

//             if (dev_var_in[betaz + IDX(i,j,5)] > 0.0 ) {
//                 output[IDX(i,j,5)] = (-  3.0 * dev_var_in[u_offset + IDX(i,j,4)]
//                             - 10.0 * dev_var_in[u_offset + IDX(i,j,5)]
//                             + 18.0 * dev_var_in[u_offset + IDX(i,j,6)]
//                             -  6.0 * dev_var_in[u_offset + IDX(i,j,7)]
//                             +        dev_var_in[u_offset + IDX(i,j,8)]
//                             ) * idz_by_12;
//             }
//             else {
//                 output[IDX(i,j,5)] = (           dev_var_in[u_offset + IDX(i,j,3)]
//                                 -  4.0 * dev_var_in[u_offset + IDX(i,j,4)]
//                                 +  3.0 * dev_var_in[u_offset + IDX(i,j,5)]
//                             ) * idz_by_2;
//             }
//         }

//         if ((bflag & (1u<<OCT_DIR_FRONT)) && (k == 4)) {
            
//             const int ke = host_sz_z - 3; // Here I changed
            
//             if ( dev_var_in[betaz + IDX(i,j,ke-3)] < 0.0 ) {
//                 output[IDX(i,j,ke-3)] = (  - 3.0 * dev_var_in[u_offset + IDX(i,j,ke-3)]
//                                         + 4.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
//                                         -       dev_var_in[u_offset + IDX(i,j,ke-1)]
//                                         ) * idz_by_2;
//             }
//             else {
//                 output[IDX(i,j,ke-3)] = ( -   dev_var_in[u_offset + IDX(i,j,ke-6)]
//                                     +  6.0 * dev_var_in[u_offset + IDX(i,j,ke-5)]
//                                     - 18.0 * dev_var_in[u_offset + IDX(i,j,ke-4)]
//                                     + 10.0 * dev_var_in[u_offset + IDX(i,j,ke-3)]
//                                     +  3.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
//                                 ) * idz_by_12;
//             }
        
//                 if (dev_var_in[betaz + IDX(i,j,ke-2)] > 0.0 ) {
//                 output[IDX(i,j,ke-2)] = (  -  dev_var_in[u_offset + IDX(i,j,ke-3)]
//                                         +  dev_var_in[u_offset + IDX(i,j,ke-1)]
//                                         ) * idz_by_2;
//                 }
//                 else {
//                 output[IDX(i,j,ke-2)] = (     dev_var_in[u_offset + IDX(i,j,ke-4)]
//                                     - 4.0 * dev_var_in[u_offset + IDX(i,j,ke-3)]
//                                     + 3.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
//                                         ) * idz_by_2;
//                 }
        
//                 output[IDX(i,j,ke-1)]  = (          dev_var_in[u_offset + IDX(i,j,ke-3)]
//                                         - 4.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
//                                         + 3.0 * dev_var_in[u_offset + IDX(i,j,ke-1)]
//                                     ) * idz_by_2;
//         }
//     }
// }

// void cuda_deriv42_adv_z(double * output, double * dev_var_in, 
//     int u_offset, double dz, int betaz, int bflag, const unsigned int * host_sz, cudaStream_t stream)
// {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 3;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 3;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];
    
//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_adv_deriv);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_adv_deriv);
  
//     calc_deriv42_adv_z <<< number_of_blocks, threads_per_block_adv_deriv, 0, stream >>> (output, dev_var_in, betaz, dz, bflag, host_sz_x, host_sz_y, host_sz_z, u_offset);
    
//     CHECK_ERROR(cudaGetLastError(), "calc_deriv42_adv_z Kernel launch failed");
// }

// __global__ void calc_ko_deriv42_x(double * output, double * dev_var_in,
//     double dx, int bflag, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int u_offset)
// {
//     int thread_id = blockIdx.x*threads_per_block_ko_deriv + threadIdx.x;

//     for (int id = thread_id*thread_load_ko_deriv; id<(thread_id+1)*thread_load_ko_deriv; id++){

//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    
//         if (k>=host_sz_z-3) return;

//         int nx = host_sz_x;
//         int ny = host_sz_y;

//             if(i==4) {
//                 int ib=3;
//                 output[IDX(3, j, k)] = (-1.0 / 64.0 / dx) *
//                                 (
//                                 -      dev_var_in[u_offset + IDX(ib+4,j,k)]
//                                 +  6.0*dev_var_in[u_offset + IDX(ib+3,j,k)]
//                                 - 15.0*dev_var_in[u_offset + IDX(ib+2,j,k)]
//                                 + 20.0*dev_var_in[u_offset + IDX(ib+1,j,k)]
//                                 - 15.0*dev_var_in[u_offset + IDX(ib,j,k)]
//                                 +  6.0*dev_var_in[u_offset + IDX(ib-1,j,k)]
//                                 -      dev_var_in[u_offset + IDX(ib-2,j,k)]
//                                 );
//             }

//         int pp = IDX(i, j, k);
        
//         output[pp] = (-1.0 / 64.0 / dx) *
//                                 (
//                                 -      dev_var_in[u_offset + pp - 3]
//                                 +  6.0*dev_var_in[u_offset + pp - 2]
//                                 - 15.0*dev_var_in[u_offset + pp - 1]
//                                 + 20.0*dev_var_in[u_offset + pp ]
//                                 - 15.0*dev_var_in[u_offset + pp + 1]
//                                 +  6.0*dev_var_in[u_offset + pp + 2]
//                                 -      dev_var_in[u_offset + pp + 3]
//                                 );

//             if(i==5) {
//                 int ie = nx-3;
//                 output[IDX(ie-1, j, k)] = (-1.0 / 64.0 / dx) *
//                                 (
//                                 -      dev_var_in[u_offset + IDX(ie+1,j,k)]
//                                 +  6.0*dev_var_in[u_offset + IDX(ie,j,k)]
//                                 - 15.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
//                                 + 20.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
//                                 - 15.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
//                                 +  6.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
//                                 -      dev_var_in[u_offset + IDX(ie-5,j,k)]
//                                 );
//             }
        
//         if ((bflag & (1u<<OCT_DIR_LEFT)) && (i == 4)) {

//             output[IDX(3,j,k)] =  (      dev_var_in[u_offset + IDX(6,j,k)]
//                                         - 3.0*dev_var_in[u_offset + IDX(5,j,k)]
//                                         + 3.0*dev_var_in[u_offset + IDX(4,j,k)]
//                                         -     dev_var_in[u_offset + IDX(3,j,k)]
//                                     )/59.0/48.0*64*dx;
//             output[IDX(4,j,k)] =  (     dev_var_in[u_offset + IDX(7,j,k)]
//                                         -  6.0*dev_var_in[u_offset + IDX(6,j,k)]
//                                         + 12.0*dev_var_in[u_offset + IDX(5,j,k)]
//                                         - 10.0*dev_var_in[u_offset + IDX(4,j,k)]
//                                         +  3.0*dev_var_in[u_offset + IDX(3,j,k)]
//                                         )/43.0/48.0*64*dx;
//             output[IDX(5,j,k)] =  (     dev_var_in[u_offset + IDX(8,j,k)]
//                                         -  6.0*dev_var_in[u_offset + IDX(7,j,k)]
//                                         + 15.0*dev_var_in[u_offset + IDX(6,j,k)]
//                                         - 19.0*dev_var_in[u_offset + IDX(5,j,k)]
//                                         + 12.0*dev_var_in[u_offset + IDX(4,j,k)]
//                                         -  3.0*dev_var_in[u_offset + IDX(3,j,k)]
//                                         )/49.0/48.0*64*dx;
//             }

//         if ((bflag & (1u<<OCT_DIR_RIGHT)) && (i == 5)) {
            
//             const int ie = nx - 3;
//             output[IDX(ie-3,j,k)] = ( dev_var_in[u_offset + IDX(ie-6,j,k)]
//                                         - 6.0*dev_var_in[u_offset + IDX(ie-5,j,k)]
//                                         + 15.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
//                                         - 19.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
//                                         + 12.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
//                                         -  3.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
//                                         )/49.0/48.0*64*dx;
                
//                 output[IDX(ie-2,j,k)] =  ( dev_var_in[u_offset + IDX(ie-5,j,k)]
//                                         -  6.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
//                                         + 12.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
//                                         - 10.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
//                                         +  3.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
//                                         )/43.0/48.0*64*dx;
            
        
//                 output[IDX(ie-1,j,k)] = ( dev_var_in[u_offset + IDX(ie-4,j,k)]
//                                         -  3.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
//                                         +  3.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
//                                         -      dev_var_in[u_offset + IDX(ie-1,j,k)]
//                                         )/59.0/48.0*64*dx;
//         }
//     }
// }

// void cuda_ko_deriv42_x(double * output, double * dev_var_in, 
//    int u_offset, double dx, int bflag, const unsigned int * host_sz, cudaStream_t stream)
// {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 3;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 3;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];

//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_ko_deriv);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_ko_deriv);

//     calc_ko_deriv42_x <<< number_of_blocks, threads_per_block_ko_deriv, 0, stream >>> (output, dev_var_in, dx, bflag, host_sz_x, host_sz_y, host_sz_z, u_offset);

//     CHECK_ERROR(cudaGetLastError(), "calc_ko_deriv42_x Kernel launch failed");
// }

// __global__ void calc_ko_deriv42_y(double * output, double * dev_var_in,
//     double dy, int bflag, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int u_offset)
// {
//     int thread_id = blockIdx.x*threads_per_block_ko_deriv + threadIdx.x;

//     for (int id = thread_id*thread_load_ko_deriv; id<(thread_id+1)*thread_load_ko_deriv; id++){
        
//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    
//         if (k>=host_sz_z-3) return;

//         int nx = host_sz_x;
//         int ny = host_sz_y;

//         if(j==4) {
//             int jb=3;
//             output[IDX(i,jb,k)] = (-1.0 / 64.0 / dy) *
//                         (
//                             -      dev_var_in[u_offset + IDX(i,jb+4,k)]
//                             +  6.0*dev_var_in[u_offset + IDX(i,jb+3,k)]
//                             - 15.0*dev_var_in[u_offset + IDX(i,jb+2,k)]
//                             + 20.0*dev_var_in[u_offset + IDX(i,jb+1,k)]
//                             - 15.0*dev_var_in[u_offset + IDX(i,jb,k)]
//                             +  6.0*dev_var_in[u_offset + IDX(i,jb-1,k)]
//                             -      dev_var_in[u_offset + IDX(i,jb-2,k)]
//                             );
//             }

//         int pp = IDX(i, j, k);
        
//         output[pp] = (-1.0 / 64.0 / dy) *
//                         (
//                             -      dev_var_in[u_offset + pp-3*nx]
//                             +  6.0*dev_var_in[u_offset + pp-2*nx]
//                             - 15.0*dev_var_in[u_offset + pp-nx]
//                             + 20.0*dev_var_in[u_offset + pp]
//                             - 15.0*dev_var_in[u_offset + pp+nx]
//                             +  6.0*dev_var_in[u_offset + pp+2*nx]
//                             -      dev_var_in[u_offset + pp+3*nx]
//                             );

//             if(j==5) {
//                 int je = ny - 3;
//                 output[IDX(i,je-1,k)] = (-1.0 / 64.0 / dy) *
//                         (
//                             -      dev_var_in[u_offset + IDX(i,je+1,k)]
//                             +  6.0*dev_var_in[u_offset + IDX(i,je,k)]
//                             - 15.0*dev_var_in[u_offset + IDX(i,je-1,k)]
//                             + 20.0*dev_var_in[u_offset + IDX(i,je-2,k)]
//                             - 15.0*dev_var_in[u_offset + IDX(i,je-3,k)]
//                             +  6.0*dev_var_in[u_offset + IDX(i,je-4,k)]
//                             -      dev_var_in[u_offset + IDX(i,je-5,k)]
//                             );                   
//             }
//         if ((bflag & (1u<<OCT_DIR_DOWN)) && (j == 4)) {

//             output[IDX(i,3,k)] =  (      dev_var_in[u_offset +IDX(i,6,k)]
//                                         - 3.0*dev_var_in[u_offset +IDX(i,5,k)]
//                                         + 3.0*dev_var_in[u_offset + IDX(i,4,k)]
//                                         -     dev_var_in[u_offset + IDX(i,3,k)]
//                                     )/59.0/48.0*64*dy;
//             output[IDX(i,4,k)] =  (     dev_var_in[u_offset + IDX(i,7,k)]
//                                         -  6.0*dev_var_in[u_offset + IDX(i,6,k)]
//                                         + 12.0*dev_var_in[u_offset + IDX(i,5,k)]
//                                         - 10.0*dev_var_in[u_offset + IDX(i,4,k)]
//                                         +  3.0*dev_var_in[u_offset + IDX(i,3,k)]
//                                         )/43.0/48.0*64*dy;
//             output[IDX(i,5,k)] =  (     dev_var_in[u_offset + IDX(i,8,k)]
//                                         -  6.0*dev_var_in[u_offset + IDX(i,7,k)]
//                                         + 15.0*dev_var_in[u_offset + IDX(i,6,k)]
//                                         - 19.0*dev_var_in[u_offset + IDX(i,5,k)]
//                                         + 12.0*dev_var_in[u_offset + IDX(i,4,k)]
//                                         -  3.0*dev_var_in[u_offset + IDX(i,3,k)]
//                                         )/49.0/48.0*64*dy;
//             }

//         if ((bflag & (1u<<OCT_DIR_UP)) && (j == 5)) {
            
//             const int je = ny - 3;
//             output[IDX(i,je-3,k)] = (dev_var_in[u_offset + IDX(i,je-6,k)]
//                                         -  6.0*dev_var_in[u_offset + IDX(i,je-5,k)]
//                                         + 15.0*dev_var_in[u_offset + IDX(i,je-4,k)]
//                                         - 19.0*dev_var_in[u_offset + IDX(i,je-3,k)]
//                                         + 12.0*dev_var_in[u_offset + IDX(i,je-2,k)]
//                                         -  3.0*dev_var_in[u_offset + IDX(i,je-1,k)]
//                                         )/49.0/48.0*64*dy;
                
//                 output[IDX(i,je-2,k)] = (dev_var_in[u_offset + IDX(i,je-5,k)]
//                                         -  6.0*dev_var_in[u_offset + IDX(i,je-4,k)]
//                                         + 12.0*dev_var_in[u_offset + IDX(i,je-3,k)]
//                                         - 10.0*dev_var_in[u_offset + IDX(i,je-2,k)]
//                                         +  3.0*dev_var_in[u_offset + IDX(i,je-1,k)]
//                                         )/43.0/48.0*64*dy;
            
        
//                 output[IDX(i,je-1,k)] = ( dev_var_in[u_offset + IDX(i,je-4,k)]
//                                         -  3.0*dev_var_in[u_offset + IDX(i,je-3,k)]
//                                         +  3.0*dev_var_in[u_offset + IDX(i,je-2,k)]
//                                         -      dev_var_in[u_offset + IDX(i,je-1,k)]
//                                         )/59.0/48.0*64*dy;
//         }
//     }
// }

// void cuda_ko_deriv42_y(double * output, double * dev_var_in, 
//     int u_offset, double dy, int bflag, const unsigned int * host_sz, cudaStream_t stream)
// {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 3;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 3;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];

//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_ko_deriv);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_ko_deriv);

//     calc_ko_deriv42_y <<< number_of_blocks, threads_per_block_ko_deriv, 0, stream >>> (output, dev_var_in, dy, bflag, host_sz_x, host_sz_y, host_sz_z, u_offset);

//     CHECK_ERROR(cudaGetLastError(), "calc_ko_deriv42_y Kernel launch failed");

// }

// __global__ void calc_ko_deriv42_z(double * output, double * dev_var_in,
//     double dz, int bflag, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int u_offset)
// {
//     int thread_id = blockIdx.x*threads_per_block_ko_deriv + threadIdx.x;

//     for (int id = thread_id*thread_load_ko_deriv; id<(thread_id+1)*thread_load_ko_deriv; id++){
            
//         int i = id%(host_sz_x-6) + 3;
//         int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
//         int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    
//         if (k>=host_sz_z-3) return;

//         int nx = host_sz_x;
//         int ny = host_sz_y;

//         if(k==4) {
//             int kb=3;
//             output[IDX(i,j,kb)] = (-1.0 / 64.0 / dz) *
//                         (
//                             -      dev_var_in[u_offset + IDX(i,j,kb+4)]
//                             +  6.0*dev_var_in[u_offset + IDX(i,j,kb+3)]
//                             - 15.0*dev_var_in[u_offset + IDX(i,j,kb+2)]
//                             + 20.0*dev_var_in[u_offset + IDX(i,j,kb+1)]
//                             - 15.0*dev_var_in[u_offset + IDX(i,j,kb)]
//                             +  6.0*dev_var_in[u_offset + IDX(i,j,kb-1)]
//                             -      dev_var_in[u_offset + IDX(i,j,kb-2)]
//                             );
//             }

//             int pp = IDX(i, j, k);
//             int n = nx * ny;
//             output[pp] = (-1.0 / 64.0 / dz) *
//                         (
//                             -      dev_var_in[u_offset + pp-3*n]
//                             +  6.0*dev_var_in[u_offset + pp-2*n]
//                             - 15.0*dev_var_in[u_offset + pp-n]
//                             + 20.0*dev_var_in[u_offset + pp]
//                             - 15.0*dev_var_in[u_offset + pp+n]
//                             +  6.0*dev_var_in[u_offset + pp+2*n]
//                             -      dev_var_in[u_offset + pp+3*n]
//                             );

//             if(k==5) {
//                 int ke = host_sz_z - 3;
//                 output[IDX(i,j,ke-1)] = (-1.0 / 64.0 / dz) *
//                 (
//                     -      dev_var_in[u_offset + IDX(i,j,ke+1)]
//                     +  6.0*dev_var_in[u_offset + IDX(i,j,ke)]
//                     - 15.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
//                     + 20.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
//                     - 15.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
//                     +  6.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
//                     -      dev_var_in[u_offset + IDX(i,j,ke-5)]
//                     );               
//             }
        
        

        
//         if ((bflag & (1u<<OCT_DIR_BACK)) && (k == 4)) {

//             output[IDX(i,3,k)] =  (      dev_var_in[u_offset +IDX(i,k,6)]
//                                         - 3.0*dev_var_in[u_offset +IDX(i,k,5)]
//                                         + 3.0*dev_var_in[u_offset + IDX(i,k,4)]
//                                         -     dev_var_in[u_offset + IDX(i,k,3)]
//                                     )/59.0/48.0*64*dz;

//             output[IDX(i,j,4)] =  (     dev_var_in[u_offset + IDX(i,j,7)]
//                                         -  6.0*dev_var_in[u_offset + IDX(i,j,6)]
//                                         + 12.0*dev_var_in[u_offset + IDX(i,j,5)]
//                                         - 10.0*dev_var_in[u_offset + IDX(i,j,4)]
//                                         +  3.0*dev_var_in[u_offset + IDX(i,j,3)]
//                                         )/43.0/48.0*64*dz;

//             output[IDX(i,j,5)] =  (     dev_var_in[u_offset + IDX(i,j,8)]
//                                         -  6.0*dev_var_in[u_offset + IDX(i,j,7)]
//                                         + 15.0*dev_var_in[u_offset + IDX(i,j,6)]
//                                         - 19.0*dev_var_in[u_offset + IDX(i,j,5)]
//                                         + 12.0*dev_var_in[u_offset + IDX(i,j,4)]
//                                         -  3.0*dev_var_in[u_offset + IDX(i,j,3)]
//                                         )/49.0/48.0*64*dz;
//             }

//         if ((bflag & (1u<<OCT_DIR_FRONT)) && (k == 5)) {
            
//             const int ke = host_sz_z - 3;
//             output[IDX(i,j,ke-3)] = (    dev_var_in[u_offset + IDX(i,j,ke-6)]
//                                             -  6.0*dev_var_in[u_offset + IDX(i,j,ke-5)]
//                                             + 15.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
//                                             - 19.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
//                                             + 12.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
//                                             -  3.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
//                                             )/49.0/48.0*64*dz;
                
//                 output[IDX(i,j,ke-2)] = (   dev_var_in[u_offset + IDX(i,j,ke-5)]
//                                             -  6.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
//                                             + 12.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
//                                             - 10.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
//                                             +  3.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
//                                             )/43.0/48.0*64*dz;
            
        
//                 output[IDX(i,j,ke-1)] = (   dev_var_in[u_offset + IDX(i,j,ke-4)]
//                                             -  3.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
//                                             +  3.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
//                                             -      dev_var_in[u_offset + IDX(i,j,ke-1)]
//                                             )/59.0/48.0*64*dz;
//         }
//     }
// }

// void cuda_ko_deriv42_z(double * output, double * dev_var_in, 
//     int u_offset, double dz, int bflag, const unsigned int * host_sz, cudaStream_t stream)
// {
//     const int ib = 3;
//     const int jb = 3;
//     const int kb = 3;
//     const int ie = host_sz[0] - 3;
//     const int je = host_sz[1] - 3;
//     const int ke = host_sz[2] - 3;
//     const unsigned int host_sz_x = host_sz[0];
//     const unsigned int host_sz_y = host_sz[1];
//     const unsigned int host_sz_z = host_sz[2];

//     const int number_of_threads_required = ceil((ie-ib)*(je-jb)*(ke-kb)/thread_load_ko_deriv);
//     int number_of_blocks = ceil(1.0*number_of_threads_required/threads_per_block_ko_deriv);

//     calc_ko_deriv42_z <<< number_of_blocks, threads_per_block_ko_deriv, 0, stream >>> (output, dev_var_in, dz, bflag, host_sz_x, host_sz_y, host_sz_z, u_offset);


//     CHECK_ERROR(cudaGetLastError(), "calc_ko_deriv42_z Kernel launch failed");
// }