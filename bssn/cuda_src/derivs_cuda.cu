#include "hip/hip_runtime.h"
#include "derivs_cuda.h"

 
__device__ void device_calc_deriv_x(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
    
    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-2)) + 1;
    int k = (id/(sz_y-2)/(sz_x-6)) + 1;

    if(i >= nx-3 || j >= ny-1 || k >= nz-1) return;

    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[offset + pp - 2] - 8.0*dev_var_in[offset
                                            + pp - 1] + 8.0*dev_var_in[offset + pp + 1]
                                            - dev_var_in[offset + pp + 2] )*((1.0/hx)/12.0);

    if ((bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
        int pp3 = IDX(3, j, k);
        int pp4 = IDX(4, j, k);
        int pp5 = IDX(5, j, k);
        output[pp3] = ((-3)*dev_var_in[offset + pp3] + 4*dev_var_in[offset
                                                    + pp4] - dev_var_in[offset + pp5]) * 0.5 / hx;
        output[pp4] = (dev_var_in[offset + pp5] - dev_var_in[offset
                                            + pp3]) * (0.50/hx);
    }

    if ((bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
        int pp2 = IDX(nx-5, j, k); // IDX(ie-2,j,k)
        int pp3 = IDX(nx-6, j, k); // IDX(ie-3,j,k)
        int pp1 = IDX(nx-4,j,k); // IDX(ie-1,j,k)
        output[pp2] = (dev_var_in[offset + pp1] - dev_var_in[offset + pp3])
        * 0.50 / hx;
        output[pp1] = (dev_var_in[offset + pp3]- 4.0 * dev_var_in[offset + pp2]
        + 3.0 * dev_var_in[offset + pp1]) * 0.50 / hx;

    }

} 
 
 __device__ void device_calc_deriv_y(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
        
    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
    int k = (id/(sz_y-6)/(sz_x-6)) + 1;

    if(i >= nx-3 || j >= ny-3 || k >= nz-1) return;

    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[offset + pp - 2*nx]
    - 8.0*dev_var_in[offset + pp - nx]
    + 8.0*dev_var_in[offset + pp + nx]
    - dev_var_in[offset + pp + 2*nx] )*((1.0/hx)/12.0);


    if ((bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
        int pp3 = IDX(i, 3, k);
        int pp4 = IDX(i, 4, k);
        int pp5 = IDX(i, 5, k);

        output[pp3] = ((-3)*dev_var_in[(offset) + pp3] +  4*dev_var_in[(offset) + pp4]
        - dev_var_in[(offset) + pp5]) * 0.5 / hx;
        output[pp4] = (dev_var_in[(offset) + pp5] - dev_var_in[(offset) + pp3])
        * (0.50/hx);

    }

    if ((bflag & (1u<<OCT_DIR_UP)) && j==4)  {
        int pp2 = IDX(i, ny-5, k); // IDX(i,je-2,k)
        int pp3 = IDX(i, ny-6, k); // IDX(i,je-3,k)
        int pp1 = IDX(i, ny-4, k); // IDX(i,je-1,k)

        output[pp2] = (dev_var_in[(offset) + pp1] - dev_var_in[(offset) + pp3])
        * 0.50 / hx;
        output[pp1] = (dev_var_in[(offset) + pp3]- 4.0 * dev_var_in[(offset) + pp2]
        + 3.0 * dev_var_in[(offset) + pp1]) * 0.50 / hx;

    }

}

 
 __device__ void device_calc_deriv_z(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
        
    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
    int k = (id/(sz_y-6)/(sz_x-6)) + 3;
    
    if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

    int pp = IDX(i, j, k);

    int n = nx * ny;

    output[pp] = (dev_var_in[offset + pp - 2*n] - 8.0*dev_var_in[offset + pp - n]
    + 8.0*dev_var_in[offset + pp + n] - dev_var_in[offset + pp + 2*n])
    * ((1.0/hx)/12);

    if ((bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
        int pp3 = IDX(i, j, 3); // IDX(i, j, 3)
        int pp4 = IDX(i, j, 4); // IDX(i,j,4)
        int pp5 = IDX(i, j, 5); // IDX(i,j,5)

        output[pp3] = ((-3)*dev_var_in[offset + pp3] + 4*dev_var_in[offset + pp4]
        - dev_var_in[offset + pp5]) * 0.5 / hx;
        output[pp4] = (dev_var_in[offset + pp5] - dev_var_in[offset + pp3])
        * (0.50/hx);
    }

    if ((bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
        int pp2 = IDX(i, j, nz-5); // IDX(i,j,ke-2)
        int pp3 = IDX(i, j, nz-6); // IDX(i,j,ke-3)
        int pp1 = IDX(i, j, nz-4); // IDX(i,j,ke-1)

        output[pp2] = (dev_var_in[offset + pp1] - dev_var_in[offset + pp3])
        * 0.50 / hx;
        output[pp1] = (dev_var_in[offset + pp3]- 4.0 * dev_var_in[offset + pp2]
        + 3.0 * dev_var_in[offset + pp1]) * 0.50 / hx;
    }


}

__device__ void device_calc_deriv_xx(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
        
    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
    int k = (id/(sz_y-6)/(sz_x-6)) + 3;

    if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

    int pp = IDX(i, j, k);

    output[pp] = ((-1)*dev_var_in[offset + pp - 2]
    + 16.0*dev_var_in[offset + pp - 1]
    - 30.0*dev_var_in[offset + pp]
    + 16.0*dev_var_in[offset + pp + 1]
    - dev_var_in[offset + pp + 2]
    )*(1.0/(hx*hx))/12.0;

    if ((bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
        int pp3 = IDX(3, j, k);
        int pp4 = IDX(4, j, k);
        int pp5 = IDX(5, j, k);
        int pp6 = IDX(6, j, k);

        output[pp3] = (
        2.0 *   dev_var_in[offset + pp3]
        -   5.0 *   dev_var_in[offset + pp4]
        +   4.0 *   dev_var_in[offset + pp5]
        -           dev_var_in[offset + pp6]
        ) * 1.0/(hx*hx);

        output[pp4] = (
        dev_var_in[offset + pp3]
        -   2.0 *   dev_var_in[offset + pp4]
        +           dev_var_in[offset + pp5]
        ) * 1.0/(hx*hx);

    }

    if ((bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
        int pp1 = IDX(nx - 4, j, k); // IDX(ie-1,j,k)
        int pp2 = IDX(nx - 5, j, k); // IDX(ie-2,j,k)
        int pp3 = IDX(nx - 6, j, k); // IDX(ie-3,j,k)
        int pp4 = IDX(nx - 7, j, k); // IDX(ie-4,j,k)

        output[pp2] = (
        dev_var_in[offset + pp3]
        -   2.0 *   dev_var_in[offset + pp2]
        +           dev_var_in[offset + pp1]
        ) * 1.0/(hx*hx);


        output[pp1] = (
        -   1.0 *   dev_var_in[offset + pp4]
        +   4.0 *   dev_var_in[offset + pp3]
        -   5.0 *   dev_var_in[offset + pp2]
        +   2.0 *   dev_var_in[offset + pp1]
        ) * 1.0/(hx*hx);
    }
}
 
__device__ void device_calc_deriv_yy(double * output, double * dev_var_in,
    const int offset, double hy, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
        
    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
    int k = (id/(sz_y-6)/(sz_x-6)) + 3;
    
    if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

    int pp = IDX(i, j, k);

    output[pp] = ((-1)*dev_var_in[offset + pp - 2*nx]
    + 16.0*dev_var_in[offset + pp - nx]
    - 30.0*dev_var_in[offset + pp]
    + 16.0*dev_var_in[offset + pp + nx]
    - dev_var_in[offset + pp + 2*nx]
    )*(1.0/(hy*hy))/12.0;

    if ((bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
        int pp3 = IDX(i, 3, k);
        int pp4 = IDX(i, 4, k);
        int pp5 = IDX(i, 5, k);
        int pp6 = IDX(i, 6, k);

        output[pp3] = (
        2.0 *   dev_var_in[offset + pp3]
        -   5.0 *   dev_var_in[offset + pp4]
        +   4.0 *   dev_var_in[offset + pp5]
        -           dev_var_in[offset + pp6]
        ) * 1.0/(hy*hy);

        output[pp4] = (
        dev_var_in[offset + pp3]
        -   2.0 *   dev_var_in[offset + pp4]
        +           dev_var_in[offset + pp5]
        ) * 1.0/(hy*hy);
    }

    if ((bflag & (1u<<OCT_DIR_UP)) && j==4)  {
        int pp1 = IDX(i, ny - 4, k);
        int pp2 = IDX(i, ny - 5, k);
        int pp3 = IDX(i, ny - 6, k);
        int pp4 = IDX(i, ny - 7, k);

        output[pp2] = (
        dev_var_in[offset + pp3]
        -   2.0 *   dev_var_in[offset + pp2]
        +           dev_var_in[offset + pp1]
        ) * 1.0/(hy*hy);


        output[pp1] = (
        -   1.0 *   dev_var_in[offset + pp4]
        +   4.0 *   dev_var_in[offset + pp3]
        -   5.0 *   dev_var_in[offset + pp2]
        +   2.0 *   dev_var_in[offset + pp1]
        ) * 1.0/(hy*hy);

    }
}

__device__ void device_calc_deriv_zz(double * output, double * dev_var_in,
    const int offset, double hz, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
        
    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
    int k = (id/(sz_y-6)/(sz_x-6)) + 3;
    
        if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

        int pp = IDX(i, j, k);

        int n = nx * ny;

        output[pp] = ((-1)*dev_var_in[offset + pp - 2*n]
        + 16.0*dev_var_in[offset + pp - n]
        - 30.0*dev_var_in[offset + pp]
        + 16.0*dev_var_in[offset + pp + n]
        - dev_var_in[offset + pp + 2*n]
        )*(1.0/(hz*hz))/12.0;

        if ((bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
            int pp3 = IDX(i, j, 3);
            int pp4 = IDX(i, j, 4);
            int pp5 = IDX(i, j, 5);
            int pp6 = IDX(i, j, 6);

            output[pp3] = (
            2.0 *   dev_var_in[offset + pp3]
            -   5.0 *   dev_var_in[offset + pp4]
            +   4.0 *   dev_var_in[offset + pp5]
            -           dev_var_in[offset + pp6]
            ) * 1.0/(hz*hz);

            output[pp4] = (
            dev_var_in[offset + pp3]
            -   2.0 *   dev_var_in[offset + pp4]
            +           dev_var_in[offset + pp5]
            ) * 1.0/(hz*hz);
        }

        if ((bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
            int pp1 = IDX(i, j, nz - 4);
            int pp2 = IDX(i, j, nz - 5);
            int pp3 = IDX(i, j, nz - 6);
            int pp4 = IDX(i, j, nz - 7);

            output[pp2] = (
            dev_var_in[offset + pp3]
            -   2.0 *   dev_var_in[offset + pp2]
            +           dev_var_in[offset + pp1]
            ) * 1.0/(hz*hz);


            output[pp1] = (
            -   1.0 *   dev_var_in[offset + pp4]
            +   4.0 *   dev_var_in[offset + pp3]
            -   5.0 *   dev_var_in[offset + pp2]
            +   2.0 *   dev_var_in[offset + pp1]
            ) * 1.0/(hz*hz);
        }
}
__global__ void calc_deriv42_first_part(double * dev_var_in, double hx, double hy, double hz, 
    int sz_x, int sz_y, int sz_z, int bflag,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
 ){
 
     int nx = sz_x;
     int ny = sz_y;
     int nz = sz_z;

 
 #include "bssnrhs_cuda_derivs_first_part.h"
 
 }
 
__global__ void calc_deriv42_second_part(double * dev_var_in, double hx, double hy, 
    double hz, int sz_x, int sz_y, int sz_z, int bflag,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
){
 
     int nx = sz_x;
     int ny = sz_y;
     int nz = sz_z;
 
 #include "bssnrhs_cuda_derivs_secondd_part.h"
 
 }
 
void cuda_calc_all(double * dev_var_in, double hx, double hy, double hz, int sz_x, 
    int sz_y, int sz_z, int bflag, hipStream_t stream,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
    ){
 
    const int ie = sz_x - 1;//x direction
    const int je = sz_y - 1;//y direction
    const int ke = sz_z - 1;//z direction
 
    int total_points = ie*je*ke;
    int blocks = ceil(1.0*total_points/threads_per_block);

    calc_deriv42_first_part <<< blocks, threads_per_block, 0, stream >>> (
                      dev_var_in, hx, hy, hz, sz_x, sz_y, sz_z, bflag,
                    #include "list_of_args.h"
                    ,
                    #include "list_of_offset_args.h"
             );

    calc_deriv42_second_part <<< blocks, threads_per_block, 0, stream >>> (
                      dev_var_in, hx, hy, hz, sz_x, sz_y, sz_z, bflag,
                    #include "list_of_args.h"
                    ,
                    #include "list_of_offset_args.h"
             );
}



__device__ void device_calc_adv_x(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int betax, int sz_x, int sz_y, int sz_z){

    int id = blockIdx.x*threads_per_block + threadIdx.x;
    
    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
    int k = (id/(sz_y-6)/(sz_x-6)) + 3;

    double idx_by_2 = 0.50 * (1.0 / hx);
    double idx_by_12 = (1.0 / hx)/12;

    if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

    int pp = IDX(i, j, k);

    if (dev_var_in[betax + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[offset + pp - 1]
        - 10.0 * dev_var_in[offset + pp]
        + 18.0 * dev_var_in[offset + pp + 1]
        -  6.0 * dev_var_in[offset + pp + 2]
        +        dev_var_in[offset + pp + 3]
        ) * idx_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[offset + pp - 3]
        +  6.0 * dev_var_in[offset + pp - 2]
        - 18.0 * dev_var_in[offset + pp - 1]
        + 10.0 * dev_var_in[offset + pp]
        +  3.0 * dev_var_in[offset + pp +1]
        ) * idx_by_12;
    }

    if ((bflag & (1u<<OCT_DIR_LEFT)) && (i == 3)) {

        output[IDX(3,j,k)] = ( -  3.0 * dev_var_in[offset + IDX(3,j,k)]
        +  4.0 * dev_var_in[offset + IDX(4,j,k)]
        -        dev_var_in[offset + IDX(5,j,k)]
        ) * idx_by_2;

        if (dev_var_in[betax + IDX(4,j,k)] > 0.0) {
        output[IDX(4,j,k)] = ( -  3.0 * dev_var_in[offset + IDX(4,j,k)]
            +  4.0 * dev_var_in[offset + IDX(5,j,k)]
            -        dev_var_in[offset + IDX(6,j,k)]
        ) * idx_by_2;
        }
        else {
        output[IDX(4,j,k)] = ( -         dev_var_in[offset + IDX(3,j,k)]
            +        dev_var_in[offset + IDX(5,j,k)]
        ) * idx_by_2;
        }

        if (dev_var_in[betax + IDX(5,j,k)] > 0.0 ) {
        output[IDX(5,j,k)] = (-  3.0 * dev_var_in[offset + IDX(4,j,k)]
            - 10.0 * dev_var_in[offset + IDX(5,j,k)]
            + 18.0 * dev_var_in[offset + IDX(6,j,k)]
            -  6.0 * dev_var_in[offset + IDX(7,j,k)]
            +        dev_var_in[offset + IDX(8,j,k)]
        ) * idx_by_12;
        }
        else {
        output[IDX(5,j,k)] = (           dev_var_in[offset + IDX(3,j,k)]
                    -  4.0 * dev_var_in[offset + IDX(4,j,k)]
                    +  3.0 * dev_var_in[offset + IDX(5,j,k)]
        ) * idx_by_2;
        }
    }

    if ((bflag & (1u<<OCT_DIR_RIGHT)) && (i == 4)) {

        const int ie = nx - 3;

        if ( dev_var_in[betax + IDX(ie-3,j,k)] < 0.0 ) {
            output[IDX(ie-3,j,k)] = (  - 3.0 * dev_var_in[offset + IDX(ie-3,j,k)]
                    + 4.0 * dev_var_in[offset + IDX(ie-2,j,k)]
                    -       dev_var_in[offset + IDX(ie-1,j,k)]
                ) * idx_by_2;
        }
        else {
            output[IDX(ie-3,j,k)] = ( -   dev_var_in[offset + IDX(ie-6,j,k)]
                    +  6.0 * dev_var_in[offset + IDX(ie-5,j,k)]
                    - 18.0 * dev_var_in[offset + IDX(ie-4,j,k)]
                    + 10.0 * dev_var_in[offset + IDX(ie-3  ,j,k)]
                    +  3.0 * dev_var_in[offset + IDX(ie-2,j,k)]
                ) * idx_by_12;
        }

        if (dev_var_in[betax + IDX(ie-2,j,k)] > 0.0 ) {
            output[IDX(ie-2,j,k)] = (  -  dev_var_in[offset + IDX(ie-3,j,k)]
                    +  dev_var_in[offset + IDX(ie-1,j,k)]
                ) * idx_by_2;
        }
        else {
            output[IDX(ie-2,j,k)] = (     dev_var_in[offset + IDX(ie-4,j,k)]
                        - 4.0 * dev_var_in[offset + IDX(ie-3,j,k)]
                        + 3.0 * dev_var_in[offset + IDX(ie-2,j,k)]
                ) * idx_by_2;
        }

        output[IDX(ie-1,j,k)] = (          dev_var_in[offset + IDX(ie-3,j,k)]
                    - 4.0 * dev_var_in[offset + IDX(ie-2,j,k)]
                    + 3.0 * dev_var_in[offset + IDX(ie-1,j,k)]
        ) * idx_by_2;
    }
}
 
__device__ void device_calc_adv_y(double * output, double * dev_var_in,
    const int offset, double hy, int bflag,
    int nx,int ny,int nz, int betay, int sz_x, int sz_y, int sz_z){

    int id = blockIdx.x*threads_per_block + threadIdx.x;
    
    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
    int k = (id/(sz_y-6)/(sz_x-6)) + 3;

    double idy_by_2 = 0.50 * (1.0 / hy);
    double idy_by_12 = (1.0 / hy)/12.0;

    if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

    int pp = IDX(i, j, k);

    if (dev_var_in[betay + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[offset + pp - nx]
        - 10.0 * dev_var_in[offset + pp]
        + 18.0 * dev_var_in[offset + pp + nx]
        -  6.0 * dev_var_in[offset + pp + 2*nx]
        +        dev_var_in[offset + pp + 3*nx]
        ) * idy_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[offset + pp - 3*nx]
        +  6.0 * dev_var_in[offset + pp - 2*nx]
        - 18.0 * dev_var_in[offset + pp - nx]
        + 10.0 * dev_var_in[offset + pp]
        +  3.0 * dev_var_in[offset + pp +nx]
        ) * idy_by_12;

    }

    if ((bflag & (1u<<OCT_DIR_DOWN)) && (j == 3)) {

        output[IDX(i,3,k)] = ( -  3.0 * dev_var_in[offset + IDX(i,3,k)]
        +  4.0 * dev_var_in[offset + IDX(i,4,k)]
        -        dev_var_in[offset + IDX(i,5,k)]
        ) * idy_by_2;

        if (dev_var_in[betay + IDX(i,4,k)] > 0.0) {
            output[IDX(i,4,k)] = ( -  3.0 * dev_var_in[offset + IDX(i,4,k)]
                +  4.0 * dev_var_in[offset + IDX(i,5,k)]
                -        dev_var_in[offset + IDX(i,6,k)]
            ) * idy_by_2;

        }
        else {
            output[IDX(i,4,k)] = ( -         dev_var_in[offset + IDX(i,3,k)]
                +        dev_var_in[offset + IDX(i,5,k)]
            ) * idy_by_2;

        }

        if (dev_var_in[betay + IDX(i,5,k)] > 0.0 ) {
            output[IDX(i,5,k)] = (-  3.0 * dev_var_in[offset + IDX(i,4,k)]
                - 10.0 * dev_var_in[offset + IDX(i,5,k)]
                + 18.0 * dev_var_in[offset + IDX(i,6,k)]
                -  6.0 * dev_var_in[offset + IDX(i,7,k)]
                +        dev_var_in[offset + IDX(i,8,k)]
            ) * idy_by_12;
        }
        else {
            output[IDX(i,5,k)] = (           dev_var_in[offset + IDX(i,3,k)]
                        -  4.0 * dev_var_in[offset + IDX(i,4,k)]
                        +  3.0 * dev_var_in[offset + IDX(i,5,k)]
            ) * idy_by_2;
        }
    }

    if ((bflag & (1u<<OCT_DIR_UP)) && (j == 4)) {

        const int je = ny - 3;

        if ( dev_var_in[betay + IDX(i,je-3,k)] < 0.0 ) {
        output[IDX(i,je-3,k)] = (  - 3.0 * dev_var_in[offset + IDX(i,je-3,k)]
                + 4.0 * dev_var_in[offset + IDX(i,je-2,k)]
                -       dev_var_in[offset + IDX(i,je-1,k)]
            ) * idy_by_2;
        }
        else {
        output[IDX(i,je-3,k)] = ( -   dev_var_in[offset + IDX(i,je-6,k)]
                +  6.0 * dev_var_in[offset + IDX(i,je-5,k)]
                - 18.0 * dev_var_in[offset + IDX(i,je-4,k)]
                + 10.0 * dev_var_in[offset + IDX(i,je-3,k)]
                +  3.0 * dev_var_in[offset + IDX(i,je-2,k)]
            ) * idy_by_12;
        }

        if (dev_var_in[betay + IDX(i,je-2,k)] > 0.0 ) {
        output[IDX(i,je-2,k)] = (  -  dev_var_in[offset + IDX(i,je-3,k)]
                +  dev_var_in[offset + IDX(i,je-1,k)]
            ) * idy_by_2;
        }
        else {
        output[IDX(i,je-2,k)] = (     dev_var_in[offset + IDX(i,je-4,k)]
                    - 4.0 * dev_var_in[offset + IDX(i,je-3,k)]
                    + 3.0 * dev_var_in[offset + IDX(i,je-2,k)]
            ) * idy_by_2;
        }

        output[IDX(i,je-1,k)]  = (          dev_var_in[offset + IDX(i,je-3,k)]
                    - 4.0 * dev_var_in[offset + IDX(i,je-2,k)]
                    + 3.0 * dev_var_in[offset + IDX(i,je-1,k)]
        ) * idy_by_2;
    }
}

 
__device__ void device_calc_adv_z(double * output, double * dev_var_in,
    const int offset, double hz, int bflag,
    int nx,int ny,int nz, int betaz, int sz_x, int sz_y, int sz_z){

    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
    int k = (id/(sz_y-6)/(sz_x-6)) + 3;
    

    double idz_by_2 = 0.50 * (1.0 / hz);
    double idz_by_12 = (1.0 / hz)/12.0;

    if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

    int n = nx * ny;
    int pp = IDX(i, j, k);

    if (dev_var_in[betaz + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[offset + pp - n]
        - 10.0 * dev_var_in[offset + pp]
        + 18.0 * dev_var_in[offset + pp + n]
        -  6.0 * dev_var_in[offset + pp + 2*n]
        +        dev_var_in[offset + pp + 3*n]
        ) * idz_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[offset + pp - 3*n]
        +  6.0 * dev_var_in[offset + pp - 2*n]
        - 18.0 * dev_var_in[offset + pp - n]
        + 10.0 * dev_var_in[offset + pp]
        +  3.0 * dev_var_in[offset + pp +n]
        ) * idz_by_12;

    }

    if ((bflag & (1u<<OCT_DIR_BACK)) && (k == 3)) {

        output[IDX(i,j,3)] = ( -  3.0 * dev_var_in[offset + IDX(i,j,3)]
        +  4.0 * dev_var_in[offset + IDX(i,j,4)]
        -        dev_var_in[offset + IDX(i,j,5)]
        ) * idz_by_2;

        if (dev_var_in[betaz + IDX(i,j,4)] > 0.0) {
        output[IDX(i,j,4)] = ( -  3.0 * dev_var_in[offset + IDX(i,j,4)]
            +  4.0 * dev_var_in[offset + IDX(i,j,5)]
            -        dev_var_in[offset + IDX(i,j,6)]
        ) * idz_by_2;

        }
        else {
        output[IDX(i,j,4)] = ( -         dev_var_in[offset + IDX(i,j,3)]
            +        dev_var_in[offset + IDX(i,j,5)]
        ) * idz_by_2;

        }

        if (dev_var_in[betaz + IDX(i,j,5)] > 0.0 ) {
        output[IDX(i,j,5)] = (-  3.0 * dev_var_in[offset + IDX(i,j,4)]
            - 10.0 * dev_var_in[offset + IDX(i,j,5)]
            + 18.0 * dev_var_in[offset + IDX(i,j,6)]
            -  6.0 * dev_var_in[offset + IDX(i,j,7)]
            +        dev_var_in[offset + IDX(i,j,8)]
        ) * idz_by_12;
        }
        else {
        output[IDX(i,j,5)] = (           dev_var_in[offset + IDX(i,j,3)]
                    -  4.0 * dev_var_in[offset + IDX(i,j,4)]
                    +  3.0 * dev_var_in[offset + IDX(i,j,5)]
        ) * idz_by_2;
        }
    }

    if ((bflag & (1u<<OCT_DIR_FRONT)) && (k == 4)) {

        const int ke = nz - 3;

        if ( dev_var_in[betaz + IDX(i,j,ke-3)] < 0.0 ) {
        output[IDX(i,j,ke-3)] = (  - 3.0 * dev_var_in[offset + IDX(i,j,ke-3)]
                + 4.0 * dev_var_in[offset + IDX(i,j,ke-2)]
                -       dev_var_in[offset + IDX(i,j,ke-1)]
            ) * idz_by_2;
        }
        else {
        output[IDX(i,j,ke-3)] = ( -   dev_var_in[offset + IDX(i,j,ke-6)]
                +  6.0 * dev_var_in[offset + IDX(i,j,ke-5)]
                - 18.0 * dev_var_in[offset + IDX(i,j,ke-4)]
                + 10.0 * dev_var_in[offset + IDX(i,j,ke-3)]
                +  3.0 * dev_var_in[offset + IDX(i,j,ke-2)]
            ) * idz_by_12;
        }

        if (dev_var_in[betaz + IDX(i,j,ke-2)] > 0.0 ) {
        output[IDX(i,j,ke-2)] = (  -  dev_var_in[offset + IDX(i,j,ke-3)]
                +  dev_var_in[offset + IDX(i,j,ke-1)]
            ) * idz_by_2;
        }
        else {
        output[IDX(i,j,ke-2)] = (     dev_var_in[offset + IDX(i,j,ke-4)]
                    - 4.0 * dev_var_in[offset + IDX(i,j,ke-3)]
                    + 3.0 * dev_var_in[offset + IDX(i,j,ke-2)]
            ) * idz_by_2;
        }

        output[IDX(i,j,ke-1)]  = (          dev_var_in[offset + IDX(i,j,ke-3)]
                    - 4.0 * dev_var_in[offset + IDX(i,j,ke-2)]
                    + 3.0 * dev_var_in[offset + IDX(i,j,ke-1)]
        ) * idz_by_2;
    }
}
 

 
 __global__ void calc_all_adv(double * dev_var_in, double hx, double hy, double hz, 
    int sz_x, int sz_y, int sz_z, int bflag,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
 ) {
    int nx = sz_x;
    int ny = sz_y;
    int nz = sz_z;
 
     #include "bssnrhs_cuda_derivs_adv.h"
     //ib, jb, kb values are accumulated to the x, y, z
 
 }

 void cuda_deriv_calc_all_adv(double * dev_var_in, double hx, double hy, double hz, int sz_x, 
    int sz_y, int sz_z, int bflag, hipStream_t stream,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
    ){
     const int ie = sz_x - 3;//x direction
     const int je = sz_y - 3;//y direction
     const int ke = sz_z - 3;//z direction
     
     int total_points = ie*je*ke;
     int blocks = ceil(1.0*total_points/threads_per_block);

     calc_all_adv <<< blocks, threads_per_block, 0, stream >>> (
                    dev_var_in, hx, hy, hz, sz_x, sz_y, sz_z, bflag,
                  #include "list_of_args.h"
                  ,
                  #include "list_of_offset_args.h"
           );
}

__device__ void device_calc_ko_deriv_x(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){

    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(sz_x-8) + 4;
    int j = ((id/(sz_x-8))%(sz_y-6)) + 3;
    int k = (id/(sz_y-6)/(sz_x-8)) + 3;
    
    if(i >= nx-4 || j >= ny-3 || k >= nz-3) return;

    if(i==4) {
        int ib=3;
        output[IDX(3, j, k)] = (-1.0 / 64.0 / hx) *
        (
                -      dev_var_in[offset + IDX(ib+4,j,k)]
                +  6.0*dev_var_in[offset + IDX(ib+3,j,k)]
                - 15.0*dev_var_in[offset + IDX(ib+2,j,k)]
                + 20.0*dev_var_in[offset + IDX(ib+1,j,k)]
                - 15.0*dev_var_in[offset + IDX(ib,j,k)]
                +  6.0*dev_var_in[offset + IDX(ib-1,j,k)]
                -      dev_var_in[offset + IDX(ib-2,j,k)]
        );
    }

    int pp = IDX(i, j, k);

    output[pp] = (-1.0 / 64.0 / hx) *
    (
    -      dev_var_in[offset + pp - 3]
    +  6.0*dev_var_in[offset + pp - 2]
    - 15.0*dev_var_in[offset + pp - 1]
    + 20.0*dev_var_in[offset + pp ]
    - 15.0*dev_var_in[offset + pp + 1]
    +  6.0*dev_var_in[offset + pp + 2]
    -      dev_var_in[offset + pp + 3]
    );

    if(i==5) {
        int ie = nx-3;
        output[IDX(ie-1, j, k)] = (-1.0 / 64.0 / hx) *
            (
                    -      dev_var_in[offset + IDX(ie+1,j,k)]
                    +  6.0*dev_var_in[offset + IDX(ie,j,k)]
                    - 15.0*dev_var_in[offset + IDX(ie-1,j,k)]
                    + 20.0*dev_var_in[offset + IDX(ie-2,j,k)]
                    - 15.0*dev_var_in[offset + IDX(ie-3,j,k)]
                    +  6.0*dev_var_in[offset + IDX(ie-4,j,k)]
                    -      dev_var_in[offset + IDX(ie-5,j,k)]
            );
    }

        if ((bflag & (1u<<OCT_DIR_LEFT)) && (i == 4)) {

        output[IDX(3,j,k)] =  (      dev_var_in[offset + IDX(6,j,k)]
            - 3.0*dev_var_in[offset + IDX(5,j,k)]
            + 3.0*dev_var_in[offset + IDX(4,j,k)]
            -     dev_var_in[offset + IDX(3,j,k)]
        )/59.0/48.0*64*hx;
        output[IDX(4,j,k)] =  (     dev_var_in[offset + IDX(7,j,k)]
            -  6.0*dev_var_in[offset + IDX(6,j,k)]
            + 12.0*dev_var_in[offset + IDX(5,j,k)]
            - 10.0*dev_var_in[offset + IDX(4,j,k)]
            +  3.0*dev_var_in[offset + IDX(3,j,k)]
        )/43.0/48.0*64*hx;
        output[IDX(5,j,k)] =  (     dev_var_in[offset + IDX(8,j,k)]
            -  6.0*dev_var_in[offset + IDX(7,j,k)]
            + 15.0*dev_var_in[offset + IDX(6,j,k)]
            - 19.0*dev_var_in[offset + IDX(5,j,k)]
            + 12.0*dev_var_in[offset + IDX(4,j,k)]
            -  3.0*dev_var_in[offset + IDX(3,j,k)]
        )/49.0/48.0*64*hx;
    }

        if ((bflag & (1u<<OCT_DIR_RIGHT)) && (i == 5)) {

        const int ie = nx - 3;
        output[IDX(ie-3,j,k)] = ( dev_var_in[offset + IDX(ie-6,j,k)]
            - 6.0*dev_var_in[offset + IDX(ie-5,j,k)]
            + 15.0*dev_var_in[offset + IDX(ie-4,j,k)]
            - 19.0*dev_var_in[offset + IDX(ie-3,j,k)]
            + 12.0*dev_var_in[offset + IDX(ie-2,j,k)]
            -  3.0*dev_var_in[offset + IDX(ie-1,j,k)]
        )/49.0/48.0*64*hx;

        output[IDX(ie-2,j,k)] =  ( dev_var_in[offset + IDX(ie-5,j,k)]
            -  6.0*dev_var_in[offset + IDX(ie-4,j,k)]
            + 12.0*dev_var_in[offset + IDX(ie-3,j,k)]
            - 10.0*dev_var_in[offset + IDX(ie-2,j,k)]
            +  3.0*dev_var_in[offset + IDX(ie-1,j,k)]
        )/43.0/48.0*64*hx;


        output[IDX(ie-1,j,k)] = ( dev_var_in[offset + IDX(ie-4,j,k)]
            -  3.0*dev_var_in[offset + IDX(ie-3,j,k)]
            +  3.0*dev_var_in[offset + IDX(ie-2,j,k)]
            -      dev_var_in[offset + IDX(ie-1,j,k)]
        )/59.0/48.0*64*hx;
    }
} 

 
__device__ void device_calc_ko_deriv_y(double * output, double * dev_var_in,
    const int offset, double hy, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){

    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-8)) + 4;
    int k = (id/(sz_y-8)/(sz_x-6)) + 3;

    if(i >= nx-3 || j >= ny-4 || k >= nz-3) return;

    if(j==4) {
        int jb=3;
        output[IDX(i,jb,k)] = (-1.0 / 64.0 / hy) *
        (
                -      dev_var_in[offset + IDX(i,jb+4,k)]
                +  6.0*dev_var_in[offset + IDX(i,jb+3,k)]
                - 15.0*dev_var_in[offset + IDX(i,jb+2,k)]
                + 20.0*dev_var_in[offset + IDX(i,jb+1,k)]
                - 15.0*dev_var_in[offset + IDX(i,jb,k)]
                +  6.0*dev_var_in[offset + IDX(i,jb-1,k)]
                -      dev_var_in[offset + IDX(i,jb-2,k)]
        );
    }

    int pp = IDX(i, j, k);

    output[pp] = (-1.0 / 64.0 / hy) *
    (
    -      dev_var_in[offset + pp-3*nx]
    +  6.0*dev_var_in[offset + pp-2*nx]
    - 15.0*dev_var_in[offset + pp-nx]
    + 20.0*dev_var_in[offset + pp]
    - 15.0*dev_var_in[offset + pp+nx]
    +  6.0*dev_var_in[offset + pp+2*nx]
    -      dev_var_in[offset + pp+3*nx]
    );

    if(j==5) {
        int je = ny - 3;
        output[IDX(i,je-1,k)] = (-1.0 / 64.0 / hy) *
        (
                -      dev_var_in[offset + IDX(i,je+1,k)]
                +  6.0*dev_var_in[offset + IDX(i,je,k)]
                - 15.0*dev_var_in[offset + IDX(i,je-1,k)]
                + 20.0*dev_var_in[offset + IDX(i,je-2,k)]
                - 15.0*dev_var_in[offset + IDX(i,je-3,k)]
                +  6.0*dev_var_in[offset + IDX(i,je-4,k)]
                -      dev_var_in[offset + IDX(i,je-5,k)]
        );
    }
        if ((bflag & (1u<<OCT_DIR_DOWN)) && (j == 4)) {

        output[IDX(i,3,k)] =  (      dev_var_in[offset +IDX(i,6,k)]
            - 3.0*dev_var_in[offset +IDX(i,5,k)]
            + 3.0*dev_var_in[offset + IDX(i,4,k)]
            -     dev_var_in[offset + IDX(i,3,k)]
        )/59.0/48.0*64*hy;
        output[IDX(i,4,k)] =  (     dev_var_in[offset + IDX(i,7,k)]
            -  6.0*dev_var_in[offset + IDX(i,6,k)]
            + 12.0*dev_var_in[offset + IDX(i,5,k)]
            - 10.0*dev_var_in[offset + IDX(i,4,k)]
            +  3.0*dev_var_in[offset + IDX(i,3,k)]
        )/43.0/48.0*64*hy;
        output[IDX(i,5,k)] =  (     dev_var_in[offset + IDX(i,8,k)]
            -  6.0*dev_var_in[offset + IDX(i,7,k)]
            + 15.0*dev_var_in[offset + IDX(i,6,k)]
            - 19.0*dev_var_in[offset + IDX(i,5,k)]
            + 12.0*dev_var_in[offset + IDX(i,4,k)]
            -  3.0*dev_var_in[offset + IDX(i,3,k)]
        )/49.0/48.0*64*hy;
    }

        if ((bflag & (1u<<OCT_DIR_UP)) && (j == 5)) {

        const int je = ny - 3;
        output[IDX(i,je-3,k)] = (dev_var_in[offset + IDX(i,je-6,k)]
        -  6.0*dev_var_in[offset + IDX(i,je-5,k)]
        + 15.0*dev_var_in[offset + IDX(i,je-4,k)]
        - 19.0*dev_var_in[offset + IDX(i,je-3,k)]
        + 12.0*dev_var_in[offset + IDX(i,je-2,k)]
        -  3.0*dev_var_in[offset + IDX(i,je-1,k)]
        )/49.0/48.0*64*hy;

        output[IDX(i,je-2,k)] = (dev_var_in[offset + IDX(i,je-5,k)]
        -  6.0*dev_var_in[offset + IDX(i,je-4,k)]
        + 12.0*dev_var_in[offset + IDX(i,je-3,k)]
        - 10.0*dev_var_in[offset + IDX(i,je-2,k)]
        +  3.0*dev_var_in[offset + IDX(i,je-1,k)]
        )/43.0/48.0*64*hy;


        output[IDX(i,je-1,k)] = ( dev_var_in[offset + IDX(i,je-4,k)]
            -  3.0*dev_var_in[offset + IDX(i,je-3,k)]
            +  3.0*dev_var_in[offset + IDX(i,je-2,k)]
            -      dev_var_in[offset + IDX(i,je-1,k)]
        )/59.0/48.0*64*hy;
    }
}



__device__ void device_calc_ko_deriv_z(double * output, double * dev_var_in,
    const int offset, double hz, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){

    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(sz_x-6) + 3;
    int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
    int k = (id/(sz_y-6)/(sz_x-6)) + 4;

    if(i >= nx-3 || j >= ny-3 || k >= nz-4) return;

    if(k==4) {
        int kb=3;
        output[IDX(i,j,kb)] = (-1.0 / 64.0 / hz) *
        (
                -      dev_var_in[offset + IDX(i,j,kb+4)]
                +  6.0*dev_var_in[offset + IDX(i,j,kb+3)]
                - 15.0*dev_var_in[offset + IDX(i,j,kb+2)]
                + 20.0*dev_var_in[offset + IDX(i,j,kb+1)]
                - 15.0*dev_var_in[offset + IDX(i,j,kb)]
                +  6.0*dev_var_in[offset + IDX(i,j,kb-1)]
                -      dev_var_in[offset + IDX(i,j,kb-2)]
        );
    }

    int pp = IDX(i, j, k);
    int n = nx * ny;
    output[pp] = (-1.0 / 64.0 / hz) *
    (
    -      dev_var_in[offset + pp-3*n]
    +  6.0*dev_var_in[offset + pp-2*n]
    - 15.0*dev_var_in[offset + pp-n]
    + 20.0*dev_var_in[offset + pp]
    - 15.0*dev_var_in[offset + pp+n]
    +  6.0*dev_var_in[offset + pp+2*n]
    -      dev_var_in[offset + pp+3*n]
    );

    if(k==5) {
        int ke = nz - 3;
        output[IDX(i,j,ke-1)] = (-1.0 / 64.0 / hz) *
        (
                -      dev_var_in[offset + IDX(i,j,ke+1)]
                +  6.0*dev_var_in[offset + IDX(i,j,ke)]
                - 15.0*dev_var_in[offset + IDX(i,j,ke-1)]
                + 20.0*dev_var_in[offset + IDX(i,j,ke-2)]
                - 15.0*dev_var_in[offset + IDX(i,j,ke-3)]
                +  6.0*dev_var_in[offset + IDX(i,j,ke-4)]
                -      dev_var_in[offset + IDX(i,j,ke-5)]
        );
    }

    if ((bflag & (1u<<OCT_DIR_BACK)) && (k == 4)) {

        output[IDX(i,3,k)] =  (      dev_var_in[offset +IDX(i,k,6)]
            - 3.0*dev_var_in[offset +IDX(i,k,5)]
            + 3.0*dev_var_in[offset + IDX(i,k,4)]
            -     dev_var_in[offset + IDX(i,k,3)]
        )/59.0/48.0*64*hz;
        output[IDX(i,j,4)] =  (     dev_var_in[offset + IDX(i,j,7)]
            -  6.0*dev_var_in[offset + IDX(i,j,6)]
            + 12.0*dev_var_in[offset + IDX(i,j,5)]
            - 10.0*dev_var_in[offset + IDX(i,j,4)]
            +  3.0*dev_var_in[offset + IDX(i,j,3)]
        )/43.0/48.0*64*hz;
        output[IDX(i,j,5)] =  (     dev_var_in[offset + IDX(i,j,8)]
            -  6.0*dev_var_in[offset + IDX(i,j,7)]
            + 15.0*dev_var_in[offset + IDX(i,j,6)]
            - 19.0*dev_var_in[offset + IDX(i,j,5)]
            + 12.0*dev_var_in[offset + IDX(i,j,4)]
            -  3.0*dev_var_in[offset + IDX(i,j,3)]
        )/49.0/48.0*64*hz;
    }

    if ((bflag & (1u<<OCT_DIR_FRONT)) && (k == 5)) {

        const int ke = nz - 3;
        output[IDX(i,j,ke-3)] = (    dev_var_in[offset + IDX(i,j,ke-6)]
            -  6.0*dev_var_in[offset + IDX(i,j,ke-5)]
            + 15.0*dev_var_in[offset + IDX(i,j,ke-4)]
            - 19.0*dev_var_in[offset + IDX(i,j,ke-3)]
            + 12.0*dev_var_in[offset + IDX(i,j,ke-2)]
            -  3.0*dev_var_in[offset + IDX(i,j,ke-1)]
        )/49.0/48.0*64*hz;

        output[IDX(i,j,ke-2)] = (   dev_var_in[offset + IDX(i,j,ke-5)]
            -  6.0*dev_var_in[offset + IDX(i,j,ke-4)]
            + 12.0*dev_var_in[offset + IDX(i,j,ke-3)]
            - 10.0*dev_var_in[offset + IDX(i,j,ke-2)]
            +  3.0*dev_var_in[offset + IDX(i,j,ke-1)]
        )/43.0/48.0*64*hz;


        output[IDX(i,j,ke-1)] = (   dev_var_in[offset + IDX(i,j,ke-4)]
            -  3.0*dev_var_in[offset + IDX(i,j,ke-3)]
            +  3.0*dev_var_in[offset + IDX(i,j,ke-2)]
            -      dev_var_in[offset + IDX(i,j,ke-1)]
        )/59.0/48.0*64*hz;
    }
}

 
 __global__ void cuda_calc_ko_deriv_all(double * dev_var_in, double hx, double hy, double hz, 
    int sz_x, int sz_y, int sz_z, int bflag,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
 ) {
    int nx = sz_x;
    int ny = sz_y;
    int nz = sz_z;

    #include "bssnrhs_cuda_ko_derivs.h"
 }

 void calc_ko_deriv_all( double * dev_var_in, double hx, double hy, double hz, int sz_x, 
    int sz_y, int sz_z, int bflag, hipStream_t stream,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
 )
 {
    const int ie = sz_x - 3;//x direction
    const int je = sz_y - 3;//y direction
    const int ke = sz_z - 3;//z direction
 
    int total_points = ie*je*ke;
    int blocks = ceil(1.0*total_points/threads_per_block);

    cuda_calc_ko_deriv_all <<< blocks, threads_per_block, 0, stream >>> (
                    dev_var_in, hx, hy, hz, sz_x, sz_y, sz_z, bflag,
                  #include "list_of_args.h"
                  ,
                  #include "list_of_offset_args.h"
                );
 }