#include "hip/hip_runtime.h"
#include "derivs_cuda.h"

 
__device__ void device_calc_deriv_x(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
    
    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_deriv; id<(thread_id+1)*thread_load_deriv; id++){

        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-2)) + 1;
        int k = (id/(sz_y-2)/(sz_x-6)) + 1;

        if(i >= nx-3 || j >= ny-1 || k >= nz-1) return;

        int pp = IDX(i, j, k);

        output[pp] = (dev_var_in[offset + pp - 2] - 8.0*dev_var_in[offset
                                                + pp - 1] + 8.0*dev_var_in[offset + pp + 1]
                                                - dev_var_in[offset + pp + 2] )*((1.0/hx)/12.0);

        if ((bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
            int pp3 = IDX(3, j, k);
            int pp4 = IDX(4, j, k);
            int pp5 = IDX(5, j, k);
            output[pp3] = ((-3)*dev_var_in[offset + pp3] + 4*dev_var_in[offset
                                                        + pp4] - dev_var_in[offset + pp5]) * 0.5 / hx;
            output[pp4] = (dev_var_in[offset + pp5] - dev_var_in[offset
                                                + pp3]) * (0.50/hx);
        }

        if ((bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
            int pp2 = IDX(nx-5, j, k); // IDX(ie-2,j,k)
            int pp3 = IDX(nx-6, j, k); // IDX(ie-3,j,k)
            int pp1 = IDX(nx-4,j,k); // IDX(ie-1,j,k)
            output[pp2] = (dev_var_in[offset + pp1] - dev_var_in[offset + pp3])
            * 0.50 / hx;
            output[pp1] = (dev_var_in[offset + pp3]- 4.0 * dev_var_in[offset + pp2]
            + 3.0 * dev_var_in[offset + pp1]) * 0.50 / hx;

        }
    }

} 
 
 __device__ void device_calc_deriv_y(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
        
    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_deriv; id<(thread_id+1)*thread_load_deriv; id++){

        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
        int k = (id/(sz_y-6)/(sz_x-6)) + 1;

        if(i >= nx-3 || j >= ny-3 || k >= nz-1) return;

        int pp = IDX(i, j, k);

        output[pp] = (dev_var_in[offset + pp - 2*nx]
        - 8.0*dev_var_in[offset + pp - nx]
        + 8.0*dev_var_in[offset + pp + nx]
        - dev_var_in[offset + pp + 2*nx] )*((1.0/hx)/12.0);


        if ((bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
            int pp3 = IDX(i, 3, k);
            int pp4 = IDX(i, 4, k);
            int pp5 = IDX(i, 5, k);

            output[pp3] = ((-3)*dev_var_in[(offset) + pp3] +  4*dev_var_in[(offset) + pp4]
            - dev_var_in[(offset) + pp5]) * 0.5 / hx;
            output[pp4] = (dev_var_in[(offset) + pp5] - dev_var_in[(offset) + pp3])
            * (0.50/hx);

        }

        if ((bflag & (1u<<OCT_DIR_UP)) && j==4)  {
            int pp2 = IDX(i, ny-5, k); // IDX(i,je-2,k)
            int pp3 = IDX(i, ny-6, k); // IDX(i,je-3,k)
            int pp1 = IDX(i, ny-4, k); // IDX(i,je-1,k)

            output[pp2] = (dev_var_in[(offset) + pp1] - dev_var_in[(offset) + pp3])
            * 0.50 / hx;
            output[pp1] = (dev_var_in[(offset) + pp3]- 4.0 * dev_var_in[(offset) + pp2]
            + 3.0 * dev_var_in[(offset) + pp1]) * 0.50 / hx;

        }
    }
}

 
 __device__ void device_calc_deriv_z(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
        
    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_deriv; id<(thread_id+1)*thread_load_deriv; id++){
    
        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
        int k = (id/(sz_y-6)/(sz_x-6)) + 3;
        
        if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

        int pp = IDX(i, j, k);

        int n = nx * ny;

        output[pp] = (dev_var_in[offset + pp - 2*n] - 8.0*dev_var_in[offset + pp - n]
        + 8.0*dev_var_in[offset + pp + n] - dev_var_in[offset + pp + 2*n])
        * ((1.0/hx)/12);

        if ((bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
            int pp3 = IDX(i, j, 3); // IDX(i, j, 3)
            int pp4 = IDX(i, j, 4); // IDX(i,j,4)
            int pp5 = IDX(i, j, 5); // IDX(i,j,5)

            output[pp3] = ((-3)*dev_var_in[offset + pp3] + 4*dev_var_in[offset + pp4]
            - dev_var_in[offset + pp5]) * 0.5 / hx;
            output[pp4] = (dev_var_in[offset + pp5] - dev_var_in[offset + pp3])
            * (0.50/hx);
        }

        if ((bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
            int pp2 = IDX(i, j, nz-5); // IDX(i,j,ke-2)
            int pp3 = IDX(i, j, nz-6); // IDX(i,j,ke-3)
            int pp1 = IDX(i, j, nz-4); // IDX(i,j,ke-1)

            output[pp2] = (dev_var_in[offset + pp1] - dev_var_in[offset + pp3])
            * 0.50 / hx;
            output[pp1] = (dev_var_in[offset + pp3]- 4.0 * dev_var_in[offset + pp2]
            + 3.0 * dev_var_in[offset + pp1]) * 0.50 / hx;
        }
    }
}

__device__ void device_calc_deriv_xx(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
        
    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_deriv; id<(thread_id+1)*thread_load_deriv; id++){

        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
        int k = (id/(sz_y-6)/(sz_x-6)) + 3;

        if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

        int pp = IDX(i, j, k);

        output[pp] = ((-1)*dev_var_in[offset + pp - 2]
        + 16.0*dev_var_in[offset + pp - 1]
        - 30.0*dev_var_in[offset + pp]
        + 16.0*dev_var_in[offset + pp + 1]
        - dev_var_in[offset + pp + 2]
        )*(1.0/(hx*hx))/12.0;

        if ((bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
            int pp3 = IDX(3, j, k);
            int pp4 = IDX(4, j, k);
            int pp5 = IDX(5, j, k);
            int pp6 = IDX(6, j, k);

            output[pp3] = (
            2.0 *   dev_var_in[offset + pp3]
            -   5.0 *   dev_var_in[offset + pp4]
            +   4.0 *   dev_var_in[offset + pp5]
            -           dev_var_in[offset + pp6]
            ) * 1.0/(hx*hx);

            output[pp4] = (
            dev_var_in[offset + pp3]
            -   2.0 *   dev_var_in[offset + pp4]
            +           dev_var_in[offset + pp5]
            ) * 1.0/(hx*hx);

        }

        if ((bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
            int pp1 = IDX(nx - 4, j, k); // IDX(ie-1,j,k)
            int pp2 = IDX(nx - 5, j, k); // IDX(ie-2,j,k)
            int pp3 = IDX(nx - 6, j, k); // IDX(ie-3,j,k)
            int pp4 = IDX(nx - 7, j, k); // IDX(ie-4,j,k)

            output[pp2] = (
            dev_var_in[offset + pp3]
            -   2.0 *   dev_var_in[offset + pp2]
            +           dev_var_in[offset + pp1]
            ) * 1.0/(hx*hx);


            output[pp1] = (
            -   1.0 *   dev_var_in[offset + pp4]
            +   4.0 *   dev_var_in[offset + pp3]
            -   5.0 *   dev_var_in[offset + pp2]
            +   2.0 *   dev_var_in[offset + pp1]
            ) * 1.0/(hx*hx);
        }
    }
}
 
__device__ void device_calc_deriv_yy(double * output, double * dev_var_in,
    const int offset, double hy, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
        
    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_deriv; id<(thread_id+1)*thread_load_deriv; id++){
        
        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
        int k = (id/(sz_y-6)/(sz_x-6)) + 3;
        
        if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

        int pp = IDX(i, j, k);

        output[pp] = ((-1)*dev_var_in[offset + pp - 2*nx]
        + 16.0*dev_var_in[offset + pp - nx]
        - 30.0*dev_var_in[offset + pp]
        + 16.0*dev_var_in[offset + pp + nx]
        - dev_var_in[offset + pp + 2*nx]
        )*(1.0/(hy*hy))/12.0;

        if ((bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
            int pp3 = IDX(i, 3, k);
            int pp4 = IDX(i, 4, k);
            int pp5 = IDX(i, 5, k);
            int pp6 = IDX(i, 6, k);

            output[pp3] = (
            2.0 *   dev_var_in[offset + pp3]
            -   5.0 *   dev_var_in[offset + pp4]
            +   4.0 *   dev_var_in[offset + pp5]
            -           dev_var_in[offset + pp6]
            ) * 1.0/(hy*hy);

            output[pp4] = (
            dev_var_in[offset + pp3]
            -   2.0 *   dev_var_in[offset + pp4]
            +           dev_var_in[offset + pp5]
            ) * 1.0/(hy*hy);
        }

        if ((bflag & (1u<<OCT_DIR_UP)) && j==4)  {
            int pp1 = IDX(i, ny - 4, k);
            int pp2 = IDX(i, ny - 5, k);
            int pp3 = IDX(i, ny - 6, k);
            int pp4 = IDX(i, ny - 7, k);

            output[pp2] = (
            dev_var_in[offset + pp3]
            -   2.0 *   dev_var_in[offset + pp2]
            +           dev_var_in[offset + pp1]
            ) * 1.0/(hy*hy);


            output[pp1] = (
            -   1.0 *   dev_var_in[offset + pp4]
            +   4.0 *   dev_var_in[offset + pp3]
            -   5.0 *   dev_var_in[offset + pp2]
            +   2.0 *   dev_var_in[offset + pp1]
            ) * 1.0/(hy*hy);

        }
    }
}

__device__ void device_calc_deriv_zz(double * output, double * dev_var_in,
    const int offset, double hz, int bflag,
    int nx,int ny,int nz, int sz_x, int sz_y, int sz_z){
        
    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_deriv; id<(thread_id+1)*thread_load_deriv; id++){
        
        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
        int k = (id/(sz_y-6)/(sz_x-6)) + 3;
        
            if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

            int pp = IDX(i, j, k);

            int n = nx * ny;

            output[pp] = ((-1)*dev_var_in[offset + pp - 2*n]
            + 16.0*dev_var_in[offset + pp - n]
            - 30.0*dev_var_in[offset + pp]
            + 16.0*dev_var_in[offset + pp + n]
            - dev_var_in[offset + pp + 2*n]
            )*(1.0/(hz*hz))/12.0;

            if ((bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
                int pp3 = IDX(i, j, 3);
                int pp4 = IDX(i, j, 4);
                int pp5 = IDX(i, j, 5);
                int pp6 = IDX(i, j, 6);

                output[pp3] = (
                2.0 *   dev_var_in[offset + pp3]
                -   5.0 *   dev_var_in[offset + pp4]
                +   4.0 *   dev_var_in[offset + pp5]
                -           dev_var_in[offset + pp6]
                ) * 1.0/(hz*hz);

                output[pp4] = (
                dev_var_in[offset + pp3]
                -   2.0 *   dev_var_in[offset + pp4]
                +           dev_var_in[offset + pp5]
                ) * 1.0/(hz*hz);
            }

            if ((bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
                int pp1 = IDX(i, j, nz - 4);
                int pp2 = IDX(i, j, nz - 5);
                int pp3 = IDX(i, j, nz - 6);
                int pp4 = IDX(i, j, nz - 7);

                output[pp2] = (
                dev_var_in[offset + pp3]
                -   2.0 *   dev_var_in[offset + pp2]
                +           dev_var_in[offset + pp1]
                ) * 1.0/(hz*hz);


                output[pp1] = (
                -   1.0 *   dev_var_in[offset + pp4]
                +   4.0 *   dev_var_in[offset + pp3]
                -   5.0 *   dev_var_in[offset + pp2]
                +   2.0 *   dev_var_in[offset + pp1]
                ) * 1.0/(hz*hz);
            }
    }
}
__global__ void calc_deriv42_first_part(double * dev_var_in, double hx, double hy, double hz, 
    int sz_x, int sz_y, int sz_z, int bflag,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
 ){
 
     int nx = sz_x;
     int ny = sz_y;
     int nz = sz_z;

 
 #include "bssnrhs_cuda_derivs_first_part.h"
 
 }
 
__global__ void calc_deriv42_second_part(double * dev_var_in, double hx, double hy, 
    double hz, int sz_x, int sz_y, int sz_z, int bflag,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
){
 
     int nx = sz_x;
     int ny = sz_y;
     int nz = sz_z;
 
 #include "bssnrhs_cuda_derivs_secondd_part.h"
 
 }
 
void cuda_calc_all(double * dev_var_in, double hx, double hy, double hz, int sz_x, 
    int sz_y, int sz_z, int bflag, hipStream_t stream,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
    ){
 
    const int ie = sz_x - 1;//x direction
    const int je = sz_y - 1;//y direction
    const int ke = sz_z - 1;//z direction
 
    int total_points = ceil(1.0*ie*je*ke/thread_load_deriv);
    int blocks = ceil(1.0*total_points/threads_per_block);

    calc_deriv42_first_part <<< blocks, threads_per_block, 0, stream >>> (
                      dev_var_in, hx, hy, hz, sz_x, sz_y, sz_z, bflag,
                    #include "list_of_args.h"
                    ,
                    #include "list_of_offset_args.h"
             );

    calc_deriv42_second_part <<< blocks, threads_per_block, 0, stream >>> (
                      dev_var_in, hx, hy, hz, sz_x, sz_y, sz_z, bflag,
                    #include "list_of_args.h"
                    ,
                    #include "list_of_offset_args.h"
             );
}

