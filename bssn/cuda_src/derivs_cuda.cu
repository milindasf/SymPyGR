#include "hip/hip_runtime.h"
/**
 * Created on: Feb 12, 2018
 * 		Author: Akila
 **/

 #include "derivs_cuda.h"
 
 __global__ void firstThreeForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*10;
    int y = threadIdx.y + blockIdx.x*10;
    int z = threadIdx.z + blockIdx.x*10;

    int i;
    int j;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[1]-3-3)<=y ){ return; } else { j = y+3; } //j handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[(*dev_u_offset) + pp - 2*dev_sz[0]] - 8.0*dev_var_in[(*dev_u_offset) + pp - dev_sz[0]] + 8.0*dev_var_in[(*dev_u_offset) + pp + dev_sz[0]] - dev_var_in[(*dev_u_offset) + pp + 2*dev_sz[0]] )*((1.0/dev_dy[0])/12.0);
    // printf("%f\n", output[pp]);
}

 __global__ void secondTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*30;
    int z = threadIdx.y + blockIdx.x*30;

    int i;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0];
    int ny = dev_sz[1];

    int pp3 = IDX(i, 3, k);
    int pp4 = IDX(i, 4, k);
    int pp5 = IDX(i, 5, k);

    output[pp3] = ((-3)*dev_var_in[(*dev_u_offset) + pp3] +  4*dev_var_in[(*dev_u_offset) + pp4] - dev_var_in[(*dev_u_offset) + pp5]) * 0.5 / dev_dy[0];
    output[pp4] = (dev_var_in[(*dev_u_offset) + pp5] - dev_var_in[(*dev_u_offset) + pp3]) * (0.50/dev_dy[0]);
    // printf("%f\n", output[pp3]);
 }

 __global__ void thirdTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*30;
    int z = threadIdx.y + blockIdx.x*30;

    int i;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0];
    int ny = dev_sz[1];

    int pp2 = IDX(i, dev_sz[1]-5, k); // IDX(i,je-2,k)
    int pp3 = IDX(i, dev_sz[1]-6, k); // IDX(i,je-3,k)
    int pp1 = IDX(i, dev_sz[1]-4, k); // IDX(i,je-1,k)

    output[pp2] = (dev_var_in[(*dev_u_offset) + pp1] - dev_var_in[(*dev_u_offset) + pp3]) * 0.50 / dev_dy[0];
    output[pp1] = (dev_var_in[(*dev_u_offset) + pp3]- 4.0 * dev_var_in[(*dev_u_offset) + pp2]+ 3.0 * dev_var_in[(*dev_u_offset) + pp1]) * 0.50 / dev_dy[0];
    printf("%f\n", output[pp1]);
 }
 
void cuda_deriv42_y(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, int * dev_sz, unsigned bflag, const unsigned int * host_sz)
 {
    int zblocks = ((host_sz[2]-1)/10)+1;
    int yblocks = ((host_sz[0]-3)/10)+1;
    int xblocks = ((host_sz[1]-3)/10)+1;
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }


    if (bflag & (1u<<OCT_DIR_DOWN)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-1)/30)+1;
        int xblocks = ((host_sz[0]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        secondTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "secondTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching secondTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    }
    
    if (bflag & (1u<<OCT_DIR_UP)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-1)/30)+1;
        int xblocks = ((host_sz[0]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        thirdTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "thirdTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching thirdTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    } 

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    //   for (int k = 3; k < sz[2]-3; k++) {
    //     for (int j = 3; j < sz[1]-3; j++) {
    //       for (int i = 3; i < sz[0]-3; i++) {
    //         int pp = IDX(i,j,k);
    //         if(std::isnan(Dyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //       }
    //     }
    //   }
    // #endif
 }
 
 