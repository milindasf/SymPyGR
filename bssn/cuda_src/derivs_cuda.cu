#include "hip/hip_runtime.h"
/**
 * Created on: March 15, 2018
 * 		Author: Akila
 **/

 #include "derivs_cuda.h"
 
 __global__ void cuda_deriv42_y_firstThreeForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*10;
    int y = threadIdx.y + blockIdx.x*10;
    int z = threadIdx.z + blockIdx.x*10;

    int i;
    int j;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[1]-3-3)<=y ){ return; } else { j = y+3; } //j handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[(*dev_u_offset) + pp - 2*dev_sz[0]] - 8.0*dev_var_in[(*dev_u_offset) + pp - dev_sz[0]] + 8.0*dev_var_in[(*dev_u_offset) + pp + dev_sz[0]] - dev_var_in[(*dev_u_offset) + pp + 2*dev_sz[0]] )*((1.0/dev_dy[0])/12.0);
    // printf("%f\n", output[pp]);
}

 __global__ void cuda_deriv42_y_secondTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*30;
    int z = threadIdx.y + blockIdx.x*30;

    int i;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0];
    int ny = dev_sz[1];

    int pp3 = IDX(i, 3, k);
    int pp4 = IDX(i, 4, k);
    int pp5 = IDX(i, 5, k);

    output[pp3] = ((-3)*dev_var_in[(*dev_u_offset) + pp3] +  4*dev_var_in[(*dev_u_offset) + pp4] - dev_var_in[(*dev_u_offset) + pp5]) * 0.5 / dev_dy[0];
    output[pp4] = (dev_var_in[(*dev_u_offset) + pp5] - dev_var_in[(*dev_u_offset) + pp3]) * (0.50/dev_dy[0]);
    // printf("%f\n", output[pp3]);
 }

 __global__ void cuda_deriv42_y_thirdTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*30;
    int z = threadIdx.y + blockIdx.x*30;

    int i;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0];
    int ny = dev_sz[1];

    int pp2 = IDX(i, dev_sz[1]-5, k); // IDX(i,je-2,k)
    int pp3 = IDX(i, dev_sz[1]-6, k); // IDX(i,je-3,k)
    int pp1 = IDX(i, dev_sz[1]-4, k); // IDX(i,je-1,k)

    output[pp2] = (dev_var_in[(*dev_u_offset) + pp1] - dev_var_in[(*dev_u_offset) + pp3]) * 0.50 / dev_dy[0];
    output[pp1] = (dev_var_in[(*dev_u_offset) + pp3]- 4.0 * dev_var_in[(*dev_u_offset) + pp2]+ 3.0 * dev_var_in[(*dev_u_offset) + pp1]) * 0.50 / dev_dy[0];
    // printf("%f\n", output[pp1]);
 }
 
 // Please some one verify the below kernals carefully -------------------------------------------------------------------------------
 __global__ void cuda_deriv42_x_firstThreeForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
    int x = threadIdx.x + blockIdx.x*10;
    int y = threadIdx.y + blockIdx.x*10;
    int z = threadIdx.z + blockIdx.x*10;

    int i;
    int j;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[1]-1-1)<=y ){ return; } else { j = y+1; } //j handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[(*dev_u_offset) + pp - 2] - 8.0*dev_var_in[(*dev_u_offset) + pp - 1] + 8.0*dev_var_in[(*dev_u_offset) + pp + 1] - dev_var_in[(*dev_u_offset) + pp + 2] )*((1.0/dev_dy[0])/12.0);
    // printf("%f\n", output[pp]);
}

__global__ void cuda_deriv42_x_secondTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int j;
   int k;

   if( (dev_sz[1]-1-1)<=x ){ return; } else { j = x+1; } 
   if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp3 = IDX(3, j, k);
   int pp4 = IDX(4, j, k);
   int pp5 = IDX(5, j, k);

   output[pp3] = ((-3)*dev_var_in[(*dev_u_offset) + pp3] + 4*dev_var_in[(*dev_u_offset) + pp4] - dev_var_in[(*dev_u_offset) + pp5]) * 0.5 / dev_dy[0];
   output[pp4] = (dev_var_in[(*dev_u_offset) + pp5] - dev_var_in[(*dev_u_offset) + pp3]) * (0.50/dev_dy[0]);
   // printf("%f\n", output[pp3]);
}

__global__ void cuda_deriv42_x_thirdTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int j;
   int k;

   if( (dev_sz[1]-1-1)<=x ){ return; } else { j = x+1; } 
   if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp2 = IDX(dev_sz[0]-5, j, k); // IDX(ie-2,j,k)
   int pp3 = IDX(dev_sz[0]-6, j, k); // IDX(ie-3,j,k)
   int pp1 = IDX(dev_sz[0]-4,j,k); // IDX(ie-1,j,k)

   output[pp2] = (dev_var_in[(*dev_u_offset) + pp1] - dev_var_in[(*dev_u_offset) + pp3]) * 0.50 / dev_dy[0];
   output[pp1] = (dev_var_in[(*dev_u_offset) + pp3]- 4.0 * dev_var_in[(*dev_u_offset) + pp2]+ 3.0 * dev_var_in[(*dev_u_offset) + pp1]) * 0.50 / dev_dy[0];
   // printf("%f\n", output[pp1]);
}

__global__ void cuda_deriv42_z_firstThreeForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*10;
   int y = threadIdx.y + blockIdx.x*10;
   int z = threadIdx.z + blockIdx.x*10;

   int i;
   int j;
   int k;

   if( (dev_sz[2]-3-3)<=x ){ return; } else { k = x+3; }
   if( (dev_sz[0]-3-3)<=y ){ return; } else { i = y+3; }
   if( (dev_sz[1]-3-3)<=z ){ return; } else { j = z+3; }

   int nx = dev_sz[0]; 
   int ny = dev_sz[1]; 
   int n = nx * ny;
   int pp = IDX(i, j, k);
   
   output[pp] = (dev_var_in[(*dev_u_offset) + pp - 2*n] - 8.0*dev_var_in[(*dev_u_offset) + pp - n] + 8.0*dev_var_in[(*dev_u_offset) + pp + n] - dev_var_in[(*dev_u_offset) + pp + 2*n]) * ((1.0/dev_dy[0])/12);
   // printf("%f\n", output[pp]);
}

__global__ void cuda_deriv42_z_secondTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int j;
   int i;

   if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } 
   if( (dev_sz[1]-3-3)<=z ){ return; } else { j = z+3; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp3 = IDX(i, j, 3); // IDX(i, j, 3)
   int pp4 = IDX(i, j, 4); // IDX(i,j,4)
   int pp5 = IDX(i, j, 5); // IDX(i,j,5)

   output[pp3] = ((-3)*dev_var_in[(*dev_u_offset) + pp3] + 4*dev_var_in[(*dev_u_offset) + pp4] - dev_var_in[(*dev_u_offset) + pp5]) * 0.5 / dev_dy[0];
   output[pp4] = (dev_var_in[(*dev_u_offset) + pp5] - dev_var_in[(*dev_u_offset) + pp3]) * (0.50/dev_dy[0]);
   // printf("%f\n", output[pp3]);
}

__global__ void cuda_deriv42_z_thirdTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int i;
   int j;

   if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } 
   if( (dev_sz[1]-3-3)<=z ){ return; } else { j = z+3; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp2 = IDX(i, j, dev_sz[2]-5); // IDX(i,j,ke-2)
   int pp3 = IDX(i, j, dev_sz[2]-6); // IDX(i,j,ke-3)
   int pp1 = IDX(i, j, dev_sz[2]-4); // IDX(i,j,ke-1)

   output[pp2] = (dev_var_in[(*dev_u_offset) + pp1] - dev_var_in[(*dev_u_offset) + pp3]) * 0.50 / dev_dy[0];
   output[pp1] = (dev_var_in[(*dev_u_offset) + pp3]- 4.0 * dev_var_in[(*dev_u_offset) + pp2]+ 3.0 * dev_var_in[(*dev_u_offset) + pp1]) * 0.50 / dev_dy[0];
   // printf("%f\n", output[pp1]);
}

void cuda_deriv42_y(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, int * dev_sz, unsigned bflag, const unsigned int * host_sz)
 {
    int zblocks = ((host_sz[2]-1)/10)+1;
    int yblocks = ((host_sz[0]-3)/10)+1;
    int xblocks = ((host_sz[1]-3)/10)+1;
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    cuda_deriv42_y_firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_y_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_y_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }


    if (bflag & (1u<<OCT_DIR_DOWN)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-1)/30)+1;
        int xblocks = ((host_sz[0]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_y_secondTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_y_secondTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_y_secondTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    }
    
    if (bflag & (1u<<OCT_DIR_UP)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-1)/30)+1;
        int xblocks = ((host_sz[0]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_y_thirdTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_y_thirdTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_y_thirdTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    } 

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    //   for (int k = 3; k < sz[2]-3; k++) {
    //     for (int j = 3; j < sz[1]-3; j++) {
    //       for (int i = 3; i < sz[0]-3; i++) {
    //         int pp = IDX(i,j,k);
    //         if(std::isnan(Dyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //       }
    //     }
    //   }
    // #endif
 }
 
void cuda_deriv42_x(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, int * dev_sz, unsigned bflag, const unsigned int * host_sz)
{
    int zblocks = ((host_sz[2]-1)/10)+1; // k
    int yblocks = ((host_sz[1]-1)/10)+1; // j
    int xblocks = ((host_sz[0]-3)/10)+1; // i
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    cuda_deriv42_x_firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_x_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_x_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    if (bflag & (1u<<OCT_DIR_LEFT)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-1)/30)+1;
        int xblocks = ((host_sz[1]-1)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_x_secondTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_x_secondTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_x_secondTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    }

    if (bflag & (1u<<OCT_DIR_RIGHT)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-1)/30)+1;
        int xblocks = ((host_sz[1]-1)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_x_thirdTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_x_thirdTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_x_thirdTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    } 

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    //     #pragma message("DEBUG_DERIVS_COMP: ON")
    //     for (int k = 3; k < sz[2]-3; k++) {
    //         for (int j = 3; j < sz[1]-3; j++) {
    //             for (int i = 3; i < sz[0]-3; i++) {
    //                 int pp = IDX(i,j,k);
    //                 if(isnan(Dxu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //                 }
    //             }
    //         }
    // #endif
}

void cuda_deriv42_z(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, int * dev_sz, unsigned bflag, const unsigned int * host_sz)
{
    int zblocks = ((host_sz[2]-1)/10)+1; // k
    int yblocks = ((host_sz[1]-1)/10)+1; // j
    int xblocks = ((host_sz[0]-3)/10)+1; // i
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    cuda_deriv42_z_firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_z_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_z_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    if (bflag & (1u<<OCT_DIR_BACK)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[0]-3)/30)+1;
        int xblocks = ((host_sz[0]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_z_secondTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_z_secondTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_z_secondTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    }

    if (bflag & (1u<<OCT_DIR_FRONT)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[0]-3)/30)+1;
        int xblocks = ((host_sz[0]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_z_thirdTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_z_thirdTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_z_thirdTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    } 

    //   #ifdef DEBUG_DERIVS_COMP
    //     for (int k = kb; k < ke; k++) {
    //       for (int j = jb; j < je; j++) {
    //         for (int i = ib; i < ie; i++) {
    //           int pp = IDX(i,j,k);
    //           if(std::isnan(Dzu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //         }
    //       }
    //     }
    //   #endif
}
 