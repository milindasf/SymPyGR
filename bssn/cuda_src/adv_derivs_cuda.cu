#include "hip/hip_runtime.h"
#include "derivs_cuda.h"

__device__ void device_calc_adv_x(double * output, double * dev_var_in,
    const int offset, double hx, int bflag,
    int nx,int ny,int nz, int betax, int sz_x, int sz_y, int sz_z){

    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_adv_deriv; id<(thread_id+1)*thread_load_adv_deriv; id++){
            
        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
        int k = (id/(sz_y-6)/(sz_x-6)) + 3;

        double idx_by_2 = 0.50 * (1.0 / hx);
        double idx_by_12 = (1.0 / hx)/12;

        if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

        int pp = IDX(i, j, k);

        if (dev_var_in[betax + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[offset + pp - 1]
            - 10.0 * dev_var_in[offset + pp]
            + 18.0 * dev_var_in[offset + pp + 1]
            -  6.0 * dev_var_in[offset + pp + 2]
            +        dev_var_in[offset + pp + 3]
            ) * idx_by_12;
        }
        else {
            output[pp] = ( -        dev_var_in[offset + pp - 3]
            +  6.0 * dev_var_in[offset + pp - 2]
            - 18.0 * dev_var_in[offset + pp - 1]
            + 10.0 * dev_var_in[offset + pp]
            +  3.0 * dev_var_in[offset + pp +1]
            ) * idx_by_12;
        }

        if ((bflag & (1u<<OCT_DIR_LEFT)) && (i == 3)) {

            output[IDX(3,j,k)] = ( -  3.0 * dev_var_in[offset + IDX(3,j,k)]
            +  4.0 * dev_var_in[offset + IDX(4,j,k)]
            -        dev_var_in[offset + IDX(5,j,k)]
            ) * idx_by_2;

            if (dev_var_in[betax + IDX(4,j,k)] > 0.0) {
            output[IDX(4,j,k)] = ( -  3.0 * dev_var_in[offset + IDX(4,j,k)]
                +  4.0 * dev_var_in[offset + IDX(5,j,k)]
                -        dev_var_in[offset + IDX(6,j,k)]
            ) * idx_by_2;
            }
            else {
            output[IDX(4,j,k)] = ( -         dev_var_in[offset + IDX(3,j,k)]
                +        dev_var_in[offset + IDX(5,j,k)]
            ) * idx_by_2;
            }

            if (dev_var_in[betax + IDX(5,j,k)] > 0.0 ) {
            output[IDX(5,j,k)] = (-  3.0 * dev_var_in[offset + IDX(4,j,k)]
                - 10.0 * dev_var_in[offset + IDX(5,j,k)]
                + 18.0 * dev_var_in[offset + IDX(6,j,k)]
                -  6.0 * dev_var_in[offset + IDX(7,j,k)]
                +        dev_var_in[offset + IDX(8,j,k)]
            ) * idx_by_12;
            }
            else {
            output[IDX(5,j,k)] = (           dev_var_in[offset + IDX(3,j,k)]
                        -  4.0 * dev_var_in[offset + IDX(4,j,k)]
                        +  3.0 * dev_var_in[offset + IDX(5,j,k)]
            ) * idx_by_2;
            }
        }

        if ((bflag & (1u<<OCT_DIR_RIGHT)) && (i == 4)) {

            const int ie = nx - 3;

            if ( dev_var_in[betax + IDX(ie-3,j,k)] < 0.0 ) {
                output[IDX(ie-3,j,k)] = (  - 3.0 * dev_var_in[offset + IDX(ie-3,j,k)]
                        + 4.0 * dev_var_in[offset + IDX(ie-2,j,k)]
                        -       dev_var_in[offset + IDX(ie-1,j,k)]
                    ) * idx_by_2;
            }
            else {
                output[IDX(ie-3,j,k)] = ( -   dev_var_in[offset + IDX(ie-6,j,k)]
                        +  6.0 * dev_var_in[offset + IDX(ie-5,j,k)]
                        - 18.0 * dev_var_in[offset + IDX(ie-4,j,k)]
                        + 10.0 * dev_var_in[offset + IDX(ie-3  ,j,k)]
                        +  3.0 * dev_var_in[offset + IDX(ie-2,j,k)]
                    ) * idx_by_12;
            }

            if (dev_var_in[betax + IDX(ie-2,j,k)] > 0.0 ) {
                output[IDX(ie-2,j,k)] = (  -  dev_var_in[offset + IDX(ie-3,j,k)]
                        +  dev_var_in[offset + IDX(ie-1,j,k)]
                    ) * idx_by_2;
            }
            else {
                output[IDX(ie-2,j,k)] = (     dev_var_in[offset + IDX(ie-4,j,k)]
                            - 4.0 * dev_var_in[offset + IDX(ie-3,j,k)]
                            + 3.0 * dev_var_in[offset + IDX(ie-2,j,k)]
                    ) * idx_by_2;
            }

            output[IDX(ie-1,j,k)] = (          dev_var_in[offset + IDX(ie-3,j,k)]
                        - 4.0 * dev_var_in[offset + IDX(ie-2,j,k)]
                        + 3.0 * dev_var_in[offset + IDX(ie-1,j,k)]
            ) * idx_by_2;
        }
    }
}
 
__device__ void device_calc_adv_y(double * output, double * dev_var_in,
    const int offset, double hy, int bflag,
    int nx,int ny,int nz, int betay, int sz_x, int sz_y, int sz_z){

    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_adv_deriv; id<(thread_id+1)*thread_load_adv_deriv; id++){
            
        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
        int k = (id/(sz_y-6)/(sz_x-6)) + 3;

        double idy_by_2 = 0.50 * (1.0 / hy);
        double idy_by_12 = (1.0 / hy)/12.0;

        if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

        int pp = IDX(i, j, k);

        if (dev_var_in[betay + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[offset + pp - nx]
            - 10.0 * dev_var_in[offset + pp]
            + 18.0 * dev_var_in[offset + pp + nx]
            -  6.0 * dev_var_in[offset + pp + 2*nx]
            +        dev_var_in[offset + pp + 3*nx]
            ) * idy_by_12;
        }
        else {
            output[pp] = ( -        dev_var_in[offset + pp - 3*nx]
            +  6.0 * dev_var_in[offset + pp - 2*nx]
            - 18.0 * dev_var_in[offset + pp - nx]
            + 10.0 * dev_var_in[offset + pp]
            +  3.0 * dev_var_in[offset + pp +nx]
            ) * idy_by_12;

        }

        if ((bflag & (1u<<OCT_DIR_DOWN)) && (j == 3)) {

            output[IDX(i,3,k)] = ( -  3.0 * dev_var_in[offset + IDX(i,3,k)]
            +  4.0 * dev_var_in[offset + IDX(i,4,k)]
            -        dev_var_in[offset + IDX(i,5,k)]
            ) * idy_by_2;

            if (dev_var_in[betay + IDX(i,4,k)] > 0.0) {
                output[IDX(i,4,k)] = ( -  3.0 * dev_var_in[offset + IDX(i,4,k)]
                    +  4.0 * dev_var_in[offset + IDX(i,5,k)]
                    -        dev_var_in[offset + IDX(i,6,k)]
                ) * idy_by_2;

            }
            else {
                output[IDX(i,4,k)] = ( -         dev_var_in[offset + IDX(i,3,k)]
                    +        dev_var_in[offset + IDX(i,5,k)]
                ) * idy_by_2;

            }

            if (dev_var_in[betay + IDX(i,5,k)] > 0.0 ) {
                output[IDX(i,5,k)] = (-  3.0 * dev_var_in[offset + IDX(i,4,k)]
                    - 10.0 * dev_var_in[offset + IDX(i,5,k)]
                    + 18.0 * dev_var_in[offset + IDX(i,6,k)]
                    -  6.0 * dev_var_in[offset + IDX(i,7,k)]
                    +        dev_var_in[offset + IDX(i,8,k)]
                ) * idy_by_12;
            }
            else {
                output[IDX(i,5,k)] = (           dev_var_in[offset + IDX(i,3,k)]
                            -  4.0 * dev_var_in[offset + IDX(i,4,k)]
                            +  3.0 * dev_var_in[offset + IDX(i,5,k)]
                ) * idy_by_2;
            }
        }

        if ((bflag & (1u<<OCT_DIR_UP)) && (j == 4)) {

            const int je = ny - 3;

            if ( dev_var_in[betay + IDX(i,je-3,k)] < 0.0 ) {
            output[IDX(i,je-3,k)] = (  - 3.0 * dev_var_in[offset + IDX(i,je-3,k)]
                    + 4.0 * dev_var_in[offset + IDX(i,je-2,k)]
                    -       dev_var_in[offset + IDX(i,je-1,k)]
                ) * idy_by_2;
            }
            else {
            output[IDX(i,je-3,k)] = ( -   dev_var_in[offset + IDX(i,je-6,k)]
                    +  6.0 * dev_var_in[offset + IDX(i,je-5,k)]
                    - 18.0 * dev_var_in[offset + IDX(i,je-4,k)]
                    + 10.0 * dev_var_in[offset + IDX(i,je-3,k)]
                    +  3.0 * dev_var_in[offset + IDX(i,je-2,k)]
                ) * idy_by_12;
            }

            if (dev_var_in[betay + IDX(i,je-2,k)] > 0.0 ) {
            output[IDX(i,je-2,k)] = (  -  dev_var_in[offset + IDX(i,je-3,k)]
                    +  dev_var_in[offset + IDX(i,je-1,k)]
                ) * idy_by_2;
            }
            else {
            output[IDX(i,je-2,k)] = (     dev_var_in[offset + IDX(i,je-4,k)]
                        - 4.0 * dev_var_in[offset + IDX(i,je-3,k)]
                        + 3.0 * dev_var_in[offset + IDX(i,je-2,k)]
                ) * idy_by_2;
            }

            output[IDX(i,je-1,k)]  = (          dev_var_in[offset + IDX(i,je-3,k)]
                        - 4.0 * dev_var_in[offset + IDX(i,je-2,k)]
                        + 3.0 * dev_var_in[offset + IDX(i,je-1,k)]
            ) * idy_by_2;
        }
    }
}

 
__device__ void device_calc_adv_z(double * output, double * dev_var_in,
    const int offset, double hz, int bflag,
    int nx,int ny,int nz, int betaz, int sz_x, int sz_y, int sz_z){

    int thread_id = blockIdx.x*threads_per_block + threadIdx.x;

    for (int id = thread_id*thread_load_adv_deriv; id<(thread_id+1)*thread_load_adv_deriv; id++){
        
        int i = id%(sz_x-6) + 3;
        int j = ((id/(sz_x-6))%(sz_y-6)) + 3;
        int k = (id/(sz_y-6)/(sz_x-6)) + 3;
        

        double idz_by_2 = 0.50 * (1.0 / hz);
        double idz_by_12 = (1.0 / hz)/12.0;

        if(i >= nx-3 || j >= ny-3 || k >= nz-3) return;

        int n = nx * ny;
        int pp = IDX(i, j, k);

        if (dev_var_in[betaz + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[offset + pp - n]
            - 10.0 * dev_var_in[offset + pp]
            + 18.0 * dev_var_in[offset + pp + n]
            -  6.0 * dev_var_in[offset + pp + 2*n]
            +        dev_var_in[offset + pp + 3*n]
            ) * idz_by_12;
        }
        else {
            output[pp] = ( -        dev_var_in[offset + pp - 3*n]
            +  6.0 * dev_var_in[offset + pp - 2*n]
            - 18.0 * dev_var_in[offset + pp - n]
            + 10.0 * dev_var_in[offset + pp]
            +  3.0 * dev_var_in[offset + pp +n]
            ) * idz_by_12;

        }

        if ((bflag & (1u<<OCT_DIR_BACK)) && (k == 3)) {

            output[IDX(i,j,3)] = ( -  3.0 * dev_var_in[offset + IDX(i,j,3)]
            +  4.0 * dev_var_in[offset + IDX(i,j,4)]
            -        dev_var_in[offset + IDX(i,j,5)]
            ) * idz_by_2;

            if (dev_var_in[betaz + IDX(i,j,4)] > 0.0) {
            output[IDX(i,j,4)] = ( -  3.0 * dev_var_in[offset + IDX(i,j,4)]
                +  4.0 * dev_var_in[offset + IDX(i,j,5)]
                -        dev_var_in[offset + IDX(i,j,6)]
            ) * idz_by_2;

            }
            else {
            output[IDX(i,j,4)] = ( -         dev_var_in[offset + IDX(i,j,3)]
                +        dev_var_in[offset + IDX(i,j,5)]
            ) * idz_by_2;

            }

            if (dev_var_in[betaz + IDX(i,j,5)] > 0.0 ) {
            output[IDX(i,j,5)] = (-  3.0 * dev_var_in[offset + IDX(i,j,4)]
                - 10.0 * dev_var_in[offset + IDX(i,j,5)]
                + 18.0 * dev_var_in[offset + IDX(i,j,6)]
                -  6.0 * dev_var_in[offset + IDX(i,j,7)]
                +        dev_var_in[offset + IDX(i,j,8)]
            ) * idz_by_12;
            }
            else {
            output[IDX(i,j,5)] = (           dev_var_in[offset + IDX(i,j,3)]
                        -  4.0 * dev_var_in[offset + IDX(i,j,4)]
                        +  3.0 * dev_var_in[offset + IDX(i,j,5)]
            ) * idz_by_2;
            }
        }

        if ((bflag & (1u<<OCT_DIR_FRONT)) && (k == 4)) {

            const int ke = nz - 3;

            if ( dev_var_in[betaz + IDX(i,j,ke-3)] < 0.0 ) {
            output[IDX(i,j,ke-3)] = (  - 3.0 * dev_var_in[offset + IDX(i,j,ke-3)]
                    + 4.0 * dev_var_in[offset + IDX(i,j,ke-2)]
                    -       dev_var_in[offset + IDX(i,j,ke-1)]
                ) * idz_by_2;
            }
            else {
            output[IDX(i,j,ke-3)] = ( -   dev_var_in[offset + IDX(i,j,ke-6)]
                    +  6.0 * dev_var_in[offset + IDX(i,j,ke-5)]
                    - 18.0 * dev_var_in[offset + IDX(i,j,ke-4)]
                    + 10.0 * dev_var_in[offset + IDX(i,j,ke-3)]
                    +  3.0 * dev_var_in[offset + IDX(i,j,ke-2)]
                ) * idz_by_12;
            }

            if (dev_var_in[betaz + IDX(i,j,ke-2)] > 0.0 ) {
            output[IDX(i,j,ke-2)] = (  -  dev_var_in[offset + IDX(i,j,ke-3)]
                    +  dev_var_in[offset + IDX(i,j,ke-1)]
                ) * idz_by_2;
            }
            else {
            output[IDX(i,j,ke-2)] = (     dev_var_in[offset + IDX(i,j,ke-4)]
                        - 4.0 * dev_var_in[offset + IDX(i,j,ke-3)]
                        + 3.0 * dev_var_in[offset + IDX(i,j,ke-2)]
                ) * idz_by_2;
            }

            output[IDX(i,j,ke-1)]  = (          dev_var_in[offset + IDX(i,j,ke-3)]
                        - 4.0 * dev_var_in[offset + IDX(i,j,ke-2)]
                        + 3.0 * dev_var_in[offset + IDX(i,j,ke-1)]
            ) * idz_by_2;
        }
    }
}
 

 
 __global__ void calc_all_adv(double * dev_var_in, double hx, double hy, double hz, 
    int sz_x, int sz_y, int sz_z, int bflag,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
 ) {
    int nx = sz_x;
    int ny = sz_y;
    int nz = sz_z;
 
     #include "bssnrhs_cuda_derivs_adv.h"
     //ib, jb, kb values are accumulated to the x, y, z
 
 }

 void cuda_deriv_calc_all_adv(double * dev_var_in, double hx, double hy, double hz, int sz_x, 
    int sz_y, int sz_z, int bflag, hipStream_t stream,
    #include "list_of_para.h"
    ,
    #include "list_of_offset_para.h"
    ){
     const int ie = sz_x - 3;//x direction
     const int je = sz_y - 3;//y direction
     const int ke = sz_z - 3;//z direction
     
     int total_points = ceil(1.0*ie*je*ke/thread_load_adv_deriv);
     int blocks = ceil(1.0*total_points/threads_per_block);

     calc_all_adv <<< blocks, threads_per_block, 0, stream >>> (
                    dev_var_in, hx, hy, hz, sz_x, sz_y, sz_z, bflag,
                  #include "list_of_args.h"
                  ,
                  #include "list_of_offset_args.h"
           );
}
