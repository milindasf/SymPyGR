#include "hip/hip_runtime.h"
#include "bssneqn_solve.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace std;

__constant__ double ETA_CONST=0.1;
__constant__ double ETA_R0=0.1;
__constant__ double ETA_DAMPING_EXP=0.1;
__constant__ unsigned int lambda[4]={1,2,3,4};
__constant__ double lambda_f[2]={0.8,0.9};

__global__ void cuda_bssn_eqns_points(double * dev_var_in, double * dev_var_out, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z,  
    double pmin_x, double pmin_y, double pmin_z, 
    double hz, double hy, double hx, 
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
    )
{
    int thread_id = blockIdx.x*256 + threadIdx.x; 

    int i = thread_id%(host_sz_x-6) + 3;
    int j = ((thread_id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (thread_id/(host_sz_z-6)/(host_sz_y-6)) + 3;

    if (k>=host_sz_z-3) return;

    double z = pmin_z + hz*k;
    double y = pmin_y + hy*j;
    double x = pmin_x + hx*i;

    int pp = i + (host_sz_x)*(j + (host_sz_y)*k);
    double r_coord = sqrt(x*x + y*y + z*z);
    double eta = ETA_CONST;
    if (r_coord >= ETA_R0) {
        eta *= pow( (ETA_R0/r_coord), ETA_DAMPING_EXP);
    }

    #include "cuda_bssneqs.cuh"
}

void calc_bssn_eqns(double * dev_var_in, double * dev_var_out, const unsigned int * sz, const double * pmin, double hz, double hy, double hx, hipStream_t stream,
#include "list_of_offset_para.h"
, 
#include "list_of_para.h"
)
{
    double pmin_x = pmin[0];
    double pmin_y = pmin[1];
    double pmin_z = pmin[2];

    const unsigned int host_sz_x = sz[0];
    const unsigned int host_sz_y = sz[1];
    const unsigned int host_sz_z = sz[2];

    int total_points = ceil(1.0*(sz[2]-6)*(sz[1]-6)*(sz[0]-6));

    int number_of_blocks = ceil(1.0*total_points/256);

    cuda_bssn_eqns_points<<< number_of_blocks, 256, 0, stream >>>(dev_var_in, dev_var_out, 
        host_sz_x, host_sz_y, host_sz_z, 
        pmin_x, pmin_y, pmin_z, 
        hz, hy, hx, 
        #include "list_of_offset_args.h"
        ,
        #include "list_of_args.h"
    ); 
}
