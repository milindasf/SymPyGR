#include "hip/hip_runtime.h"
#include "bssneqn_solve.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace std;

int threads_per_block_cpu=250;
int blocks_cpu=64;

__constant__ int threads_per_block=250;
__constant__ int blocks=64;

__constant__ double ETA_CONST=0.1;
__constant__ double ETA_R0=0.1;
__constant__ double ETA_DAMPING_EXP=0.1;
__constant__ unsigned int lambda[4]={1,2,3,4};
__constant__ double lambda_f[2]={0.8,0.9};

__global__ void cuda_bssn_eqns_points(double * dev_var_in, double * dev_var_out, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z,  
    double pmin_x, double pmin_y, double pmin_z, 
    double hz, double hy, double hx, 
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
    )
{
    int id = blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_y-6)) + 3;

    if (k>=host_sz_z-3) return;

    double z = pmin_z + hz*k;
    double y = pmin_y + hy*j;
    double x = pmin_x + hx*i;

    int pp = i + (host_sz_x)*(j + (host_sz_y)*k);
    double r_coord = sqrt(x*x + y*y + z*z);
    double eta = ETA_CONST;
    if (r_coord >= ETA_R0) {
        eta *= pow( (ETA_R0/r_coord), ETA_DAMPING_EXP);
    }

    #include "cuda_bssneqs.h"
}

void calc_bssn_eqns(double * dev_var_in, double * dev_var_out, const unsigned int * sz, const double * pmin, double hz, double hy, double hx, hipStream_t stream,
#include "list_of_offset_para.h"
, 
#include "list_of_para.h"
)
{
    double pmin_x = pmin[0];
    double pmin_y = pmin[1];
    double pmin_z = pmin[2];

    const unsigned int host_sz_x = sz[0];
    const unsigned int host_sz_y = sz[1];
    const unsigned int host_sz_z = sz[2];

    int total_points = (sz[2]-6)*(sz[1]-6)*(sz[0]-6);

    int number_of_blocks = ceil(1.0*total_points/threads_per_block_cpu);

    cuda_bssn_eqns_points<<< number_of_blocks, threads_per_block_cpu, 0, stream >>>(dev_var_in, dev_var_out, 
        host_sz_x, host_sz_y, host_sz_z, 
        pmin_x, pmin_y, pmin_z, 
        hz, hy, hx, 
        #include "list_of_offset_args.h"
        ,
        #include "list_of_args.h"
    ); 

    // int points_at_once = threads_per_block_cpu*blocks_cpu;
    // int loops = ceil(1.0*total_points/points_at_once);

    // for(int i=0; i<loops; i++){
    //     int offset = i*points_at_once;

        // cuda_bssn_eqns_points<<< blocks_cpu, threads_per_block_cpu, 0, stream >>>(dev_var_in, dev_var_out, 
        //     offset, host_sz_x, host_sz_y, host_sz_z, 
        //     pmin_x, pmin_y, pmin_z, 
        //     hz, hy, hx, 
        //     #include "list_of_offset_args.h"
        //     ,
        //     #include "list_of_args.h"
        // );     
    // }
}
