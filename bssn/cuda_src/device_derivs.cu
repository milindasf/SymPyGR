#include "hip/hip_runtime.h"
/**
 * Created on: March 15, 2018
 * 		Author: Akila
 **/

 #include "derivs_cuda.h"
 
 __device__ void calc_deriv42_x(int id, double * output, double * dev_var_in, const int u_offset, double dx, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){

    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idx = 1.0/dx;
    const double idx_by_12 = idx / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-2)) + 1;
    int k = (id/(host_sz_z-2)/(host_sz_x-6)) + 1; 
    if (k>=host_sz_z-1) return;
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[(u_offset) + pp - 2] - 8.0*dev_var_in[(u_offset)
                    + pp - 1] + 8.0*dev_var_in[(u_offset) + pp + 1] 
                    - dev_var_in[(u_offset) + pp + 2] )*idx_by_12;
 }

 __device__ void calc_deriv42_y(int id, double* output, double * dev_var_in, const int u_offset, double dy, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){

    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idy = 1.0/dy;
    const double idy_by_12 = idy / 12.0;
    
    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 1;
    if (k>=host_sz_z-1) return;
    int pp = IDX(i, j, k);
    
    output[pp] = (dev_var_in[u_offset + pp - 2*nx] 
                - 8.0*dev_var_in[u_offset + pp - nx] 
                + 8.0*dev_var_in[u_offset + pp + nx] 
                - dev_var_in[u_offset + pp + 2*nx] )*idy_by_12;
    
}

__device__ void calc_deriv42_z(int id, double* output, double * dev_var_in, const int u_offset, double dz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 
    int n = nx * ny;

    const double idz = 1.0/dz;
    const double idz_by_12 = idz / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = (dev_var_in[(u_offset) + pp - 2*n] - 8.0*dev_var_in[(u_offset) + pp - n] 
                    + 8.0*dev_var_in[(u_offset) + pp + n] - dev_var_in[(u_offset) + pp + 2*n]) 
                    * idz_by_12;
}

__device__ void calc_deriv42_xx(int id, double* output, double * dev_var_in, const int u_offset, double dx, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idx_sqrd = 1.0/(dx*dx);
    const double idx_sqrd_by_12 = idx_sqrd / 12.0;
    
    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2] 
                + 16.0*dev_var_in[(u_offset) + pp - 1] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + 1] 
                - dev_var_in[(u_offset) + pp + 2] 
            )*idx_sqrd_by_12;

}

__device__ void calc_deriv42_yy(int id, double* output, double * dev_var_in, const int u_offset, double dy, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idy_sqrd = 1.0/(dy*dy);
    const double idy_sqrd_by_12 = idy_sqrd / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2*nx] 
                + 16.0*dev_var_in[(u_offset) + pp - nx] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + nx] 
                - dev_var_in[(u_offset) + pp + 2*nx] 
            )*idy_sqrd_by_12;
            
}

__device__ void calc_deriv42_zz(int id, double* output, double * dev_var_in, const int u_offset, double dz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 
    int n = nx * ny;

    const double idz_sqrd = 1.0/(dz*dz);
    const double idz_sqrd_by_12 = idz_sqrd / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2*n] 
                + 16.0*dev_var_in[(u_offset) + pp - n] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + n] 
                - dev_var_in[(u_offset) + pp + 2*n] 
            )*idz_sqrd_by_12;
}

__device__ void calc_deriv42_adv_x(int id, double * output, double * dev_var_in, int u_offset, double dx, int betax,
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) {
    int nx = host_sz_x;
    int ny = host_sz_y;

    const double idx = 1.0/dx;
    // const double idx_by_2 = 0.50 * idx;
    const double idx_by_12 = idx / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betax + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - 1]
                    - 10.0 * dev_var_in[u_offset + pp]
                    + 18.0 * dev_var_in[u_offset + pp + 1]
                    -  6.0 * dev_var_in[u_offset + pp + 2]
                    +        dev_var_in[u_offset + pp + 3]
                ) * idx_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3]
                    +  6.0 * dev_var_in[u_offset + pp - 2]
                    - 18.0 * dev_var_in[u_offset + pp - 1]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +1]
                ) * idx_by_12;
    }
}

__device__ void calc_deriv42_adv_y(int id, double * output, double * dev_var_in, int u_offset, double dy, int betay,
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) {
    
    int nx = host_sz_x;
    int ny = host_sz_y;

    const double idy = 1.0/dy;
    // const double idy_by_2 = 0.50 * idy;
    const double idy_by_12 = idy / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betay + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - nx]
                        - 10.0 * dev_var_in[u_offset + pp]
                        + 18.0 * dev_var_in[u_offset + pp + nx]
                        -  6.0 * dev_var_in[u_offset + pp + 2*nx]
                        +        dev_var_in[u_offset + pp + 3*nx]
                    ) * idy_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3*nx]
                    +  6.0 * dev_var_in[u_offset + pp - 2*nx]
                    - 18.0 * dev_var_in[u_offset + pp - nx]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +nx]
                    ) * idy_by_12;
                
    }
}

__device__ void calc_deriv42_adv_z(int id, double * output, double * dev_var_in, int u_offset, double dz, int betaz, 
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) {

    int nx = host_sz_x;
    int ny = host_sz_y;
    int n = nx * ny;
    
    const double idz = 1.0/dz;
    // const double idz_by_2 = 0.50 * idz;
    const double idz_by_12 = idz / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betaz + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - n]
                        - 10.0 * dev_var_in[u_offset + pp]
                        + 18.0 * dev_var_in[u_offset + pp + n]
                        -  6.0 * dev_var_in[u_offset + pp + 2*n]
                        +        dev_var_in[u_offset + pp + 3*n]
                    ) * idz_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3*n]
                    +  6.0 * dev_var_in[u_offset + pp - 2*n]
                    - 18.0 * dev_var_in[u_offset + pp - n]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +n]
                    ) * idz_by_12;
                
    }
}

__global__ void calc_derivs(
    double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, 
    int bflag,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

calc_deriv42_x(tid, grad_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_B0, dev_var_in, B0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_B0, dev_var_in, B0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_B0, dev_var_in, B0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_B1, dev_var_in, B1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_B1, dev_var_in, B1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_B1, dev_var_in, B1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_B2, dev_var_in, B2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_B2, dev_var_in, B2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_B2, dev_var_in, B2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_Gt0, dev_var_in, Gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_Gt0, dev_var_in, Gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_Gt0, dev_var_in, Gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_Gt1, dev_var_in, Gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_Gt1, dev_var_in, Gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_Gt1, dev_var_in, Gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_Gt2, dev_var_in, Gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_Gt2, dev_var_in, Gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_Gt2, dev_var_in, Gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_K, dev_var_in, KInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_K, dev_var_in, KInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_K, dev_var_in, KInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_xx(tid, grad2_0_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_yy(tid, grad2_1_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_zz(tid, grad2_2_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_At0, dev_var_in, At0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_At0, dev_var_in, At0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_At0, dev_var_in, At0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_At1, dev_var_in, At1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_At1, dev_var_in, At1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_At1, dev_var_in, At1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_At2, dev_var_in, At2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_At2, dev_var_in, At2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_At2, dev_var_in, At2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_At3, dev_var_in, At3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_At3, dev_var_in, At3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_At3, dev_var_in, At3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_At4, dev_var_in, At4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_At4, dev_var_in, At4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_At4, dev_var_in, At4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_x(tid, grad_0_At5, dev_var_in, At5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad_1_At5, dev_var_in, At5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad_2_At5, dev_var_in, At5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_y(tid, grad2_0_1_gt0, grad_0_gt0, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_gt0, grad_0_gt0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_gt0, grad_1_gt0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad2_0_1_gt1, grad_0_gt1, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_gt1, grad_0_gt1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_gt1, grad_1_gt1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad2_0_1_gt2, grad_0_gt2, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_gt2, grad_0_gt2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_gt2, grad_1_gt2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad2_0_1_gt3, grad_0_gt3, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_gt3, grad_0_gt3, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_gt3, grad_1_gt3, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad2_0_1_gt4, grad_0_gt4, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_gt4, grad_0_gt4, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_gt4, grad_1_gt4, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad2_0_1_gt5, grad_0_gt5, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_gt5, grad_0_gt5, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_gt5, grad_1_gt5, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad2_0_1_chi, grad_0_chi, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_chi, grad_0_chi, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_chi, grad_1_chi, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad2_0_1_alpha, grad_0_alpha, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_alpha, grad_0_alpha, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_alpha, grad_1_alpha, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad2_0_1_beta0, grad_0_beta0, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_beta0, grad_0_beta0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_beta0, grad_1_beta0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad2_0_1_beta1, grad_0_beta1, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_beta1, grad_0_beta1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_beta1, grad_1_beta1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_y(tid, grad2_0_1_beta2, grad_0_beta2, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_0_2_beta2, grad_0_beta2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_z(tid, grad2_1_2_beta2, grad_1_beta2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_gt0, dev_var_in, gt0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_gt0, dev_var_in, gt0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_gt1, dev_var_in, gt1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_gt1, dev_var_in, gt1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_gt2, dev_var_in, gt2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_gt2, dev_var_in, gt2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_gt3, dev_var_in, gt3Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_gt3, dev_var_in, gt3Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_gt4, dev_var_in, gt4Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_gt4, dev_var_in, gt4Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_gt5, dev_var_in, gt5Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_gt5, dev_var_in, gt5Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);


calc_deriv42_adv_x(tid, agrad_0_At0, dev_var_in, At0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_At0, dev_var_in, At0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_At1, dev_var_in, At1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_At1, dev_var_in, At1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_At2, dev_var_in, At2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_At2, dev_var_in, At2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_At3, dev_var_in, At3Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_At3, dev_var_in, At3Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_At4, dev_var_in, At4Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_At4, dev_var_in, At4Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_At5, dev_var_in, At5Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_At5, dev_var_in, At5Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_x(tid, agrad_0_alpha, dev_var_in, alphaInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_alpha, dev_var_in, alphaInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_beta0, dev_var_in, beta0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_beta1, dev_var_in, beta1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_beta2, dev_var_in, beta2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_chi, dev_var_in, chiInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_Gt0, dev_var_in, Gt0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_Gt1, dev_var_in, Gt1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_Gt2, dev_var_in, Gt2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_K, dev_var_in, KInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_B0, dev_var_in, B0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_B1, dev_var_in, B1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_x(tid, agrad_0_B2, dev_var_in, B2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);

calc_deriv42_adv_y(tid, agrad_1_beta0, dev_var_in, beta0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_beta1, dev_var_in, beta1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_beta2, dev_var_in, beta2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_chi, dev_var_in, chiInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_Gt0, dev_var_in, Gt0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_Gt1, dev_var_in, Gt1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_Gt2, dev_var_in, Gt2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_K, dev_var_in, KInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_B0, dev_var_in, B0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_B1, dev_var_in, B1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_y(tid, agrad_1_B2, dev_var_in, B2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_gt0, dev_var_in, gt0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_gt1, dev_var_in, gt1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_gt2, dev_var_in, gt2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_gt3, dev_var_in, gt3Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_gt4, dev_var_in, gt4Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_gt5, dev_var_in, gt5Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_At0, dev_var_in, At0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_At1, dev_var_in, At1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_At2, dev_var_in, At2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_At3, dev_var_in, At3Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_At4, dev_var_in, At4Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_At5, dev_var_in, At5Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_alpha, dev_var_in, alphaInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_beta0, dev_var_in, beta0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_beta1, dev_var_in, beta1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_beta2, dev_var_in, beta2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_chi, dev_var_in, chiInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_Gt0, dev_var_in, Gt0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_Gt1, dev_var_in, Gt1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_Gt2, dev_var_in, Gt2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_K, dev_var_in, KInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_B0, dev_var_in, B0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_B1, dev_var_in, B1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
calc_deriv42_adv_z(tid, agrad_2_B2, dev_var_in, B2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);


}


void calc_deriv_wrapper(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int * host_sz, int bflag, hipStream_t stream,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    const int ib = 1;
    const int jb = 1;
    const int kb = 1;
    const int ie = host_sz[0] - 1;
    const int je = host_sz[1] - 1;
    const int ke = host_sz[2] - 1;
    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];
 
    int number_of_threads_required;
    int number_of_blocks;

    number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
    number_of_blocks=ceil(1.0*number_of_threads_required/64);
    calc_derivs <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
        #include "list_of_offset_args.h"
        ,
        #include "list_of_args.h"
    );
    CHECK_ERROR(hipGetLastError(), "deriv Kernel launch failed");
}
