#include "hip/hip_runtime.h"
/**
 * Created on: March 15, 2018
 * 		Author: Akila
 **/

 #include "derivs_cuda.cuh"
 
 __device__ void calc_deriv42_x(int id, double * output, double * dev_var_in, const int u_offset, double dx, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){

    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idx = 1.0/dx;
    const double idx_by_12 = idx / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-2)) + 1;
    int k = (id/(host_sz_z-2)/(host_sz_x-6)) + 1; 
    if (k>=host_sz_z-1) return;
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[(u_offset) + pp - 2] - 8.0*dev_var_in[(u_offset)
                    + pp - 1] + 8.0*dev_var_in[(u_offset) + pp + 1] 
                    - dev_var_in[(u_offset) + pp + 2] )*idx_by_12;
 }

 __device__ void calc_deriv42_y(int id, double* output, double * dev_var_in, const int u_offset, double dy, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){

    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idy = 1.0/dy;
    const double idy_by_12 = idy / 12.0;
    
    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 1;
    if (k>=host_sz_z-1) return;
    int pp = IDX(i, j, k);
    
    output[pp] = (dev_var_in[u_offset + pp - 2*nx] 
                - 8.0*dev_var_in[u_offset + pp - nx] 
                + 8.0*dev_var_in[u_offset + pp + nx] 
                - dev_var_in[u_offset + pp + 2*nx] )*idy_by_12;
    
}

__device__ void calc_deriv42_z(int id, double* output, double * dev_var_in, const int u_offset, double dz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 
    int n = nx * ny;

    const double idz = 1.0/dz;
    const double idz_by_12 = idz / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = (dev_var_in[(u_offset) + pp - 2*n] - 8.0*dev_var_in[(u_offset) + pp - n] 
                    + 8.0*dev_var_in[(u_offset) + pp + n] - dev_var_in[(u_offset) + pp + 2*n]) 
                    * idz_by_12;
}

__device__ void calc_deriv42_xx(int id, double* output, double * dev_var_in, const int u_offset, double dx, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idx_sqrd = 1.0/(dx*dx);
    const double idx_sqrd_by_12 = idx_sqrd / 12.0;
    
    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2] 
                + 16.0*dev_var_in[(u_offset) + pp - 1] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + 1] 
                - dev_var_in[(u_offset) + pp + 2] 
            )*idx_sqrd_by_12;

}

__device__ void calc_deriv42_yy(int id, double* output, double * dev_var_in, const int u_offset, double dy, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idy_sqrd = 1.0/(dy*dy);
    const double idy_sqrd_by_12 = idy_sqrd / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2*nx] 
                + 16.0*dev_var_in[(u_offset) + pp - nx] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + nx] 
                - dev_var_in[(u_offset) + pp + 2*nx] 
            )*idy_sqrd_by_12;
            
}

__device__ void calc_deriv42_zz(int id, double* output, double * dev_var_in, const int u_offset, double dz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 
    int n = nx * ny;

    const double idz_sqrd = 1.0/(dz*dz);
    const double idz_sqrd_by_12 = idz_sqrd / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2*n] 
                + 16.0*dev_var_in[(u_offset) + pp - n] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + n] 
                - dev_var_in[(u_offset) + pp + 2*n] 
            )*idz_sqrd_by_12;
}

__device__ void calc_deriv42_adv_x(int id, double * output, double * dev_var_in, int u_offset, double dx, int betax,
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) {
    int nx = host_sz_x;
    int ny = host_sz_y;

    const double idx = 1.0/dx;
    // const double idx_by_2 = 0.50 * idx;
    const double idx_by_12 = idx / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betax + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - 1]
                    - 10.0 * dev_var_in[u_offset + pp]
                    + 18.0 * dev_var_in[u_offset + pp + 1]
                    -  6.0 * dev_var_in[u_offset + pp + 2]
                    +        dev_var_in[u_offset + pp + 3]
                ) * idx_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3]
                    +  6.0 * dev_var_in[u_offset + pp - 2]
                    - 18.0 * dev_var_in[u_offset + pp - 1]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +1]
                ) * idx_by_12;
    }
}

__device__ void calc_deriv42_adv_y(int id, double * output, double * dev_var_in, int u_offset, double dy, int betay,
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) {
    
    int nx = host_sz_x;
    int ny = host_sz_y;

    const double idy = 1.0/dy;
    // const double idy_by_2 = 0.50 * idy;
    const double idy_by_12 = idy / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betay + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - nx]
                        - 10.0 * dev_var_in[u_offset + pp]
                        + 18.0 * dev_var_in[u_offset + pp + nx]
                        -  6.0 * dev_var_in[u_offset + pp + 2*nx]
                        +        dev_var_in[u_offset + pp + 3*nx]
                    ) * idy_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3*nx]
                    +  6.0 * dev_var_in[u_offset + pp - 2*nx]
                    - 18.0 * dev_var_in[u_offset + pp - nx]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +nx]
                    ) * idy_by_12;
                
    }
}

__device__ void calc_deriv42_adv_z(int id, double * output, double * dev_var_in, int u_offset, double dz, int betaz, 
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) {

    int nx = host_sz_x;
    int ny = host_sz_y;
    int n = nx * ny;
    
    const double idz = 1.0/dz;
    // const double idz_by_2 = 0.50 * idz;
    const double idz_by_12 = idz / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betaz + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - n]
                        - 10.0 * dev_var_in[u_offset + pp]
                        + 18.0 * dev_var_in[u_offset + pp + n]
                        -  6.0 * dev_var_in[u_offset + pp + 2*n]
                        +        dev_var_in[u_offset + pp + 3*n]
                    ) * idz_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3*n]
                    +  6.0 * dev_var_in[u_offset + pp - 2*n]
                    - 18.0 * dev_var_in[u_offset + pp - n]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +n]
                    ) * idz_by_12;
                
    }
}

__global__ void calc_derivs1(
    double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, 
    int bflag,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    calc_deriv42_x(tid, grad_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_B0, dev_var_in, B0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_B0, dev_var_in, B0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_B0, dev_var_in, B0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_B1, dev_var_in, B1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_B1, dev_var_in, B1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_B1, dev_var_in, B1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_B2, dev_var_in, B2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_B2, dev_var_in, B2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_B2, dev_var_in, B2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_Gt0, dev_var_in, Gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_Gt0, dev_var_in, Gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_Gt0, dev_var_in, Gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_Gt1, dev_var_in, Gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_Gt1, dev_var_in, Gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_Gt1, dev_var_in, Gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_Gt2, dev_var_in, Gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_Gt2, dev_var_in, Gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_Gt2, dev_var_in, Gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_K, dev_var_in, KInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_K, dev_var_in, KInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_K, dev_var_in, KInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At0, dev_var_in, At0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At0, dev_var_in, At0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At0, dev_var_in, At0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At1, dev_var_in, At1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At1, dev_var_in, At1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At1, dev_var_in, At1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At2, dev_var_in, At2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At2, dev_var_in, At2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At2, dev_var_in, At2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At3, dev_var_in, At3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At3, dev_var_in, At3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At3, dev_var_in, At3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At4, dev_var_in, At4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At4, dev_var_in, At4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At4, dev_var_in, At4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At5, dev_var_in, At5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At5, dev_var_in, At5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At5, dev_var_in, At5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
}

__global__ void calc_derivs2(
    double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, 
    int bflag,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    calc_deriv42_y(tid, grad2_0_1_gt0, grad_0_gt0, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt0, grad_0_gt0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt0, grad_1_gt0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_gt1, grad_0_gt1, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt1, grad_0_gt1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt1, grad_1_gt1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_gt2, grad_0_gt2, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt2, grad_0_gt2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt2, grad_1_gt2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_gt3, grad_0_gt3, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt3, grad_0_gt3, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt3, grad_1_gt3, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_gt4, grad_0_gt4, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt4, grad_0_gt4, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt4, grad_1_gt4, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_gt5, grad_0_gt5, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt5, grad_0_gt5, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt5, grad_1_gt5, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_chi, grad_0_chi, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_chi, grad_0_chi, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_chi, grad_1_chi, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_alpha, grad_0_alpha, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_alpha, grad_0_alpha, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_alpha, grad_1_alpha, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_beta0, grad_0_beta0, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_beta0, grad_0_beta0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_beta0, grad_1_beta0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_beta1, grad_0_beta1, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_beta1, grad_0_beta1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_beta1, grad_1_beta1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_beta2, grad_0_beta2, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_beta2, grad_0_beta2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_beta2, grad_1_beta2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt0, dev_var_in, gt0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt0, dev_var_in, gt0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt1, dev_var_in, gt1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt1, dev_var_in, gt1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt2, dev_var_in, gt2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt2, dev_var_in, gt2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt3, dev_var_in, gt3Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt3, dev_var_in, gt3Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt4, dev_var_in, gt4Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt4, dev_var_in, gt4Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt5, dev_var_in, gt5Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt5, dev_var_in, gt5Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);


    calc_deriv42_adv_x(tid, agrad_0_At0, dev_var_in, At0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At0, dev_var_in, At0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_At1, dev_var_in, At1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At1, dev_var_in, At1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_At2, dev_var_in, At2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At2, dev_var_in, At2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_At3, dev_var_in, At3Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At3, dev_var_in, At3Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_At4, dev_var_in, At4Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At4, dev_var_in, At4Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_At5, dev_var_in, At5Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At5, dev_var_in, At5Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_alpha, dev_var_in, alphaInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_alpha, dev_var_in, alphaInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_beta0, dev_var_in, beta0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_beta1, dev_var_in, beta1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_beta2, dev_var_in, beta2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_chi, dev_var_in, chiInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_Gt0, dev_var_in, Gt0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_Gt1, dev_var_in, Gt1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_Gt2, dev_var_in, Gt2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_K, dev_var_in, KInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_B0, dev_var_in, B0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_B1, dev_var_in, B1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_B2, dev_var_in, B2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_beta0, dev_var_in, beta0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_beta1, dev_var_in, beta1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_beta2, dev_var_in, beta2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_chi, dev_var_in, chiInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_Gt0, dev_var_in, Gt0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_Gt1, dev_var_in, Gt1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_Gt2, dev_var_in, Gt2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_K, dev_var_in, KInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_B0, dev_var_in, B0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_B1, dev_var_in, B1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_B2, dev_var_in, B2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt0, dev_var_in, gt0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt1, dev_var_in, gt1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt2, dev_var_in, gt2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt3, dev_var_in, gt3Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt4, dev_var_in, gt4Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt5, dev_var_in, gt5Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At0, dev_var_in, At0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At1, dev_var_in, At1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At2, dev_var_in, At2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At3, dev_var_in, At3Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At4, dev_var_in, At4Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At5, dev_var_in, At5Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_alpha, dev_var_in, alphaInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_beta0, dev_var_in, beta0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_beta1, dev_var_in, beta1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_beta2, dev_var_in, beta2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_chi, dev_var_in, chiInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_Gt0, dev_var_in, Gt0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_Gt1, dev_var_in, Gt1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_Gt2, dev_var_in, Gt2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_K, dev_var_in, KInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_B0, dev_var_in, B0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_B1, dev_var_in, B1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_B2, dev_var_in, B2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
}

void calc_deriv_wrapper(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int * host_sz, int bflag, hipStream_t stream,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    const int ib = 1;
    const int jb = 1;
    const int kb = 1;
    const int ie = host_sz[0] - 1;
    const int je = host_sz[1] - 1;
    const int ke = host_sz[2] - 1;
    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];
 
    int number_of_threads_required;
    int number_of_blocks;

    number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
    number_of_blocks=ceil(1.0*number_of_threads_required/64);
    calc_derivs1 <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
        #include "list_of_offset_args.h"
        ,
        #include "list_of_args.h"
    );

    number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
    number_of_blocks=ceil(1.0*number_of_threads_required/64);
    calc_derivs2 <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
        #include "list_of_offset_args.h"
        ,
        #include "list_of_args.h"
    );
    CHECK_ERROR(hipGetLastError(), "deriv Kernel launch failed");
}

__device__ void calc_ko_deriv42_x(int id, double * output, double * dev_var_in, int u_offset, double dx, 
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x;
    int ny = host_sz_y;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    
    if(i==4) {
        int ib=3;
        output[IDX(3, j, k)] = (-1.0 / 64.0 / dx) *
                        (
                        -      dev_var_in[u_offset + IDX(ib+4,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ib+3,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ib+2,j,k)]
                        + 20.0*dev_var_in[u_offset + IDX(ib+1,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ib,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ib-1,j,k)]
                        -      dev_var_in[u_offset + IDX(ib-2,j,k)]
                        );
    }

    output[pp] = (-1.0 / 64.0 / dx) *
                            (
                            -      dev_var_in[u_offset + pp - 3]
                            +  6.0*dev_var_in[u_offset + pp - 2]
                            - 15.0*dev_var_in[u_offset + pp - 1]
                            + 20.0*dev_var_in[u_offset + pp ]
                            - 15.0*dev_var_in[u_offset + pp + 1]
                            +  6.0*dev_var_in[u_offset + pp + 2]
                            -      dev_var_in[u_offset + pp + 3]
                            );

    if(i==5) {
        int ie = nx-3;
        output[IDX(ie-1, j, k)] = (-1.0 / 64.0 / dx) *
                        (
                        -      dev_var_in[u_offset + IDX(ie+1,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ie,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
                        + 20.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
                        -      dev_var_in[u_offset + IDX(ie-5,j,k)]
                        );
    }
}

__device__ void calc_ko_deriv42_y(int id, double * output, double * dev_var_in, int u_offset, double dy,
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    
    int nx = host_sz_x;
    int ny = host_sz_y;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if(j==4) {
        int jb=3;
        output[IDX(i,jb,k)] = (-1.0 / 64.0 / dy) *
                    (
                        -      dev_var_in[u_offset + IDX(i,jb+4,k)]
                        +  6.0*dev_var_in[u_offset + IDX(i,jb+3,k)]
                        - 15.0*dev_var_in[u_offset + IDX(i,jb+2,k)]
                        + 20.0*dev_var_in[u_offset + IDX(i,jb+1,k)]
                        - 15.0*dev_var_in[u_offset + IDX(i,jb,k)]
                        +  6.0*dev_var_in[u_offset + IDX(i,jb-1,k)]
                        -      dev_var_in[u_offset + IDX(i,jb-2,k)]
                        );
        }
    output[pp] = (-1.0 / 64.0 / dy) *
                    (
                        -      dev_var_in[u_offset + pp-3*nx]
                        +  6.0*dev_var_in[u_offset + pp-2*nx]
                        - 15.0*dev_var_in[u_offset + pp-nx]
                        + 20.0*dev_var_in[u_offset + pp]
                        - 15.0*dev_var_in[u_offset + pp+nx]
                        +  6.0*dev_var_in[u_offset + pp+2*nx]
                        -      dev_var_in[u_offset + pp+3*nx]
                        );

    if(j==5) {
        int je = ny - 3;
        output[IDX(i,je-1,k)] = (-1.0 / 64.0 / dy) *
                (
                    -      dev_var_in[u_offset + IDX(i,je+1,k)]
                    +  6.0*dev_var_in[u_offset + IDX(i,je,k)]
                    - 15.0*dev_var_in[u_offset + IDX(i,je-1,k)]
                    + 20.0*dev_var_in[u_offset + IDX(i,je-2,k)]
                    - 15.0*dev_var_in[u_offset + IDX(i,je-3,k)]
                    +  6.0*dev_var_in[u_offset + IDX(i,je-4,k)]
                    -      dev_var_in[u_offset + IDX(i,je-5,k)]
                    );                   
    }
}

__device__ void calc_ko_deriv42_z(int id, double * output, double * dev_var_in, int u_offset, double dz, 
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{  
    int nx = host_sz_x;
    int ny = host_sz_y;
    int n = nx * ny;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    
    if(k==4) {
        int kb=3;
        output[IDX(i,j,kb)] = (-1.0 / 64.0 / dz) *
                    (
                        -      dev_var_in[u_offset + IDX(i,j,kb+4)]
                        +  6.0*dev_var_in[u_offset + IDX(i,j,kb+3)]
                        - 15.0*dev_var_in[u_offset + IDX(i,j,kb+2)]
                        + 20.0*dev_var_in[u_offset + IDX(i,j,kb+1)]
                        - 15.0*dev_var_in[u_offset + IDX(i,j,kb)]
                        +  6.0*dev_var_in[u_offset + IDX(i,j,kb-1)]
                        -      dev_var_in[u_offset + IDX(i,j,kb-2)]
                        );
        }

    output[pp] = (-1.0 / 64.0 / dz) *
                (
                    -      dev_var_in[u_offset + pp-3*n]
                    +  6.0*dev_var_in[u_offset + pp-2*n]
                    - 15.0*dev_var_in[u_offset + pp-n]
                    + 20.0*dev_var_in[u_offset + pp]
                    - 15.0*dev_var_in[u_offset + pp+n]
                    +  6.0*dev_var_in[u_offset + pp+2*n]
                    -      dev_var_in[u_offset + pp+3*n]
                    );
    if(k==5) {
        int ke = host_sz_z - 3;
        output[IDX(i,j,ke-1)] = (-1.0 / 64.0 / dz) *
        (
            -      dev_var_in[u_offset + IDX(i,j,ke+1)]
            +  6.0*dev_var_in[u_offset + IDX(i,j,ke)]
            - 15.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
            + 20.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
            - 15.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
            +  6.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
            -      dev_var_in[u_offset + IDX(i,j,ke-5)]
            );               
    }
        
}


__global__ void calc_ko_derivs(
    double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, 
    int bflag,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    calc_ko_deriv42_x(tid, grad_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At0, dev_var_in, At0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At0, dev_var_in, At0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At0, dev_var_in, At0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At1, dev_var_in, At1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At1, dev_var_in, At1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At1, dev_var_in, At1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At2, dev_var_in, At2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At2, dev_var_in, At2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At2, dev_var_in, At2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At3, dev_var_in, At3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At3, dev_var_in, At3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At3, dev_var_in, At3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At4, dev_var_in, At4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At4, dev_var_in, At4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At4, dev_var_in, At4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At5, dev_var_in, At5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At5, dev_var_in, At5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At5, dev_var_in, At5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_Gt0, dev_var_in, Gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_Gt0, dev_var_in, Gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_Gt0, dev_var_in, Gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_Gt1, dev_var_in, Gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_Gt1, dev_var_in, Gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_Gt1, dev_var_in, Gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_Gt2, dev_var_in, Gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_Gt2, dev_var_in, Gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_Gt2, dev_var_in, Gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_K, dev_var_in, KInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_K, dev_var_in, KInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_K, dev_var_in, KInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_B0, dev_var_in, B0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_B0, dev_var_in, B0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_B0, dev_var_in, B0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_B1, dev_var_in, B1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_B1, dev_var_in, B1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_B1, dev_var_in, B1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_B2, dev_var_in, B2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_B2, dev_var_in, B2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_B2, dev_var_in, B2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);


}

void calc_ko_deriv_wrapper(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int * host_sz, int bflag, hipStream_t stream,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
    )
{
    const int ib = 1;
    const int jb = 1;
    const int kb = 1;
    const int ie = host_sz[0] - 1;
    const int je = host_sz[1] - 1;
    const int ke = host_sz[2] - 1;
    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];

    int number_of_threads_required;
    int number_of_blocks;

    number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
    number_of_blocks=ceil(1.0*number_of_threads_required/64);

    calc_ko_derivs <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
        #include "list_of_offset_args.h"
        ,
        #include "list_of_args.h"
    );
    CHECK_ERROR(hipGetLastError(), "ko deriv Kernel launch failed");
}

// device methods with bflag

__device__ void calc_deriv42_x_bflag(int id, double * output, double * dev_var_in, const int u_offset, double dx, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){

    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idx = 1.0/dx;
    const double idx_by_2 = 0.50 * idx;
    const double idx_by_12 = idx / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-2)) + 1;
    int k = (id/(host_sz_z-2)/(host_sz_x-6)) + 1; 
    if (k>=host_sz_z-1) return;
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[(u_offset) + pp - 2] - 8.0*dev_var_in[(u_offset)
                    + pp - 1] + 8.0*dev_var_in[(u_offset) + pp + 1] 
                    - dev_var_in[(u_offset) + pp + 2] )*idx_by_12;

    if ((bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
        int pp3 = IDX(3, j, k);
        int pp4 = IDX(4, j, k);
        int pp5 = IDX(5, j, k);
        output[pp3] = ((-3)*dev_var_in[(u_offset) + pp3] + 4*dev_var_in[(u_offset) + pp4] - dev_var_in[(u_offset) + pp5]) * idx_by_2;
        output[pp4] = (dev_var_in[(u_offset) + pp5] - dev_var_in[(u_offset) + pp3]) * idx_by_2;
    }

    if ((bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
        int pp2 = IDX(nx-5, j, k);
        int pp3 = IDX(nx-6, j, k);
        int pp1 = IDX(nx-4,j,k);
        output[pp2] = (dev_var_in[(u_offset) + pp1] - dev_var_in[(u_offset) + pp3]) * idx_by_2;
        output[pp1] = (dev_var_in[(u_offset) + pp3]- 4.0 * dev_var_in[(u_offset) + pp2]+ 3.0 * dev_var_in[(u_offset) + pp1]) * idx_by_2;
    }
 }

 __device__ void calc_deriv42_y_bflag(int id, double* output, double * dev_var_in, const int u_offset, double dy, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){

    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idy = 1.0/dy;
    const double idy_by_2 = 0.50 * idy;
    const double idy_by_12 = idy / 12.0;
    
    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 1;
    if (k>=host_sz_z-1) return;
    int pp = IDX(i, j, k);
    
    output[pp] = (dev_var_in[u_offset + pp - 2*nx] 
                - 8.0*dev_var_in[u_offset + pp - nx] 
                + 8.0*dev_var_in[u_offset + pp + nx] 
                - dev_var_in[u_offset + pp + 2*nx] )*idy_by_12;
    
    if ((bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
        int pp3 = IDX(i, 3, k);
        int pp4 = IDX(i, 4, k);
        int pp5 = IDX(i, 5, k);
        output[pp3] = ((-3)*dev_var_in[(u_offset) + pp3] +  4*dev_var_in[(u_offset) + pp4] - dev_var_in[(u_offset) + pp5]) * idy_by_2;
        output[pp4] = (dev_var_in[(u_offset) + pp5] - dev_var_in[(u_offset) + pp3]) * idy_by_2;
    }

    if ((bflag & (1u<<OCT_DIR_UP)) && j==4)  {
        int pp2 = IDX(i, ny-5, k); // IDX(i,je-2,k)
        int pp3 = IDX(i, ny-6, k); // IDX(i,je-3,k)
        int pp1 = IDX(i, ny-4, k); // IDX(i,je-1,k)
        output[pp2] = (dev_var_in[(u_offset) + pp1] - dev_var_in[(u_offset) + pp3]) * idy_by_2;
        output[pp1] = (dev_var_in[(u_offset) + pp3]- 4.0 * dev_var_in[(u_offset) + pp2] + 3.0 * dev_var_in[(u_offset) + pp1]) * idy_by_2;
    }
    
}

__device__ void calc_deriv42_z_bflag(int id, double* output, double * dev_var_in, const int u_offset, double dz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 
    int n = nx * ny;

    const double idz = 1.0/dz;
    const double idz_by_2 = 0.50 * idz;
    const double idz_by_12 = idz / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = (dev_var_in[(u_offset) + pp - 2*n] - 8.0*dev_var_in[(u_offset) + pp - n] 
                    + 8.0*dev_var_in[(u_offset) + pp + n] - dev_var_in[(u_offset) + pp + 2*n]) 
                    * idz_by_12;
            
    if ((bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
        int pp3 = IDX(i, j, 3); 
        int pp4 = IDX(i, j, 4); 
        int pp5 = IDX(i, j, 5);
        output[pp3] = ((-3)*dev_var_in[(u_offset) + pp3] + 4*dev_var_in[(u_offset) + pp4] - dev_var_in[(u_offset) + pp5]) * idz_by_2;
        output[pp4] = (dev_var_in[(u_offset) + pp5] - dev_var_in[(u_offset) + pp3]) * idz_by_2;
    }
                
    if ((bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
        int pp2 = IDX(i, j, host_sz_z-5); 
        int pp3 = IDX(i, j, host_sz_z-6); 
        int pp1 = IDX(i, j, host_sz_z-4);
        output[pp2] = (dev_var_in[(u_offset) + pp1] - dev_var_in[(u_offset) + pp3]) * idz_by_2;
        output[pp1] = (dev_var_in[(u_offset) + pp3]- 4.0 * dev_var_in[(u_offset) + pp2] + 3.0 * dev_var_in[(u_offset) + pp1]) * idz_by_2;
    }
}

__device__ void calc_deriv42_xx_bflag(int id, double* output, double * dev_var_in, const int u_offset, double dx, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idx_sqrd = 1.0/(dx*dx);
    const double idx_sqrd_by_12 = idx_sqrd / 12.0;
    
    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2] 
                + 16.0*dev_var_in[(u_offset) + pp - 1] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + 1] 
                - dev_var_in[(u_offset) + pp + 2] 
            )*idx_sqrd_by_12;

    if ((bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
        int pp3 = IDX(3, j, k); 
        int pp4 = IDX(4, j, k); 
        int pp5 = IDX(5, j, k); 
        int pp6 = IDX(6, j, k); 
    
        output[pp3] = (
                2.0     *       dev_var_in[(u_offset) + pp3] 
            -   5.0     *       dev_var_in[(u_offset) + pp4] 
            +   4.0     *       dev_var_in[(u_offset) + pp5] 
            -                   dev_var_in[(u_offset) + pp6]
            )*idx_sqrd;
    
        output[pp4] = (
                            dev_var_in[(u_offset) + pp3]
            -   2.0     *   dev_var_in[(u_offset) + pp4]
            +               dev_var_in[(u_offset) + pp5]
        )*idx_sqrd;
    }
                        
    if ((bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
        int pp1 = IDX(host_sz_x - 4, j, k); // IDX(ie-1,j,k)
        int pp2 = IDX(host_sz_x - 5, j, k); // IDX(ie-2,j,k)
        int pp3 = IDX(host_sz_x - 6, j, k); // IDX(ie-3,j,k)
        int pp4 = IDX(host_sz_x - 7, j, k); // IDX(ie-4,j,k)

        output[pp2] = (
                                dev_var_in[(u_offset) + pp3] 
                -   2.0     *   dev_var_in[(u_offset) + pp2] 
                +               dev_var_in[(u_offset) + pp1] 
                )*idx_sqrd;


        output[pp1] = (
            -   1.0 *   dev_var_in[(u_offset) + pp4] 
            +   4.0 *   dev_var_in[(u_offset) + pp3] 
            -   5.0 *   dev_var_in[(u_offset) + pp2] 
            +   2.0 *   dev_var_in[(u_offset) + pp1]
            )*idx_sqrd;
    }

}

__device__ void calc_deriv42_yy_bflag(int id, double* output, double * dev_var_in, const int u_offset, double dy, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idy_sqrd = 1.0/(dy*dy);
    const double idy_sqrd_by_12 = idy_sqrd / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2*nx] 
                + 16.0*dev_var_in[(u_offset) + pp - nx] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + nx] 
                - dev_var_in[(u_offset) + pp + 2*nx] 
            )*idy_sqrd_by_12;

    if ((bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
        int pp3 = IDX(i, 3, k); 
        int pp4 = IDX(i, 4, k); 
        int pp5 = IDX(i, 5, k); 
        int pp6 = IDX(i, 6, k); 
    
        output[pp3] = (
                2.0 *   dev_var_in[(u_offset) + pp3] 
            -   5.0 *   dev_var_in[(u_offset) + pp4] 
            +   4.0 *   dev_var_in[(u_offset) + pp5] 
            -           dev_var_in[(u_offset) + pp6]
            ) * idy_sqrd;
    
        output[pp4] = (
                        dev_var_in[(u_offset) + pp3]
            -   2.0 *   dev_var_in[(u_offset) + pp4]
            +           dev_var_in[(u_offset) + pp5]
        ) * idy_sqrd;
    }
                            
    if ((bflag & (1u<<OCT_DIR_UP)) && j==4)  {
        int pp1 = IDX(i, host_sz_y - 4, k); 
        int pp2 = IDX(i, host_sz_y - 5, k); 
        int pp3 = IDX(i, host_sz_y - 6, k); 
        int pp4 = IDX(i, host_sz_y - 7, k); 
    
        output[pp2] = (
                        dev_var_in[(u_offset) + pp3] 
            -   2.0 *   dev_var_in[(u_offset) + pp2] 
            +           dev_var_in[(u_offset) + pp1] 
            ) * idy_sqrd;
    
    
        output[pp1] = (
            -   1.0 *   dev_var_in[(u_offset) + pp4] 
            +   4.0 *   dev_var_in[(u_offset) + pp3] 
            -   5.0 *   dev_var_in[(u_offset) + pp2] 
            +   2.0 *   dev_var_in[(u_offset) + pp1]
            ) * idy_sqrd;
    
    }
            
}

__device__ void calc_deriv42_zz_bflag(int id, double* output, double * dev_var_in, const int u_offset, double dz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag){
    int nx = host_sz_x; 
    int ny = host_sz_y; 
    int n = nx * ny;

    const double idz_sqrd = 1.0/(dz*dz);
    const double idz_sqrd_by_12 = idz_sqrd / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2*n] 
                + 16.0*dev_var_in[(u_offset) + pp - n] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + n] 
                - dev_var_in[(u_offset) + pp + 2*n] 
            )*idz_sqrd_by_12;
    
    if ((bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
        int pp3 = IDX(i, j, 3); 
        int pp4 = IDX(i, j, 4); 
        int pp5 = IDX(i, j, 5); 
        int pp6 = IDX(i, j, 6); 
    
        output[pp3] = (
                2.0 *   dev_var_in[(u_offset) + pp3] 
            -   5.0 *   dev_var_in[(u_offset) + pp4] 
            +   4.0 *   dev_var_in[(u_offset) + pp5] 
            -           dev_var_in[(u_offset) + pp6]
            ) * idz_sqrd;
    
        output[pp4] = (
                        dev_var_in[(u_offset) + pp3]
            -   2.0 *   dev_var_in[(u_offset) + pp4]
            +           dev_var_in[(u_offset) + pp5]
        ) * idz_sqrd;
    }
                                        
    if ((bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
        int pp1 = IDX(i, j, host_sz_z - 4); 
        int pp2 = IDX(i, j, host_sz_z - 5); 
        int pp3 = IDX(i, j, host_sz_z - 6); 
        int pp4 = IDX(i, j, host_sz_z - 7); 

        output[pp2] = (
                            dev_var_in[(u_offset) + pp3] 
                -   2.0 *   dev_var_in[(u_offset) + pp2] 
                +           dev_var_in[(u_offset) + pp1] 
                ) * idz_sqrd;


        output[pp1] = (
            -   1.0 *   dev_var_in[(u_offset) + pp4] 
            +   4.0 *   dev_var_in[(u_offset) + pp3] 
            -   5.0 *   dev_var_in[(u_offset) + pp2] 
            +   2.0 *   dev_var_in[(u_offset) + pp1]
            ) * idz_sqrd;
    }
}

__device__ void calc_deriv42_adv_x_bflag(int id, double * output, double * dev_var_in, int u_offset, double dx, int betax,
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) {
    int nx = host_sz_x;
    int ny = host_sz_y;

    const double idx = 1.0/dx;
    const double idx_by_2 = 0.50 * idx;
    const double idx_by_12 = idx / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betax + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - 1]
                    - 10.0 * dev_var_in[u_offset + pp]
                    + 18.0 * dev_var_in[u_offset + pp + 1]
                    -  6.0 * dev_var_in[u_offset + pp + 2]
                    +        dev_var_in[u_offset + pp + 3]
                ) * idx_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3]
                    +  6.0 * dev_var_in[u_offset + pp - 2]
                    - 18.0 * dev_var_in[u_offset + pp - 1]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +1]
                ) * idx_by_12;
    }

    if ((bflag & (1u<<OCT_DIR_LEFT)) && (i == 3)) {
        output[IDX(3,j,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(3,j,k)]
                +  4.0 * dev_var_in[u_offset + IDX(4,j,k)]
                -        dev_var_in[u_offset + IDX(5,j,k)]
                ) * idx_by_2;

        if (dev_var_in[betax + IDX(4,j,k)] > 0.0) {
            output[IDX(4,j,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(4,j,k)]
                            +  4.0 * dev_var_in[u_offset + IDX(5,j,k)]
                            -        dev_var_in[u_offset + IDX(6,j,k)]
                        ) * idx_by_2;
        } else {
            output[IDX(4,j,k)] = ( -         dev_var_in[u_offset + IDX(3,j,k)]
                            +        dev_var_in[u_offset + IDX(5,j,k)]
                        ) * idx_by_2;
        }

        if (dev_var_in[betax + IDX(5,j,k)] > 0.0 ) {
            output[IDX(5,j,k)] = (-  3.0 * dev_var_in[u_offset + IDX(4,j,k)]
                        - 10.0 * dev_var_in[u_offset + IDX(5,j,k)]
                        + 18.0 * dev_var_in[u_offset + IDX(6,j,k)]
                        -  6.0 * dev_var_in[u_offset + IDX(7,j,k)]
                        +        dev_var_in[u_offset + IDX(8,j,k)]
                        ) * idx_by_12;
        } else {
            output[IDX(5,j,k)] = (           dev_var_in[u_offset + IDX(3,j,k)]
                            -  4.0 * dev_var_in[u_offset + IDX(4,j,k)]
                            +  3.0 * dev_var_in[u_offset + IDX(5,j,k)]
                        ) * idx_by_2;
        }
    }

    if ((bflag & (1u<<OCT_DIR_RIGHT)) && (i == 4)) {
        const int ie = nx - 3;
        if ( dev_var_in[betax + IDX(ie-3,j,k)] < 0.0 ) {
            output[IDX(ie-3,j,k)] = (  - 3.0 * dev_var_in[u_offset + IDX(ie-3,j,k)]
                                    + 4.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
                                    -       dev_var_in[u_offset + IDX(ie-1,j,k)]
                                ) * idx_by_2;
        } else {
            output[IDX(ie-3,j,k)] = ( -   dev_var_in[u_offset + IDX(ie-6,j,k)]
                            +  6.0 * dev_var_in[u_offset + IDX(ie-5,j,k)]
                            - 18.0 * dev_var_in[u_offset + IDX(ie-4,j,k)]
                            + 10.0 * dev_var_in[u_offset + IDX(ie-3  ,j,k)]
                            +  3.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
                            ) * idx_by_12;
        }

        if (dev_var_in[betax + IDX(ie-2,j,k)] > 0.0 ) {
            output[IDX(ie-2,j,k)] = (  -  dev_var_in[u_offset + IDX(ie-3,j,k)]
                                    +  dev_var_in[u_offset + IDX(ie-1,j,k)]
                                ) * idx_by_2;
        } else {
            output[IDX(ie-2,j,k)] = (     dev_var_in[u_offset + IDX(ie-4,j,k)]
                            - 4.0 * dev_var_in[u_offset + IDX(ie-3,j,k)]
                            + 3.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
                                ) * idx_by_2;
        }

        output[IDX(ie-1,j,k)] = (          dev_var_in[u_offset + IDX(ie-3,j,k)]
                                - 4.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
                                + 3.0 * dev_var_in[u_offset + IDX(ie-1,j,k)]
                            ) * idx_by_2;
    }
}

__device__ void calc_deriv42_adv_y_bflag(int id, double * output, double * dev_var_in, int u_offset, double dy, int betay,
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) {
    
    int nx = host_sz_x;
    int ny = host_sz_y;

    const double idy = 1.0/dy;
    const double idy_by_2 = 0.50 * idy;
    const double idy_by_12 = idy / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betay + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - nx]
                        - 10.0 * dev_var_in[u_offset + pp]
                        + 18.0 * dev_var_in[u_offset + pp + nx]
                        -  6.0 * dev_var_in[u_offset + pp + 2*nx]
                        +        dev_var_in[u_offset + pp + 3*nx]
                    ) * idy_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3*nx]
                    +  6.0 * dev_var_in[u_offset + pp - 2*nx]
                    - 18.0 * dev_var_in[u_offset + pp - nx]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +nx]
                    ) * idy_by_12;
                
    }

    if ((bflag & (1u<<OCT_DIR_DOWN)) && (j == 3)) {
            
        output[IDX(i,3,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,3,k)]
                +  4.0 * dev_var_in[u_offset + IDX(i,4,k)]
                -        dev_var_in[u_offset + IDX(i,5,k)]
                ) * idy_by_2;
                
        if (dev_var_in[betay + IDX(i,4,k)] > 0.0) {
            output[IDX(i,4,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,4,k)]
                            +  4.0 * dev_var_in[u_offset + IDX(i,5,k)]
                            -        dev_var_in[u_offset + IDX(i,6,k)]
                        ) * idy_by_2;

        }
        else {
            output[IDX(i,4,k)] = ( -         dev_var_in[u_offset + IDX(i,3,k)]
                            +        dev_var_in[u_offset + IDX(i,5,k)]
                        ) * idy_by_2;
                        
        }

        if (dev_var_in[betay + IDX(i,5,k)] > 0.0 ) {
            output[IDX(i,5,k)] = (-  3.0 * dev_var_in[u_offset + IDX(i,4,k)]
                        - 10.0 * dev_var_in[u_offset + IDX(i,5,k)]
                        + 18.0 * dev_var_in[u_offset + IDX(i,6,k)]
                        -  6.0 * dev_var_in[u_offset + IDX(i,7,k)]
                        +        dev_var_in[u_offset + IDX(i,8,k)]
                        ) * idy_by_12;
        }
        else {
            output[IDX(i,5,k)] = (           dev_var_in[u_offset + IDX(i,3,k)]
                            -  4.0 * dev_var_in[u_offset + IDX(i,4,k)]
                            +  3.0 * dev_var_in[u_offset + IDX(i,5,k)]
                        ) * idy_by_2;
        }
    }

    if ((bflag & (1u<<OCT_DIR_UP)) && (j == 4)) {
        
        const int je = host_sz_y - 3;
        
        if ( dev_var_in[betay + IDX(i,je-3,k)] < 0.0 ) {
            output[IDX(i,je-3,k)] = (  - 3.0 * dev_var_in[u_offset + IDX(i,je-3,k)]
                                    + 4.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
                                    -       dev_var_in[u_offset + IDX(i,je-1,k)]
                                    ) * idy_by_2;
        }
        else {
            output[IDX(i,je-3,k)] = ( -   dev_var_in[u_offset + IDX(i,je-6,k)]
                                +  6.0 * dev_var_in[u_offset + IDX(i,je-5,k)]
                                - 18.0 * dev_var_in[u_offset + IDX(i,je-4,k)]
                                + 10.0 * dev_var_in[u_offset + IDX(i,je-3,k)]
                                +  3.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
                            ) * idy_by_12;
        }
    
            if (dev_var_in[betay + IDX(i,je-2,k)] > 0.0 ) {
            output[IDX(i,je-2,k)] = (  -  dev_var_in[u_offset + IDX(i,je-3,k)]
                                    +  dev_var_in[u_offset + IDX(i,je-1,k)]
                                    ) * idy_by_2;
            }
            else {
            output[IDX(i,je-2,k)] = (     dev_var_in[u_offset + IDX(i,je-4,k)]
                                - 4.0 * dev_var_in[u_offset + IDX(i,je-3,k)]
                                + 3.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
                                    ) * idy_by_2;
            }
    
            output[IDX(i,je-1,k)]  = (          dev_var_in[u_offset + IDX(i,je-3,k)]
                                    - 4.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
                                    + 3.0 * dev_var_in[u_offset + IDX(i,je-1,k)]
                                ) * idy_by_2;
    }
}

__device__ void calc_deriv42_adv_z_bflag(int id, double * output, double * dev_var_in, int u_offset, double dz, int betaz, 
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) {

    int nx = host_sz_x;
    int ny = host_sz_y;
    int n = nx * ny;
    
    const double idz = 1.0/dz;
    const double idz_by_2 = 0.50 * idz;
    const double idz_by_12 = idz / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betaz + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - n]
                        - 10.0 * dev_var_in[u_offset + pp]
                        + 18.0 * dev_var_in[u_offset + pp + n]
                        -  6.0 * dev_var_in[u_offset + pp + 2*n]
                        +        dev_var_in[u_offset + pp + 3*n]
                    ) * idz_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3*n]
                    +  6.0 * dev_var_in[u_offset + pp - 2*n]
                    - 18.0 * dev_var_in[u_offset + pp - n]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +n]
                    ) * idz_by_12;
                
    }

    if ((bflag & (1u<<OCT_DIR_BACK)) && (k == 3)) {
            
        output[IDX(i,j,3)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,j,3)]
                +  4.0 * dev_var_in[u_offset + IDX(i,j,4)]
                -        dev_var_in[u_offset + IDX(i,j,5)]
                ) * idz_by_2;
                
        if (dev_var_in[betaz + IDX(i,j,4)] > 0.0) {
            output[IDX(i,j,4)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,j,4)]
                            +  4.0 * dev_var_in[u_offset + IDX(i,j,5)]
                            -        dev_var_in[u_offset + IDX(i,j,6)]
                        ) * idz_by_2;

        }
        else {
            output[IDX(i,j,4)] = ( -         dev_var_in[u_offset + IDX(i,j,3)]
                            +        dev_var_in[u_offset + IDX(i,j,5)]
                        ) * idz_by_2;
                        
        }

        if (dev_var_in[betaz + IDX(i,j,5)] > 0.0 ) {
            output[IDX(i,j,5)] = (-  3.0 * dev_var_in[u_offset + IDX(i,j,4)]
                        - 10.0 * dev_var_in[u_offset + IDX(i,j,5)]
                        + 18.0 * dev_var_in[u_offset + IDX(i,j,6)]
                        -  6.0 * dev_var_in[u_offset + IDX(i,j,7)]
                        +        dev_var_in[u_offset + IDX(i,j,8)]
                        ) * idz_by_12;
        }
        else {
            output[IDX(i,j,5)] = (           dev_var_in[u_offset + IDX(i,j,3)]
                            -  4.0 * dev_var_in[u_offset + IDX(i,j,4)]
                            +  3.0 * dev_var_in[u_offset + IDX(i,j,5)]
                        ) * idz_by_2;
        }
    }

    if ((bflag & (1u<<OCT_DIR_FRONT)) && (k == 4)) {
        
        const int ke = host_sz_z - 3; // Here I changed
        
        if ( dev_var_in[betaz + IDX(i,j,ke-3)] < 0.0 ) {
            output[IDX(i,j,ke-3)] = (  - 3.0 * dev_var_in[u_offset + IDX(i,j,ke-3)]
                                    + 4.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
                                    -       dev_var_in[u_offset + IDX(i,j,ke-1)]
                                    ) * idz_by_2;
        }
        else {
            output[IDX(i,j,ke-3)] = ( -   dev_var_in[u_offset + IDX(i,j,ke-6)]
                                +  6.0 * dev_var_in[u_offset + IDX(i,j,ke-5)]
                                - 18.0 * dev_var_in[u_offset + IDX(i,j,ke-4)]
                                + 10.0 * dev_var_in[u_offset + IDX(i,j,ke-3)]
                                +  3.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
                            ) * idz_by_12;
        }
    
            if (dev_var_in[betaz + IDX(i,j,ke-2)] > 0.0 ) {
            output[IDX(i,j,ke-2)] = (  -  dev_var_in[u_offset + IDX(i,j,ke-3)]
                                    +  dev_var_in[u_offset + IDX(i,j,ke-1)]
                                    ) * idz_by_2;
            }
            else {
            output[IDX(i,j,ke-2)] = (     dev_var_in[u_offset + IDX(i,j,ke-4)]
                                - 4.0 * dev_var_in[u_offset + IDX(i,j,ke-3)]
                                + 3.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
                                    ) * idz_by_2;
            }
    
            output[IDX(i,j,ke-1)]  = (          dev_var_in[u_offset + IDX(i,j,ke-3)]
                                    - 4.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
                                    + 3.0 * dev_var_in[u_offset + IDX(i,j,ke-1)]
                                ) * idz_by_2;
    }
}


__device__ void calc_ko_deriv42_x_bflag(int id, double * output, double * dev_var_in, int u_offset, double dx, 
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x;
    int ny = host_sz_y;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    
    if(i==4) {
        int ib=3;
        output[IDX(3, j, k)] = (-1.0 / 64.0 / dx) *
                        (
                        -      dev_var_in[u_offset + IDX(ib+4,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ib+3,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ib+2,j,k)]
                        + 20.0*dev_var_in[u_offset + IDX(ib+1,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ib,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ib-1,j,k)]
                        -      dev_var_in[u_offset + IDX(ib-2,j,k)]
                        );
    }

    output[pp] = (-1.0 / 64.0 / dx) *
                            (
                            -      dev_var_in[u_offset + pp - 3]
                            +  6.0*dev_var_in[u_offset + pp - 2]
                            - 15.0*dev_var_in[u_offset + pp - 1]
                            + 20.0*dev_var_in[u_offset + pp ]
                            - 15.0*dev_var_in[u_offset + pp + 1]
                            +  6.0*dev_var_in[u_offset + pp + 2]
                            -      dev_var_in[u_offset + pp + 3]
                            );

    if(i==5) {
        int ie = nx-3;
        output[IDX(ie-1, j, k)] = (-1.0 / 64.0 / dx) *
                        (
                        -      dev_var_in[u_offset + IDX(ie+1,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ie,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
                        + 20.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
                        -      dev_var_in[u_offset + IDX(ie-5,j,k)]
                        );
    }

    if ((bflag & (1u<<OCT_DIR_LEFT)) && (i == 4)) {

        output[IDX(3,j,k)] =  (      dev_var_in[u_offset + IDX(6,j,k)]
                                    - 3.0*dev_var_in[u_offset + IDX(5,j,k)]
                                    + 3.0*dev_var_in[u_offset + IDX(4,j,k)]
                                    -     dev_var_in[u_offset + IDX(3,j,k)]
                                )/59.0/48.0*64*dx;
        output[IDX(4,j,k)] =  (     dev_var_in[u_offset + IDX(7,j,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(6,j,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(5,j,k)]
                                    - 10.0*dev_var_in[u_offset + IDX(4,j,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(3,j,k)]
                                    )/43.0/48.0*64*dx;
        output[IDX(5,j,k)] =  (     dev_var_in[u_offset + IDX(8,j,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(7,j,k)]
                                    + 15.0*dev_var_in[u_offset + IDX(6,j,k)]
                                    - 19.0*dev_var_in[u_offset + IDX(5,j,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(4,j,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(3,j,k)]
                                    )/49.0/48.0*64*dx;
        }

    if ((bflag & (1u<<OCT_DIR_RIGHT)) && (i == 5)) {
        
        const int ie = nx - 3;
        output[IDX(ie-3,j,k)] = ( dev_var_in[u_offset + IDX(ie-6,j,k)]
                                    - 6.0*dev_var_in[u_offset + IDX(ie-5,j,k)]
                                    + 15.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
                                    - 19.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
                                    )/49.0/48.0*64*dx;
            
            output[IDX(ie-2,j,k)] =  ( dev_var_in[u_offset + IDX(ie-5,j,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
                                    - 10.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
                                    )/43.0/48.0*64*dx;
        
    
            output[IDX(ie-1,j,k)] = ( dev_var_in[u_offset + IDX(ie-4,j,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
                                    -      dev_var_in[u_offset + IDX(ie-1,j,k)]
                                    )/59.0/48.0*64*dx;
    }
}

__device__ void calc_ko_deriv42_y_bflag(int id, double * output, double * dev_var_in, int u_offset, double dy,
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    
    int nx = host_sz_x;
    int ny = host_sz_y;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if(j==4) {
        int jb=3;
        output[IDX(i,jb,k)] = (-1.0 / 64.0 / dy) *
                    (
                        -      dev_var_in[u_offset + IDX(i,jb+4,k)]
                        +  6.0*dev_var_in[u_offset + IDX(i,jb+3,k)]
                        - 15.0*dev_var_in[u_offset + IDX(i,jb+2,k)]
                        + 20.0*dev_var_in[u_offset + IDX(i,jb+1,k)]
                        - 15.0*dev_var_in[u_offset + IDX(i,jb,k)]
                        +  6.0*dev_var_in[u_offset + IDX(i,jb-1,k)]
                        -      dev_var_in[u_offset + IDX(i,jb-2,k)]
                        );
        }
    output[pp] = (-1.0 / 64.0 / dy) *
                    (
                        -      dev_var_in[u_offset + pp-3*nx]
                        +  6.0*dev_var_in[u_offset + pp-2*nx]
                        - 15.0*dev_var_in[u_offset + pp-nx]
                        + 20.0*dev_var_in[u_offset + pp]
                        - 15.0*dev_var_in[u_offset + pp+nx]
                        +  6.0*dev_var_in[u_offset + pp+2*nx]
                        -      dev_var_in[u_offset + pp+3*nx]
                        );

    if(j==5) {
        int je = ny - 3;
        output[IDX(i,je-1,k)] = (-1.0 / 64.0 / dy) *
                (
                    -      dev_var_in[u_offset + IDX(i,je+1,k)]
                    +  6.0*dev_var_in[u_offset + IDX(i,je,k)]
                    - 15.0*dev_var_in[u_offset + IDX(i,je-1,k)]
                    + 20.0*dev_var_in[u_offset + IDX(i,je-2,k)]
                    - 15.0*dev_var_in[u_offset + IDX(i,je-3,k)]
                    +  6.0*dev_var_in[u_offset + IDX(i,je-4,k)]
                    -      dev_var_in[u_offset + IDX(i,je-5,k)]
                    );                   
    }

    if ((bflag & (1u<<OCT_DIR_DOWN)) && (j == 4)) {

        output[IDX(i,3,k)] =  (      dev_var_in[u_offset +IDX(i,6,k)]
                                    - 3.0*dev_var_in[u_offset +IDX(i,5,k)]
                                    + 3.0*dev_var_in[u_offset + IDX(i,4,k)]
                                    -     dev_var_in[u_offset + IDX(i,3,k)]
                                )/59.0/48.0*64*dy;
        output[IDX(i,4,k)] =  (     dev_var_in[u_offset + IDX(i,7,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,6,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,5,k)]
                                    - 10.0*dev_var_in[u_offset + IDX(i,4,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(i,3,k)]
                                    )/43.0/48.0*64*dy;
        output[IDX(i,5,k)] =  (     dev_var_in[u_offset + IDX(i,8,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,7,k)]
                                    + 15.0*dev_var_in[u_offset + IDX(i,6,k)]
                                    - 19.0*dev_var_in[u_offset + IDX(i,5,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,4,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(i,3,k)]
                                    )/49.0/48.0*64*dy;
        }

    if ((bflag & (1u<<OCT_DIR_UP)) && (j == 5)) {
        
        const int je = ny - 3;
        output[IDX(i,je-3,k)] = (dev_var_in[u_offset + IDX(i,je-6,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,je-5,k)]
                                    + 15.0*dev_var_in[u_offset + IDX(i,je-4,k)]
                                    - 19.0*dev_var_in[u_offset + IDX(i,je-3,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,je-2,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(i,je-1,k)]
                                    )/49.0/48.0*64*dy;
            
            output[IDX(i,je-2,k)] = (dev_var_in[u_offset + IDX(i,je-5,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,je-4,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,je-3,k)]
                                    - 10.0*dev_var_in[u_offset + IDX(i,je-2,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(i,je-1,k)]
                                    )/43.0/48.0*64*dy;
        
    
            output[IDX(i,je-1,k)] = ( dev_var_in[u_offset + IDX(i,je-4,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(i,je-3,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(i,je-2,k)]
                                    -      dev_var_in[u_offset + IDX(i,je-1,k)]
                                    )/59.0/48.0*64*dy;
    }
}

__device__ void calc_ko_deriv42_z_bflag(int id, double * output, double * dev_var_in, int u_offset, double dz, 
const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{  
    int nx = host_sz_x;
    int ny = host_sz_y;
    int n = nx * ny;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    
    if(k==4) {
        int kb=3;
        output[IDX(i,j,kb)] = (-1.0 / 64.0 / dz) *
                    (
                        -      dev_var_in[u_offset + IDX(i,j,kb+4)]
                        +  6.0*dev_var_in[u_offset + IDX(i,j,kb+3)]
                        - 15.0*dev_var_in[u_offset + IDX(i,j,kb+2)]
                        + 20.0*dev_var_in[u_offset + IDX(i,j,kb+1)]
                        - 15.0*dev_var_in[u_offset + IDX(i,j,kb)]
                        +  6.0*dev_var_in[u_offset + IDX(i,j,kb-1)]
                        -      dev_var_in[u_offset + IDX(i,j,kb-2)]
                        );
        }

    output[pp] = (-1.0 / 64.0 / dz) *
                (
                    -      dev_var_in[u_offset + pp-3*n]
                    +  6.0*dev_var_in[u_offset + pp-2*n]
                    - 15.0*dev_var_in[u_offset + pp-n]
                    + 20.0*dev_var_in[u_offset + pp]
                    - 15.0*dev_var_in[u_offset + pp+n]
                    +  6.0*dev_var_in[u_offset + pp+2*n]
                    -      dev_var_in[u_offset + pp+3*n]
                    );
    if(k==5) {
        int ke = host_sz_z - 3;
        output[IDX(i,j,ke-1)] = (-1.0 / 64.0 / dz) *
        (
            -      dev_var_in[u_offset + IDX(i,j,ke+1)]
            +  6.0*dev_var_in[u_offset + IDX(i,j,ke)]
            - 15.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
            + 20.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
            - 15.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
            +  6.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
            -      dev_var_in[u_offset + IDX(i,j,ke-5)]
            );               
    }

    if ((bflag & (1u<<OCT_DIR_BACK)) && (k == 4)) {

        output[IDX(i,3,k)] =  (      dev_var_in[u_offset +IDX(i,k,6)]
                                    - 3.0*dev_var_in[u_offset +IDX(i,k,5)]
                                    + 3.0*dev_var_in[u_offset + IDX(i,k,4)]
                                    -     dev_var_in[u_offset + IDX(i,k,3)]
                                )/59.0/48.0*64*dz;

        output[IDX(i,j,4)] =  (     dev_var_in[u_offset + IDX(i,j,7)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,j,6)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,j,5)]
                                    - 10.0*dev_var_in[u_offset + IDX(i,j,4)]
                                    +  3.0*dev_var_in[u_offset + IDX(i,j,3)]
                                    )/43.0/48.0*64*dz;

        output[IDX(i,j,5)] =  (     dev_var_in[u_offset + IDX(i,j,8)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,j,7)]
                                    + 15.0*dev_var_in[u_offset + IDX(i,j,6)]
                                    - 19.0*dev_var_in[u_offset + IDX(i,j,5)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,j,4)]
                                    -  3.0*dev_var_in[u_offset + IDX(i,j,3)]
                                    )/49.0/48.0*64*dz;
        }

    if ((bflag & (1u<<OCT_DIR_FRONT)) && (k == 5)) {
        
        const int ke = host_sz_z - 3;
        output[IDX(i,j,ke-3)] = (    dev_var_in[u_offset + IDX(i,j,ke-6)]
                                        -  6.0*dev_var_in[u_offset + IDX(i,j,ke-5)]
                                        + 15.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
                                        - 19.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
                                        + 12.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
                                        -  3.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
                                        )/49.0/48.0*64*dz;
            
            output[IDX(i,j,ke-2)] = (   dev_var_in[u_offset + IDX(i,j,ke-5)]
                                        -  6.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
                                        + 12.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
                                        - 10.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
                                        +  3.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
                                        )/43.0/48.0*64*dz;
        
    
            output[IDX(i,j,ke-1)] = (   dev_var_in[u_offset + IDX(i,j,ke-4)]
                                        -  3.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
                                        +  3.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
                                        -      dev_var_in[u_offset + IDX(i,j,ke-1)]
                                        )/59.0/48.0*64*dz;
    }
        
}