//
// Created by milinda on 8/10/18.
//
#include "rhs_cuda.cuh"


namespace cuda
{



    void computeRHS(double **unzipVarsRHS, const double **uZipVars,const cuda::_Block* blkList,
            unsigned int numBlocks, hipStream_t stream, cuda::_Block* blockListReference, 
            double** referenceToInput, cuda::MemoryDerivs* derivPointer, 
            hipDeviceProp_t* cudaDeviceProperties, cuda::BSSNComputeParams* bSSNComputeParams,
            double** outputReference)
    {
        cuda::profile::t_overall.start();

        cuda::profile::t_H2D_Comm.start();

            //get GPU information.
            cuda::__CUDA_DEVICE_PROPERTIES = cudaDeviceProperties;
            cuda::__UNZIP_OUTPUT = outputReference;
            cuda::__BSSN_COMPUTE_PARMS = bSSNComputeParams;
            cuda::__DENDRO_BLOCK_LIST = blockListReference;
            cuda::__UNZIP_INPUT = referenceToInput;
            cuda::__BSSN_DERIV_WORKSPACE = derivPointer;

        cuda::profile::t_H2D_Comm.stop();

        unsigned int maxBlkSz=0;
        for(unsigned int blk=0;blk<numBlocks;blk++)
        {
            const unsigned int* sz=blkList[blk].getSz();
            if(maxBlkSz<(sz[0]*sz[1]*sz[2]))
                maxBlkSz=sz[0]*sz[1]*sz[2];
        }

        dim3 blockGrid(numBlocks,1);
        dim3 threadBlock(32,4,1);

        cuda::profile::t_derivs.start();

        cuda::__RSWS_computeDerivs <<<blockGrid,threadBlock>>> ((const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        // hipDeviceSynchronize();
        // CUDA_CHECK_ERROR();

        cuda::profile::t_derivs.stop();

        threadBlock=dim3(6,6,6);
        cuda::profile::t_rhs.start();

        cuda::__compute_a_rhs<<<blockGrid,threadBlock>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_b_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_gt_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_chi_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_At_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_K_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_Gt_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_B_rhs<<<blockGrid, threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        hipDeviceSynchronize();
        CUDA_CHECK_ERROR();

        cuda::profile::t_rhs.stop();

        // cuda::profile::t_cudaMalloc_derivs.start();
        //     derivWorkSpace.deallocateDerivMemory();
        //     CUDA_CHECK_ERROR();
        // cuda::profile::t_cudaMalloc_derivs.stop();

        // hipFree(cuda::__CUDA_DEVICE_PROPERTIES);
        // hipFree(cuda::__DENDRO_BLOCK_LIST);

        // cuda::copyBackMemory(cuda::__UNZIP_INPUT,BSSN_NUM_VARS, stream);

        cuda::profile::t_overall.stop();
    }

}
