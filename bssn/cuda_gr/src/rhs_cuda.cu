//
// Created by milinda on 8/10/18.
//
#include "rhs_cuda.cuh"


namespace cuda
{



    void computeRHS(double **unzipVarsRHS, const double **uZipVars,const cuda::_Block* blkList,unsigned int numBlocks,const cuda::BSSNComputeParams* bssnPars)
    {
        cuda::profile::t_overall.start();

        cuda::profile::t_H2D_Comm.start();

            //get GPU information.
            // assumes the if there are multiple gpus per node all have the same specification.
            cuda::__CUDA_DEVICE_PROPERTIES=getGPUDeviceInfo(0);
            // device properties for the host
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp,0);

            const double GPU_BLOCK_SHARED_MEM_UTIL=0.8;
            const unsigned int BSSN_NUM_VARS=24;
            const unsigned int BSSN_CONSTRAINT_NUM_VARS=6;

            const unsigned int UNZIP_DOF_SZ=blkList[numBlocks-1].getOffset()+(blkList[numBlocks-1].getSz()[0]*blkList[numBlocks-1].getSz()[1]*blkList[numBlocks-1].getSz()[2]);


            //send blocks to the gpu
            cuda::__DENDRO_BLOCK_LIST=cuda::copyArrayToDevice(blkList,numBlocks);
            cuda::__DENDRO_NUM_BLOCKS=cuda::copyValueToDevice(&numBlocks);

            cuda::__BSSN_NUM_VARS=cuda::copyValueToDevice(&BSSN_NUM_VARS);
            cuda::__BSSN_CONSTRAINT_NUM_VARS=cuda::copyValueToDevice(&BSSN_CONSTRAINT_NUM_VARS);

            cuda::__GPU_BLOCK_SHARED_MEM_UTIL=cuda::copyValueToDevice(&GPU_BLOCK_SHARED_MEM_UTIL);

            //allocate memory for unzip vectors
            cuda::__UNZIP_INPUT=cuda::alloc2DCudaArray<double>(uZipVars,BSSN_NUM_VARS,UNZIP_DOF_SZ);
            cuda::__UNZIP_OUTPUT=cuda::alloc2DCudaArray<double>(BSSN_NUM_VARS,UNZIP_DOF_SZ);

            cuda::__BSSN_COMPUTE_PARMS=cuda::copyValueToDevice(&(*bssnPars));


        cuda::profile::t_H2D_Comm.stop();

        unsigned int maxBlkSz=0;
        for(unsigned int blk=0;blk<numBlocks;blk++)
        {
            const unsigned int* sz=blkList[blk].getSz();
            if(maxBlkSz<(sz[0]*sz[1]*sz[2]))
                maxBlkSz=sz[0]*sz[1]*sz[2];
        }

        const unsigned int derivSz=(maxBlkSz);
        cuda::__DENDRO_BLK_MAX_SZ=cuda::copyValueToDevice(&derivSz);
        //const size_t deriv_mem_sz= derivSz*(deviceProp.multiProcessorCount);
        const unsigned int numSM=deviceProp.multiProcessorCount;

        //std::cout<<"deriv alloc begin"<<std::endl;

        cuda::profile::t_cudaMalloc_derivs.start();

            cuda::MemoryDerivs derivWorkSpace;
            derivWorkSpace.allocateDerivMemory(maxBlkSz,numSM);
            CUDA_CHECK_ERROR();

            cuda::__BSSN_DERIV_WORKSPACE=cuda::copyValueToDevice(&derivWorkSpace);
            CUDA_CHECK_ERROR();

        cuda::profile::t_cudaMalloc_derivs.stop();



        dim3 blockGrid(numBlocks,1);
        dim3 threadBlock(2,2,2);


        cuda::profile::t_derivs.start();


        cuda::__RSWS_computeDerivs <<<blockGrid,threadBlock>>> ((const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        hipDeviceSynchronize();
        CUDA_CHECK_ERROR();

        cuda::profile::t_derivs.stop();

        threadBlock=dim3(6,6,6);
        cuda::profile::t_rhs.start();

        /*cuda::__compute_a_rhs<<<blockGrid,threadBlock>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_b_rhs<<<blockGrid,threadBlock>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_gt_rhs<<<blockGrid,threadBlock>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_chi_rhs<<<blockGrid,threadBlock>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_At_rhs<<<blockGrid,threadBlock>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_K_rhs<<<blockGrid,threadBlock>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_Gt_rhs<<<blockGrid,threadBlock>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_B_rhs<<<blockGrid,threadBlock>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        hipDeviceSynchronize();
        CUDA_CHECK_ERROR();*/

        cuda::profile::t_rhs.stop();

        cuda::profile::t_cudaMalloc_derivs.start();
            derivWorkSpace.deallocateDerivMemory();
            CUDA_CHECK_ERROR();
        cuda::profile::t_cudaMalloc_derivs.stop();

        hipFree(cuda::__CUDA_DEVICE_PROPERTIES);
        hipFree(cuda::__DENDRO_BLOCK_LIST);
        hipFree(cuda::__DENDRO_NUM_BLOCKS);
        hipFree(cuda::__BSSN_NUM_VARS);
        hipFree(cuda::__BSSN_CONSTRAINT_NUM_VARS);
        hipFree(cuda::__GPU_BLOCK_SHARED_MEM_UTIL);

        cuda::dealloc2DCudaArray(cuda::__UNZIP_INPUT,BSSN_NUM_VARS);
        cuda::dealloc2DCudaArray(cuda::__UNZIP_OUTPUT,BSSN_NUM_VARS);



        cuda::profile::t_overall.stop();


    }

}
