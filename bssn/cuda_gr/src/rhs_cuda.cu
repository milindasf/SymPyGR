//
// Created by milinda on 8/10/18.
//
#include "rhs_cuda.cuh"


namespace cuda
{



    void computeRHS(double **unzipVarsRHS, const double **uZipVars,const cuda::_Block* blkList,
            unsigned int numBlocks, hipStream_t stream,
            cuda::_Block* blockListReference, double** tmp2D, double** referenceToInput,
            cuda::MemoryDerivs derivWorkSpace, cuda::MemoryDerivs* derivPointer,
            hipDeviceProp_t* cudaDeviceProperties, cuda::BSSNComputeParams* bSSNComputeParams, double** outputReference)
    {
        cuda::profile::t_overall.start();

        cuda::profile::t_H2D_Comm.start();

            //get GPU information.
            // assumes the if there are multiple gpus per node all have the same specification.
            cuda::__CUDA_DEVICE_PROPERTIES=cudaDeviceProperties;
            // device properties for the host

            const unsigned int BSSN_NUM_VARS=24;

            const unsigned int UNZIP_DOF_SZ=blkList[numBlocks-1].getOffset()+(blkList[numBlocks-1].getSz()[0]*blkList[numBlocks-1].getSz()[1]*blkList[numBlocks-1].getSz()[2]);


            //send blocks to the gpu
            cuda::__DENDRO_BLOCK_LIST=cuda::copyArrayToDevice(blockListReference,blkList,numBlocks, stream);
            cuda::__UNZIP_INPUT = referenceToInput;
            cuda::copy2DCudaArray<double>(uZipVars, tmp2D,
                    BSSN_NUM_VARS,UNZIP_DOF_SZ, cuda::__UNZIP_INPUT, stream);
            cuda::__UNZIP_OUTPUT=outputReference;

            cuda::__BSSN_COMPUTE_PARMS=bSSNComputeParams;


        cuda::profile::t_H2D_Comm.stop();

        unsigned int maxBlkSz=0;
        for(unsigned int blk=0;blk<numBlocks;blk++)
        {
            const unsigned int* sz=blkList[blk].getSz();
            if(maxBlkSz<(sz[0]*sz[1]*sz[2]))
                maxBlkSz=sz[0]*sz[1]*sz[2];
        }

        //std::cout<<"deriv alloc begin"<<std::endl;

        cuda::profile::t_cudaMalloc_derivs.start();

            cuda::__BSSN_DERIV_WORKSPACE=cuda::copyValueToDeviceAllocateDerivMemory(derivPointer, &derivWorkSpace, stream);
            CUDA_CHECK_ERROR();

        cuda::profile::t_cudaMalloc_derivs.stop();



        dim3 blockGrid(numBlocks,1);
        dim3 threadBlock(32,4,1);


        cuda::profile::t_derivs.start();


        cuda::__RSWS_computeDerivs <<<blockGrid,threadBlock>>> ((const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        // hipDeviceSynchronize();
        // CUDA_CHECK_ERROR();

        cuda::profile::t_derivs.stop();

        threadBlock=dim3(6,6,6);
        cuda::profile::t_rhs.start();

        /*cuda::__compute_a_rhs<<<blockGrid,threadBlock>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_b_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_gt_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_chi_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_At_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_K_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_Gt_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        cuda::__compute_B_rhs<<<blockGrid,threadBlock, 0, stream>>>(cuda::__UNZIP_OUTPUT,(const double**)cuda::__UNZIP_INPUT,cuda::__BSSN_DERIV_WORKSPACE,cuda::__DENDRO_BLOCK_LIST,cuda::__BSSN_COMPUTE_PARMS,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        hipDeviceSynchronize();
        CUDA_CHECK_ERROR();*/

        cuda::profile::t_rhs.stop();

        // cuda::profile::t_cudaMalloc_derivs.start();
        //     derivWorkSpace.deallocateDerivMemory();
        //     CUDA_CHECK_ERROR();
        // cuda::profile::t_cudaMalloc_derivs.stop();

        // hipFree(cuda::__CUDA_DEVICE_PROPERTIES);
        // hipFree(cuda::__DENDRO_BLOCK_LIST);

        // cuda::copyBackMemory(cuda::__UNZIP_INPUT,BSSN_NUM_VARS, stream);

        cuda::profile::t_overall.stop();
    }

}
