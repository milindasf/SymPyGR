/**
 * Created on: March 15, 2018
 * 		Author: Akila, Eranga, Eminda, Ruwan
 **/

#include "deviceDerivs.cuh"

__device__ void calc_deriv42_x(int id, double * output, double * dev_var_in, const int u_offset, double dx, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idx = 1.0/dx;
    const double idx_by_12 = idx / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-2)) + 1;
    int k = (id/(host_sz_z-2)/(host_sz_x-6)) + 1; 
    if (k>=host_sz_z-1) return;
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[u_offset+pp-2] - 8.0*dev_var_in[u_offset+pp-1] + 8.0*dev_var_in[u_offset+pp+1] - dev_var_in[u_offset+pp+2])*idx_by_12;
}

__device__ void calc_deriv42_y(int id, double* output, double * dev_var_in, const int u_offset, double dy, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idy = 1.0/dy;
    const double idy_by_12 = idy / 12.0;
    
    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 1;
    if (k>=host_sz_z-1) return;
    int pp = IDX(i, j, k);
    
    output[pp] = (dev_var_in[u_offset+pp-2*nx] - 8.0*dev_var_in[u_offset+pp-nx] + 8.0*dev_var_in[u_offset+pp+nx] - dev_var_in[u_offset+pp+2*nx])*idy_by_12;
}

__device__ void calc_deriv42_z(int id, double* output, double * dev_var_in, const int u_offset, double dz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 
    int n = nx * ny;

    const double idz = 1.0/dz;
    const double idz_by_12 = idz / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = (dev_var_in[u_offset+pp-2*n] - 8.0*dev_var_in[u_offset+pp-n] + 8.0*dev_var_in[u_offset+pp+n] - dev_var_in[u_offset+pp+2*n])*idz_by_12;
}

__device__ void calc_deriv42_xx(int id, double* output, double * dev_var_in, const int u_offset, double dx, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idx_sqrd = 1.0/(dx*dx);
    const double idx_sqrd_by_12 = idx_sqrd / 12.0;
    
    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[u_offset+pp-2] + 16.0*dev_var_in[u_offset+pp-1] - 30.0*dev_var_in[u_offset+pp] + 16.0*dev_var_in[u_offset+pp+1] - dev_var_in[u_offset+pp+2])*idx_sqrd_by_12;
}

__device__ void calc_deriv42_yy(int id, double* output, double * dev_var_in, const int u_offset, double dy, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idy_sqrd = 1.0/(dy*dy);
    const double idy_sqrd_by_12 = idy_sqrd / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[u_offset+pp-2*nx] + 16.0*dev_var_in[u_offset+pp-nx] - 30.0*dev_var_in[u_offset+pp] + 16.0*dev_var_in[u_offset+pp+nx] - dev_var_in[u_offset+pp+2*nx])*idy_sqrd_by_12;         
}

__device__ void calc_deriv42_zz(int id, double* output, double * dev_var_in, const int u_offset, double dz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 
    int n = nx * ny;

    const double idz_sqrd = 1.0/(dz*dz);
    const double idz_sqrd_by_12 = idz_sqrd / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[u_offset+pp-2*n] + 16.0*dev_var_in[u_offset+pp-n] - 30.0*dev_var_in[u_offset+pp] + 16.0*dev_var_in[u_offset+pp+n] - dev_var_in[u_offset+pp+2*n])*idz_sqrd_by_12;
}

__device__ void calc_deriv42_adv_x(int id, double * output, double * dev_var_in, int u_offset, double dx, int betax, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) 
{
    int nx = host_sz_x;
    int ny = host_sz_y;

    const double idx = 1.0/dx;
    const double idx_by_12 = idx / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betax + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - 1]
                    - 10.0 * dev_var_in[u_offset + pp]
                    + 18.0 * dev_var_in[u_offset + pp + 1]
                    -  6.0 * dev_var_in[u_offset + pp + 2]
                    +        dev_var_in[u_offset + pp + 3]
                ) * idx_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3]
                    +  6.0 * dev_var_in[u_offset + pp - 2]
                    - 18.0 * dev_var_in[u_offset + pp - 1]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +1]
                ) * idx_by_12;
    }
}

__device__ void calc_deriv42_adv_y(int id, double * output, double * dev_var_in, int u_offset, double dy, int betay, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) 
{
    int nx = host_sz_x;
    int ny = host_sz_y;

    const double idy = 1.0/dy;
    const double idy_by_12 = idy / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betay + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - nx]
                        - 10.0 * dev_var_in[u_offset + pp]
                        + 18.0 * dev_var_in[u_offset + pp + nx]
                        -  6.0 * dev_var_in[u_offset + pp + 2*nx]
                        +        dev_var_in[u_offset + pp + 3*nx]
                    ) * idy_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3*nx]
                    +  6.0 * dev_var_in[u_offset + pp - 2*nx]
                    - 18.0 * dev_var_in[u_offset + pp - nx]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +nx]
                    ) * idy_by_12;
                
    }
}

__device__ void calc_deriv42_adv_z(int id, double * output, double * dev_var_in, int u_offset, double dz, int betaz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) 
{
    int nx = host_sz_x;
    int ny = host_sz_y;
    int n = nx * ny;
    
    const double idz = 1.0/dz;
    const double idz_by_12 = idz / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betaz + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - n]
                        - 10.0 * dev_var_in[u_offset + pp]
                        + 18.0 * dev_var_in[u_offset + pp + n]
                        -  6.0 * dev_var_in[u_offset + pp + 2*n]
                        +        dev_var_in[u_offset + pp + 3*n]
                    ) * idz_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3*n]
                    +  6.0 * dev_var_in[u_offset + pp - 2*n]
                    - 18.0 * dev_var_in[u_offset + pp - n]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +n]
                    ) * idz_by_12;
                
    }
}

__device__ void calc_ko_deriv42_x(int id, double * output, double * dev_var_in, int u_offset, double dx, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag0)
{
    int nx = host_sz_x;
    int ny = host_sz_y;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    
    if(i==4) {
        int ib=3;
        output[IDX(3, j, k)] = (-1.0 / 64.0 / dx) *
                        (
                        -      dev_var_in[u_offset + IDX(ib+4,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ib+3,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ib+2,j,k)]
                        + 20.0*dev_var_in[u_offset + IDX(ib+1,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ib,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ib-1,j,k)]
                        -      dev_var_in[u_offset + IDX(ib-2,j,k)]
                        );
    }

    output[pp] = (-1.0 / 64.0 / dx) *
                            (
                            -      dev_var_in[u_offset + pp - 3]
                            +  6.0*dev_var_in[u_offset + pp - 2]
                            - 15.0*dev_var_in[u_offset + pp - 1]
                            + 20.0*dev_var_in[u_offset + pp ]
                            - 15.0*dev_var_in[u_offset + pp + 1]
                            +  6.0*dev_var_in[u_offset + pp + 2]
                            -      dev_var_in[u_offset + pp + 3]
                            );

    if(i==5) {
        int ie = nx-3;
        output[IDX(ie-1, j, k)] = (-1.0 / 64.0 / dx) *
                        (
                        -      dev_var_in[u_offset + IDX(ie+1,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ie,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
                        + 20.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
                        -      dev_var_in[u_offset + IDX(ie-5,j,k)]
                        );
    }
}

__device__ void calc_ko_deriv42_y(int id, double * output, double * dev_var_in, int u_offset, double dy, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x;
    int ny = host_sz_y;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if(j==4) {
        int jb=3;
        output[IDX(i,jb,k)] = (-1.0 / 64.0 / dy) *
                    (
                        -      dev_var_in[u_offset + IDX(i,jb+4,k)]
                        +  6.0*dev_var_in[u_offset + IDX(i,jb+3,k)]
                        - 15.0*dev_var_in[u_offset + IDX(i,jb+2,k)]
                        + 20.0*dev_var_in[u_offset + IDX(i,jb+1,k)]
                        - 15.0*dev_var_in[u_offset + IDX(i,jb,k)]
                        +  6.0*dev_var_in[u_offset + IDX(i,jb-1,k)]
                        -      dev_var_in[u_offset + IDX(i,jb-2,k)]
                        );
        }
    output[pp] = (-1.0 / 64.0 / dy) *
                    (
                        -      dev_var_in[u_offset + pp-3*nx]
                        +  6.0*dev_var_in[u_offset + pp-2*nx]
                        - 15.0*dev_var_in[u_offset + pp-nx]
                        + 20.0*dev_var_in[u_offset + pp]
                        - 15.0*dev_var_in[u_offset + pp+nx]
                        +  6.0*dev_var_in[u_offset + pp+2*nx]
                        -      dev_var_in[u_offset + pp+3*nx]
                        );

    if(j==5) {
        int je = ny - 3;
        output[IDX(i,je-1,k)] = (-1.0 / 64.0 / dy) *
                (
                    -      dev_var_in[u_offset + IDX(i,je+1,k)]
                    +  6.0*dev_var_in[u_offset + IDX(i,je,k)]
                    - 15.0*dev_var_in[u_offset + IDX(i,je-1,k)]
                    + 20.0*dev_var_in[u_offset + IDX(i,je-2,k)]
                    - 15.0*dev_var_in[u_offset + IDX(i,je-3,k)]
                    +  6.0*dev_var_in[u_offset + IDX(i,je-4,k)]
                    -      dev_var_in[u_offset + IDX(i,je-5,k)]
                    );                   
    }
}

__device__ void calc_ko_deriv42_z(int id, double * output, double * dev_var_in, int u_offset, double dz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{  
    int nx = host_sz_x;
    int ny = host_sz_y;
    int n = nx * ny;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    
    if(k==4) {
        int kb=3;
        output[IDX(i,j,kb)] = (-1.0 / 64.0 / dz) *
                    (
                        -      dev_var_in[u_offset + IDX(i,j,kb+4)]
                        +  6.0*dev_var_in[u_offset + IDX(i,j,kb+3)]
                        - 15.0*dev_var_in[u_offset + IDX(i,j,kb+2)]
                        + 20.0*dev_var_in[u_offset + IDX(i,j,kb+1)]
                        - 15.0*dev_var_in[u_offset + IDX(i,j,kb)]
                        +  6.0*dev_var_in[u_offset + IDX(i,j,kb-1)]
                        -      dev_var_in[u_offset + IDX(i,j,kb-2)]
                        );
        }

    output[pp] = (-1.0 / 64.0 / dz) *
                (
                    -      dev_var_in[u_offset + pp-3*n]
                    +  6.0*dev_var_in[u_offset + pp-2*n]
                    - 15.0*dev_var_in[u_offset + pp-n]
                    + 20.0*dev_var_in[u_offset + pp]
                    - 15.0*dev_var_in[u_offset + pp+n]
                    +  6.0*dev_var_in[u_offset + pp+2*n]
                    -      dev_var_in[u_offset + pp+3*n]
                    );
    if(k==5) {
        int ke = host_sz_z - 3;
        output[IDX(i,j,ke-1)] = (-1.0 / 64.0 / dz) *
        (
            -      dev_var_in[u_offset + IDX(i,j,ke+1)]
            +  6.0*dev_var_in[u_offset + IDX(i,j,ke)]
            - 15.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
            + 20.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
            - 15.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
            +  6.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
            -      dev_var_in[u_offset + IDX(i,j,ke-5)]
            );               
    }
        
}

// device methods with bflag

__device__ void calc_deriv42_x_bflag(int id, double * output, double * dev_var_in, const int u_offset, double dx, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idx = 1.0/dx;
    const double idx_by_2 = 0.50 * idx;
    const double idx_by_12 = idx / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-2)) + 1;
    int k = (id/(host_sz_z-2)/(host_sz_x-6)) + 1; 
    if (k>=host_sz_z-1) return;
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[(u_offset) + pp - 2] - 8.0*dev_var_in[(u_offset)
                    + pp - 1] + 8.0*dev_var_in[(u_offset) + pp + 1] 
                    - dev_var_in[(u_offset) + pp + 2] )*idx_by_12;

    if ((bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
        int pp3 = IDX(3, j, k);
        int pp4 = IDX(4, j, k);
        int pp5 = IDX(5, j, k);
        output[pp3] = ((-3)*dev_var_in[(u_offset) + pp3] + 4*dev_var_in[(u_offset) + pp4] - dev_var_in[(u_offset) + pp5]) * idx_by_2;
        output[pp4] = (dev_var_in[(u_offset) + pp5] - dev_var_in[(u_offset) + pp3]) * idx_by_2;
    }

    if ((bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
        int pp2 = IDX(nx-5, j, k);
        int pp3 = IDX(nx-6, j, k);
        int pp1 = IDX(nx-4,j,k);
        output[pp2] = (dev_var_in[(u_offset) + pp1] - dev_var_in[(u_offset) + pp3]) * idx_by_2;
        output[pp1] = (dev_var_in[(u_offset) + pp3]- 4.0 * dev_var_in[(u_offset) + pp2]+ 3.0 * dev_var_in[(u_offset) + pp1]) * idx_by_2;
    }
}

__device__ void calc_deriv42_y_bflag(int id, double* output, double * dev_var_in, const int u_offset, double dy, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idy = 1.0/dy;
    const double idy_by_2 = 0.50 * idy;
    const double idy_by_12 = idy / 12.0;
    
    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 1;
    if (k>=host_sz_z-1) return;
    int pp = IDX(i, j, k);
    
    output[pp] = (dev_var_in[u_offset + pp - 2*nx] 
                - 8.0*dev_var_in[u_offset + pp - nx] 
                + 8.0*dev_var_in[u_offset + pp + nx] 
                - dev_var_in[u_offset + pp + 2*nx] )*idy_by_12;
    
    if ((bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
        int pp3 = IDX(i, 3, k);
        int pp4 = IDX(i, 4, k);
        int pp5 = IDX(i, 5, k);
        output[pp3] = ((-3)*dev_var_in[(u_offset) + pp3] +  4*dev_var_in[(u_offset) + pp4] - dev_var_in[(u_offset) + pp5]) * idy_by_2;
        output[pp4] = (dev_var_in[(u_offset) + pp5] - dev_var_in[(u_offset) + pp3]) * idy_by_2;
    }

    if ((bflag & (1u<<OCT_DIR_UP)) && j==4)  {
        int pp2 = IDX(i, ny-5, k); // IDX(i,je-2,k)
        int pp3 = IDX(i, ny-6, k); // IDX(i,je-3,k)
        int pp1 = IDX(i, ny-4, k); // IDX(i,je-1,k)
        output[pp2] = (dev_var_in[(u_offset) + pp1] - dev_var_in[(u_offset) + pp3]) * idy_by_2;
        output[pp1] = (dev_var_in[(u_offset) + pp3]- 4.0 * dev_var_in[(u_offset) + pp2] + 3.0 * dev_var_in[(u_offset) + pp1]) * idy_by_2;
    }
    
}

__device__ void calc_deriv42_z_bflag(int id, double* output, double * dev_var_in, const int u_offset, double dz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 
    int n = nx * ny;

    const double idz = 1.0/dz;
    const double idz_by_2 = 0.50 * idz;
    const double idz_by_12 = idz / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = (dev_var_in[(u_offset) + pp - 2*n] - 8.0*dev_var_in[(u_offset) + pp - n] 
                    + 8.0*dev_var_in[(u_offset) + pp + n] - dev_var_in[(u_offset) + pp + 2*n]) 
                    * idz_by_12;
            
    if ((bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
        int pp3 = IDX(i, j, 3); 
        int pp4 = IDX(i, j, 4); 
        int pp5 = IDX(i, j, 5);
        output[pp3] = ((-3)*dev_var_in[(u_offset) + pp3] + 4*dev_var_in[(u_offset) + pp4] - dev_var_in[(u_offset) + pp5]) * idz_by_2;
        output[pp4] = (dev_var_in[(u_offset) + pp5] - dev_var_in[(u_offset) + pp3]) * idz_by_2;
    }
                
    if ((bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
        int pp2 = IDX(i, j, host_sz_z-5); 
        int pp3 = IDX(i, j, host_sz_z-6); 
        int pp1 = IDX(i, j, host_sz_z-4);
        output[pp2] = (dev_var_in[(u_offset) + pp1] - dev_var_in[(u_offset) + pp3]) * idz_by_2;
        output[pp1] = (dev_var_in[(u_offset) + pp3]- 4.0 * dev_var_in[(u_offset) + pp2] + 3.0 * dev_var_in[(u_offset) + pp1]) * idz_by_2;
    }
}

__device__ void calc_deriv42_xx_bflag(int id, double* output, double * dev_var_in, const int u_offset, double dx, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idx_sqrd = 1.0/(dx*dx);
    const double idx_sqrd_by_12 = idx_sqrd / 12.0;
    
    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2] 
                + 16.0*dev_var_in[(u_offset) + pp - 1] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + 1] 
                - dev_var_in[(u_offset) + pp + 2] 
            )*idx_sqrd_by_12;

    if ((bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
        int pp3 = IDX(3, j, k); 
        int pp4 = IDX(4, j, k); 
        int pp5 = IDX(5, j, k); 
        int pp6 = IDX(6, j, k); 
    
        output[pp3] = (
                2.0     *       dev_var_in[(u_offset) + pp3] 
            -   5.0     *       dev_var_in[(u_offset) + pp4] 
            +   4.0     *       dev_var_in[(u_offset) + pp5] 
            -                   dev_var_in[(u_offset) + pp6]
            )*idx_sqrd;
    
        output[pp4] = (
                            dev_var_in[(u_offset) + pp3]
            -   2.0     *   dev_var_in[(u_offset) + pp4]
            +               dev_var_in[(u_offset) + pp5]
        )*idx_sqrd;
    }
                        
    if ((bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
        int pp1 = IDX(host_sz_x - 4, j, k); // IDX(ie-1,j,k)
        int pp2 = IDX(host_sz_x - 5, j, k); // IDX(ie-2,j,k)
        int pp3 = IDX(host_sz_x - 6, j, k); // IDX(ie-3,j,k)
        int pp4 = IDX(host_sz_x - 7, j, k); // IDX(ie-4,j,k)

        output[pp2] = (
                                dev_var_in[(u_offset) + pp3] 
                -   2.0     *   dev_var_in[(u_offset) + pp2] 
                +               dev_var_in[(u_offset) + pp1] 
                )*idx_sqrd;


        output[pp1] = (
            -   1.0 *   dev_var_in[(u_offset) + pp4] 
            +   4.0 *   dev_var_in[(u_offset) + pp3] 
            -   5.0 *   dev_var_in[(u_offset) + pp2] 
            +   2.0 *   dev_var_in[(u_offset) + pp1]
            )*idx_sqrd;
    }

}

__device__ void calc_deriv42_yy_bflag(int id, double* output, double * dev_var_in, const int u_offset, double dy, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 

    const double idy_sqrd = 1.0/(dy*dy);
    const double idy_sqrd_by_12 = idy_sqrd / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2*nx] 
                + 16.0*dev_var_in[(u_offset) + pp - nx] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + nx] 
                - dev_var_in[(u_offset) + pp + 2*nx] 
            )*idy_sqrd_by_12;

    if ((bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
        int pp3 = IDX(i, 3, k); 
        int pp4 = IDX(i, 4, k); 
        int pp5 = IDX(i, 5, k); 
        int pp6 = IDX(i, 6, k); 
    
        output[pp3] = (
                2.0 *   dev_var_in[(u_offset) + pp3] 
            -   5.0 *   dev_var_in[(u_offset) + pp4] 
            +   4.0 *   dev_var_in[(u_offset) + pp5] 
            -           dev_var_in[(u_offset) + pp6]
            ) * idy_sqrd;
    
        output[pp4] = (
                        dev_var_in[(u_offset) + pp3]
            -   2.0 *   dev_var_in[(u_offset) + pp4]
            +           dev_var_in[(u_offset) + pp5]
        ) * idy_sqrd;
    }
                            
    if ((bflag & (1u<<OCT_DIR_UP)) && j==4)  {
        int pp1 = IDX(i, host_sz_y - 4, k); 
        int pp2 = IDX(i, host_sz_y - 5, k); 
        int pp3 = IDX(i, host_sz_y - 6, k); 
        int pp4 = IDX(i, host_sz_y - 7, k); 
    
        output[pp2] = (
                        dev_var_in[(u_offset) + pp3] 
            -   2.0 *   dev_var_in[(u_offset) + pp2] 
            +           dev_var_in[(u_offset) + pp1] 
            ) * idy_sqrd;
    
    
        output[pp1] = (
            -   1.0 *   dev_var_in[(u_offset) + pp4] 
            +   4.0 *   dev_var_in[(u_offset) + pp3] 
            -   5.0 *   dev_var_in[(u_offset) + pp2] 
            +   2.0 *   dev_var_in[(u_offset) + pp1]
            ) * idy_sqrd;
    
    }
            
}

__device__ void calc_deriv42_zz_bflag(int id, double* output, double * dev_var_in, const int u_offset, double dz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x; 
    int ny = host_sz_y; 
    int n = nx * ny;

    const double idz_sqrd = 1.0/(dz*dz);
    const double idz_sqrd_by_12 = idz_sqrd / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    output[pp] = ((-1)*dev_var_in[(u_offset) + pp - 2*n] 
                + 16.0*dev_var_in[(u_offset) + pp - n] 
                - 30.0*dev_var_in[(u_offset) + pp] 
                + 16.0*dev_var_in[(u_offset) + pp + n] 
                - dev_var_in[(u_offset) + pp + 2*n] 
            )*idz_sqrd_by_12;
    
    if ((bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
        int pp3 = IDX(i, j, 3); 
        int pp4 = IDX(i, j, 4); 
        int pp5 = IDX(i, j, 5); 
        int pp6 = IDX(i, j, 6); 
    
        output[pp3] = (
                2.0 *   dev_var_in[(u_offset) + pp3] 
            -   5.0 *   dev_var_in[(u_offset) + pp4] 
            +   4.0 *   dev_var_in[(u_offset) + pp5] 
            -           dev_var_in[(u_offset) + pp6]
            ) * idz_sqrd;
    
        output[pp4] = (
                        dev_var_in[(u_offset) + pp3]
            -   2.0 *   dev_var_in[(u_offset) + pp4]
            +           dev_var_in[(u_offset) + pp5]
        ) * idz_sqrd;
    }
                                        
    if ((bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
        int pp1 = IDX(i, j, host_sz_z - 4); 
        int pp2 = IDX(i, j, host_sz_z - 5); 
        int pp3 = IDX(i, j, host_sz_z - 6); 
        int pp4 = IDX(i, j, host_sz_z - 7); 

        output[pp2] = (
                            dev_var_in[(u_offset) + pp3] 
                -   2.0 *   dev_var_in[(u_offset) + pp2] 
                +           dev_var_in[(u_offset) + pp1] 
                ) * idz_sqrd;


        output[pp1] = (
            -   1.0 *   dev_var_in[(u_offset) + pp4] 
            +   4.0 *   dev_var_in[(u_offset) + pp3] 
            -   5.0 *   dev_var_in[(u_offset) + pp2] 
            +   2.0 *   dev_var_in[(u_offset) + pp1]
            ) * idz_sqrd;
    }
}

__device__ void calc_deriv42_adv_x_bflag(int id, double * output, double * dev_var_in, int u_offset, double dx, int betax, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) 
{
    int nx = host_sz_x;
    int ny = host_sz_y;

    const double idx = 1.0/dx;
    const double idx_by_2 = 0.50 * idx;
    const double idx_by_12 = idx / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betax + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - 1]
                    - 10.0 * dev_var_in[u_offset + pp]
                    + 18.0 * dev_var_in[u_offset + pp + 1]
                    -  6.0 * dev_var_in[u_offset + pp + 2]
                    +        dev_var_in[u_offset + pp + 3]
                ) * idx_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3]
                    +  6.0 * dev_var_in[u_offset + pp - 2]
                    - 18.0 * dev_var_in[u_offset + pp - 1]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +1]
                ) * idx_by_12;
    }

    if ((bflag & (1u<<OCT_DIR_LEFT)) && (i == 3)) {
        output[IDX(3,j,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(3,j,k)]
                +  4.0 * dev_var_in[u_offset + IDX(4,j,k)]
                -        dev_var_in[u_offset + IDX(5,j,k)]
                ) * idx_by_2;

        if (dev_var_in[betax + IDX(4,j,k)] > 0.0) {
            output[IDX(4,j,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(4,j,k)]
                            +  4.0 * dev_var_in[u_offset + IDX(5,j,k)]
                            -        dev_var_in[u_offset + IDX(6,j,k)]
                        ) * idx_by_2;
        } else {
            output[IDX(4,j,k)] = ( -         dev_var_in[u_offset + IDX(3,j,k)]
                            +        dev_var_in[u_offset + IDX(5,j,k)]
                        ) * idx_by_2;
        }

        if (dev_var_in[betax + IDX(5,j,k)] > 0.0 ) {
            output[IDX(5,j,k)] = (-  3.0 * dev_var_in[u_offset + IDX(4,j,k)]
                        - 10.0 * dev_var_in[u_offset + IDX(5,j,k)]
                        + 18.0 * dev_var_in[u_offset + IDX(6,j,k)]
                        -  6.0 * dev_var_in[u_offset + IDX(7,j,k)]
                        +        dev_var_in[u_offset + IDX(8,j,k)]
                        ) * idx_by_12;
        } else {
            output[IDX(5,j,k)] = (           dev_var_in[u_offset + IDX(3,j,k)]
                            -  4.0 * dev_var_in[u_offset + IDX(4,j,k)]
                            +  3.0 * dev_var_in[u_offset + IDX(5,j,k)]
                        ) * idx_by_2;
        }
    }

    if ((bflag & (1u<<OCT_DIR_RIGHT)) && (i == 4)) {
        const int ie = nx - 3;
        if ( dev_var_in[betax + IDX(ie-3,j,k)] < 0.0 ) {
            output[IDX(ie-3,j,k)] = (  - 3.0 * dev_var_in[u_offset + IDX(ie-3,j,k)]
                                    + 4.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
                                    -       dev_var_in[u_offset + IDX(ie-1,j,k)]
                                ) * idx_by_2;
        } else {
            output[IDX(ie-3,j,k)] = ( -   dev_var_in[u_offset + IDX(ie-6,j,k)]
                            +  6.0 * dev_var_in[u_offset + IDX(ie-5,j,k)]
                            - 18.0 * dev_var_in[u_offset + IDX(ie-4,j,k)]
                            + 10.0 * dev_var_in[u_offset + IDX(ie-3  ,j,k)]
                            +  3.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
                            ) * idx_by_12;
        }

        if (dev_var_in[betax + IDX(ie-2,j,k)] > 0.0 ) {
            output[IDX(ie-2,j,k)] = (  -  dev_var_in[u_offset + IDX(ie-3,j,k)]
                                    +  dev_var_in[u_offset + IDX(ie-1,j,k)]
                                ) * idx_by_2;
        } else {
            output[IDX(ie-2,j,k)] = (     dev_var_in[u_offset + IDX(ie-4,j,k)]
                            - 4.0 * dev_var_in[u_offset + IDX(ie-3,j,k)]
                            + 3.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
                                ) * idx_by_2;
        }

        output[IDX(ie-1,j,k)] = (          dev_var_in[u_offset + IDX(ie-3,j,k)]
                                - 4.0 * dev_var_in[u_offset + IDX(ie-2,j,k)]
                                + 3.0 * dev_var_in[u_offset + IDX(ie-1,j,k)]
                            ) * idx_by_2;
    }
}

__device__ void calc_deriv42_adv_y_bflag(int id, double * output, double * dev_var_in, int u_offset, double dy, int betay, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) 
{
    int nx = host_sz_x;
    int ny = host_sz_y;

    const double idy = 1.0/dy;
    const double idy_by_2 = 0.50 * idy;
    const double idy_by_12 = idy / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betay + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - nx]
                        - 10.0 * dev_var_in[u_offset + pp]
                        + 18.0 * dev_var_in[u_offset + pp + nx]
                        -  6.0 * dev_var_in[u_offset + pp + 2*nx]
                        +        dev_var_in[u_offset + pp + 3*nx]
                    ) * idy_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3*nx]
                    +  6.0 * dev_var_in[u_offset + pp - 2*nx]
                    - 18.0 * dev_var_in[u_offset + pp - nx]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +nx]
                    ) * idy_by_12;
                
    }

    if ((bflag & (1u<<OCT_DIR_DOWN)) && (j == 3)) {
            
        output[IDX(i,3,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,3,k)]
                +  4.0 * dev_var_in[u_offset + IDX(i,4,k)]
                -        dev_var_in[u_offset + IDX(i,5,k)]
                ) * idy_by_2;
                
        if (dev_var_in[betay + IDX(i,4,k)] > 0.0) {
            output[IDX(i,4,k)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,4,k)]
                            +  4.0 * dev_var_in[u_offset + IDX(i,5,k)]
                            -        dev_var_in[u_offset + IDX(i,6,k)]
                        ) * idy_by_2;

        }
        else {
            output[IDX(i,4,k)] = ( -         dev_var_in[u_offset + IDX(i,3,k)]
                            +        dev_var_in[u_offset + IDX(i,5,k)]
                        ) * idy_by_2;
                        
        }

        if (dev_var_in[betay + IDX(i,5,k)] > 0.0 ) {
            output[IDX(i,5,k)] = (-  3.0 * dev_var_in[u_offset + IDX(i,4,k)]
                        - 10.0 * dev_var_in[u_offset + IDX(i,5,k)]
                        + 18.0 * dev_var_in[u_offset + IDX(i,6,k)]
                        -  6.0 * dev_var_in[u_offset + IDX(i,7,k)]
                        +        dev_var_in[u_offset + IDX(i,8,k)]
                        ) * idy_by_12;
        }
        else {
            output[IDX(i,5,k)] = (           dev_var_in[u_offset + IDX(i,3,k)]
                            -  4.0 * dev_var_in[u_offset + IDX(i,4,k)]
                            +  3.0 * dev_var_in[u_offset + IDX(i,5,k)]
                        ) * idy_by_2;
        }
    }

    if ((bflag & (1u<<OCT_DIR_UP)) && (j == 4)) {
        
        const int je = host_sz_y - 3;
        
        if ( dev_var_in[betay + IDX(i,je-3,k)] < 0.0 ) {
            output[IDX(i,je-3,k)] = (  - 3.0 * dev_var_in[u_offset + IDX(i,je-3,k)]
                                    + 4.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
                                    -       dev_var_in[u_offset + IDX(i,je-1,k)]
                                    ) * idy_by_2;
        }
        else {
            output[IDX(i,je-3,k)] = ( -   dev_var_in[u_offset + IDX(i,je-6,k)]
                                +  6.0 * dev_var_in[u_offset + IDX(i,je-5,k)]
                                - 18.0 * dev_var_in[u_offset + IDX(i,je-4,k)]
                                + 10.0 * dev_var_in[u_offset + IDX(i,je-3,k)]
                                +  3.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
                            ) * idy_by_12;
        }
    
            if (dev_var_in[betay + IDX(i,je-2,k)] > 0.0 ) {
            output[IDX(i,je-2,k)] = (  -  dev_var_in[u_offset + IDX(i,je-3,k)]
                                    +  dev_var_in[u_offset + IDX(i,je-1,k)]
                                    ) * idy_by_2;
            }
            else {
            output[IDX(i,je-2,k)] = (     dev_var_in[u_offset + IDX(i,je-4,k)]
                                - 4.0 * dev_var_in[u_offset + IDX(i,je-3,k)]
                                + 3.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
                                    ) * idy_by_2;
            }
    
            output[IDX(i,je-1,k)]  = (          dev_var_in[u_offset + IDX(i,je-3,k)]
                                    - 4.0 * dev_var_in[u_offset + IDX(i,je-2,k)]
                                    + 3.0 * dev_var_in[u_offset + IDX(i,je-1,k)]
                                ) * idy_by_2;
    }
}

__device__ void calc_deriv42_adv_z_bflag(int id, double * output, double * dev_var_in, int u_offset, double dz, int betaz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) 
{
    int nx = host_sz_x;
    int ny = host_sz_y;
    int n = nx * ny;
    
    const double idz = 1.0/dz;
    const double idz_by_2 = 0.50 * idz;
    const double idz_by_12 = idz / 12.0;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if (dev_var_in[betaz + pp] > 0.0 ) {
            output[pp] = ( -  3.0 * dev_var_in[u_offset + pp - n]
                        - 10.0 * dev_var_in[u_offset + pp]
                        + 18.0 * dev_var_in[u_offset + pp + n]
                        -  6.0 * dev_var_in[u_offset + pp + 2*n]
                        +        dev_var_in[u_offset + pp + 3*n]
                    ) * idz_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[u_offset + pp - 3*n]
                    +  6.0 * dev_var_in[u_offset + pp - 2*n]
                    - 18.0 * dev_var_in[u_offset + pp - n]
                    + 10.0 * dev_var_in[u_offset + pp]
                    +  3.0 * dev_var_in[u_offset + pp +n]
                    ) * idz_by_12;
                
    }

    if ((bflag & (1u<<OCT_DIR_BACK)) && (k == 3)) {
            
        output[IDX(i,j,3)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,j,3)]
                +  4.0 * dev_var_in[u_offset + IDX(i,j,4)]
                -        dev_var_in[u_offset + IDX(i,j,5)]
                ) * idz_by_2;
                
        if (dev_var_in[betaz + IDX(i,j,4)] > 0.0) {
            output[IDX(i,j,4)] = ( -  3.0 * dev_var_in[u_offset + IDX(i,j,4)]
                            +  4.0 * dev_var_in[u_offset + IDX(i,j,5)]
                            -        dev_var_in[u_offset + IDX(i,j,6)]
                        ) * idz_by_2;

        }
        else {
            output[IDX(i,j,4)] = ( -         dev_var_in[u_offset + IDX(i,j,3)]
                            +        dev_var_in[u_offset + IDX(i,j,5)]
                        ) * idz_by_2;
                        
        }

        if (dev_var_in[betaz + IDX(i,j,5)] > 0.0 ) {
            output[IDX(i,j,5)] = (-  3.0 * dev_var_in[u_offset + IDX(i,j,4)]
                        - 10.0 * dev_var_in[u_offset + IDX(i,j,5)]
                        + 18.0 * dev_var_in[u_offset + IDX(i,j,6)]
                        -  6.0 * dev_var_in[u_offset + IDX(i,j,7)]
                        +        dev_var_in[u_offset + IDX(i,j,8)]
                        ) * idz_by_12;
        }
        else {
            output[IDX(i,j,5)] = (           dev_var_in[u_offset + IDX(i,j,3)]
                            -  4.0 * dev_var_in[u_offset + IDX(i,j,4)]
                            +  3.0 * dev_var_in[u_offset + IDX(i,j,5)]
                        ) * idz_by_2;
        }
    }

    if ((bflag & (1u<<OCT_DIR_FRONT)) && (k == 4)) {
        
        const int ke = host_sz_z - 3; // Here I changed
        
        if ( dev_var_in[betaz + IDX(i,j,ke-3)] < 0.0 ) {
            output[IDX(i,j,ke-3)] = (  - 3.0 * dev_var_in[u_offset + IDX(i,j,ke-3)]
                                    + 4.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
                                    -       dev_var_in[u_offset + IDX(i,j,ke-1)]
                                    ) * idz_by_2;
        }
        else {
            output[IDX(i,j,ke-3)] = ( -   dev_var_in[u_offset + IDX(i,j,ke-6)]
                                +  6.0 * dev_var_in[u_offset + IDX(i,j,ke-5)]
                                - 18.0 * dev_var_in[u_offset + IDX(i,j,ke-4)]
                                + 10.0 * dev_var_in[u_offset + IDX(i,j,ke-3)]
                                +  3.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
                            ) * idz_by_12;
        }
    
            if (dev_var_in[betaz + IDX(i,j,ke-2)] > 0.0 ) {
            output[IDX(i,j,ke-2)] = (  -  dev_var_in[u_offset + IDX(i,j,ke-3)]
                                    +  dev_var_in[u_offset + IDX(i,j,ke-1)]
                                    ) * idz_by_2;
            }
            else {
            output[IDX(i,j,ke-2)] = (     dev_var_in[u_offset + IDX(i,j,ke-4)]
                                - 4.0 * dev_var_in[u_offset + IDX(i,j,ke-3)]
                                + 3.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
                                    ) * idz_by_2;
            }
    
            output[IDX(i,j,ke-1)]  = (          dev_var_in[u_offset + IDX(i,j,ke-3)]
                                    - 4.0 * dev_var_in[u_offset + IDX(i,j,ke-2)]
                                    + 3.0 * dev_var_in[u_offset + IDX(i,j,ke-1)]
                                ) * idz_by_2;
    }
}


__device__ void calc_ko_deriv42_x_bflag(int id, double * output, double * dev_var_in, int u_offset, double dx, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x;
    int ny = host_sz_y;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    
    if(i==4) {
        int ib=3;
        output[IDX(3, j, k)] = (-1.0 / 64.0 / dx) *
                        (
                        -      dev_var_in[u_offset + IDX(ib+4,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ib+3,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ib+2,j,k)]
                        + 20.0*dev_var_in[u_offset + IDX(ib+1,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ib,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ib-1,j,k)]
                        -      dev_var_in[u_offset + IDX(ib-2,j,k)]
                        );
    }

    output[pp] = (-1.0 / 64.0 / dx) *
                            (
                            -      dev_var_in[u_offset + pp - 3]
                            +  6.0*dev_var_in[u_offset + pp - 2]
                            - 15.0*dev_var_in[u_offset + pp - 1]
                            + 20.0*dev_var_in[u_offset + pp ]
                            - 15.0*dev_var_in[u_offset + pp + 1]
                            +  6.0*dev_var_in[u_offset + pp + 2]
                            -      dev_var_in[u_offset + pp + 3]
                            );

    if(i==5) {
        int ie = nx-3;
        output[IDX(ie-1, j, k)] = (-1.0 / 64.0 / dx) *
                        (
                        -      dev_var_in[u_offset + IDX(ie+1,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ie,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
                        + 20.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
                        - 15.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
                        +  6.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
                        -      dev_var_in[u_offset + IDX(ie-5,j,k)]
                        );
    }

    if ((bflag & (1u<<OCT_DIR_LEFT)) && (i == 4)) {

        output[IDX(3,j,k)] =  (      dev_var_in[u_offset + IDX(6,j,k)]
                                    - 3.0*dev_var_in[u_offset + IDX(5,j,k)]
                                    + 3.0*dev_var_in[u_offset + IDX(4,j,k)]
                                    -     dev_var_in[u_offset + IDX(3,j,k)]
                                )/59.0/48.0*64*dx;
        output[IDX(4,j,k)] =  (     dev_var_in[u_offset + IDX(7,j,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(6,j,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(5,j,k)]
                                    - 10.0*dev_var_in[u_offset + IDX(4,j,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(3,j,k)]
                                    )/43.0/48.0*64*dx;
        output[IDX(5,j,k)] =  (     dev_var_in[u_offset + IDX(8,j,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(7,j,k)]
                                    + 15.0*dev_var_in[u_offset + IDX(6,j,k)]
                                    - 19.0*dev_var_in[u_offset + IDX(5,j,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(4,j,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(3,j,k)]
                                    )/49.0/48.0*64*dx;
        }

    if ((bflag & (1u<<OCT_DIR_RIGHT)) && (i == 5)) {
        
        const int ie = nx - 3;
        output[IDX(ie-3,j,k)] = ( dev_var_in[u_offset + IDX(ie-6,j,k)]
                                    - 6.0*dev_var_in[u_offset + IDX(ie-5,j,k)]
                                    + 15.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
                                    - 19.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
                                    )/49.0/48.0*64*dx;
            
            output[IDX(ie-2,j,k)] =  ( dev_var_in[u_offset + IDX(ie-5,j,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(ie-4,j,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
                                    - 10.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(ie-1,j,k)]
                                    )/43.0/48.0*64*dx;
        
    
            output[IDX(ie-1,j,k)] = ( dev_var_in[u_offset + IDX(ie-4,j,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(ie-3,j,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(ie-2,j,k)]
                                    -      dev_var_in[u_offset + IDX(ie-1,j,k)]
                                    )/59.0/48.0*64*dx;
    }
}

__device__ void calc_ko_deriv42_y_bflag(int id, double * output, double * dev_var_in, int u_offset, double dy, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{
    int nx = host_sz_x;
    int ny = host_sz_y;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);

    if(j==4) {
        int jb=3;
        output[IDX(i,jb,k)] = (-1.0 / 64.0 / dy) *
                    (
                        -      dev_var_in[u_offset + IDX(i,jb+4,k)]
                        +  6.0*dev_var_in[u_offset + IDX(i,jb+3,k)]
                        - 15.0*dev_var_in[u_offset + IDX(i,jb+2,k)]
                        + 20.0*dev_var_in[u_offset + IDX(i,jb+1,k)]
                        - 15.0*dev_var_in[u_offset + IDX(i,jb,k)]
                        +  6.0*dev_var_in[u_offset + IDX(i,jb-1,k)]
                        -      dev_var_in[u_offset + IDX(i,jb-2,k)]
                        );
        }
    output[pp] = (-1.0 / 64.0 / dy) *
                    (
                        -      dev_var_in[u_offset + pp-3*nx]
                        +  6.0*dev_var_in[u_offset + pp-2*nx]
                        - 15.0*dev_var_in[u_offset + pp-nx]
                        + 20.0*dev_var_in[u_offset + pp]
                        - 15.0*dev_var_in[u_offset + pp+nx]
                        +  6.0*dev_var_in[u_offset + pp+2*nx]
                        -      dev_var_in[u_offset + pp+3*nx]
                        );

    if(j==5) {
        int je = ny - 3;
        output[IDX(i,je-1,k)] = (-1.0 / 64.0 / dy) *
                (
                    -      dev_var_in[u_offset + IDX(i,je+1,k)]
                    +  6.0*dev_var_in[u_offset + IDX(i,je,k)]
                    - 15.0*dev_var_in[u_offset + IDX(i,je-1,k)]
                    + 20.0*dev_var_in[u_offset + IDX(i,je-2,k)]
                    - 15.0*dev_var_in[u_offset + IDX(i,je-3,k)]
                    +  6.0*dev_var_in[u_offset + IDX(i,je-4,k)]
                    -      dev_var_in[u_offset + IDX(i,je-5,k)]
                    );                   
    }

    if ((bflag & (1u<<OCT_DIR_DOWN)) && (j == 4)) {

        output[IDX(i,3,k)] =  (      dev_var_in[u_offset +IDX(i,6,k)]
                                    - 3.0*dev_var_in[u_offset +IDX(i,5,k)]
                                    + 3.0*dev_var_in[u_offset + IDX(i,4,k)]
                                    -     dev_var_in[u_offset + IDX(i,3,k)]
                                )/59.0/48.0*64*dy;
        output[IDX(i,4,k)] =  (     dev_var_in[u_offset + IDX(i,7,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,6,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,5,k)]
                                    - 10.0*dev_var_in[u_offset + IDX(i,4,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(i,3,k)]
                                    )/43.0/48.0*64*dy;
        output[IDX(i,5,k)] =  (     dev_var_in[u_offset + IDX(i,8,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,7,k)]
                                    + 15.0*dev_var_in[u_offset + IDX(i,6,k)]
                                    - 19.0*dev_var_in[u_offset + IDX(i,5,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,4,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(i,3,k)]
                                    )/49.0/48.0*64*dy;
        }

    if ((bflag & (1u<<OCT_DIR_UP)) && (j == 5)) {
        
        const int je = ny - 3;
        output[IDX(i,je-3,k)] = (dev_var_in[u_offset + IDX(i,je-6,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,je-5,k)]
                                    + 15.0*dev_var_in[u_offset + IDX(i,je-4,k)]
                                    - 19.0*dev_var_in[u_offset + IDX(i,je-3,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,je-2,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(i,je-1,k)]
                                    )/49.0/48.0*64*dy;
            
            output[IDX(i,je-2,k)] = (dev_var_in[u_offset + IDX(i,je-5,k)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,je-4,k)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,je-3,k)]
                                    - 10.0*dev_var_in[u_offset + IDX(i,je-2,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(i,je-1,k)]
                                    )/43.0/48.0*64*dy;
        
    
            output[IDX(i,je-1,k)] = ( dev_var_in[u_offset + IDX(i,je-4,k)]
                                    -  3.0*dev_var_in[u_offset + IDX(i,je-3,k)]
                                    +  3.0*dev_var_in[u_offset + IDX(i,je-2,k)]
                                    -      dev_var_in[u_offset + IDX(i,je-1,k)]
                                    )/59.0/48.0*64*dy;
    }
}

__device__ void calc_ko_deriv42_z_bflag(int id, double * output, double * dev_var_in, int u_offset, double dz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag)
{  
    int nx = host_sz_x;
    int ny = host_sz_y;
    int n = nx * ny;

    int i = id%(host_sz_x-6) + 3;
    int j = ((id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (id/(host_sz_z-6)/(host_sz_x-6)) + 3;
    if (k>=host_sz_z-3) return;
    int pp = IDX(i, j, k);
    
    if(k==4) {
        int kb=3;
        output[IDX(i,j,kb)] = (-1.0 / 64.0 / dz) *
                    (
                        -      dev_var_in[u_offset + IDX(i,j,kb+4)]
                        +  6.0*dev_var_in[u_offset + IDX(i,j,kb+3)]
                        - 15.0*dev_var_in[u_offset + IDX(i,j,kb+2)]
                        + 20.0*dev_var_in[u_offset + IDX(i,j,kb+1)]
                        - 15.0*dev_var_in[u_offset + IDX(i,j,kb)]
                        +  6.0*dev_var_in[u_offset + IDX(i,j,kb-1)]
                        -      dev_var_in[u_offset + IDX(i,j,kb-2)]
                        );
        }

    output[pp] = (-1.0 / 64.0 / dz) *
                (
                    -      dev_var_in[u_offset + pp-3*n]
                    +  6.0*dev_var_in[u_offset + pp-2*n]
                    - 15.0*dev_var_in[u_offset + pp-n]
                    + 20.0*dev_var_in[u_offset + pp]
                    - 15.0*dev_var_in[u_offset + pp+n]
                    +  6.0*dev_var_in[u_offset + pp+2*n]
                    -      dev_var_in[u_offset + pp+3*n]
                    );
    if(k==5) {
        int ke = host_sz_z - 3;
        output[IDX(i,j,ke-1)] = (-1.0 / 64.0 / dz) *
        (
            -      dev_var_in[u_offset + IDX(i,j,ke+1)]
            +  6.0*dev_var_in[u_offset + IDX(i,j,ke)]
            - 15.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
            + 20.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
            - 15.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
            +  6.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
            -      dev_var_in[u_offset + IDX(i,j,ke-5)]
            );               
    }

    if ((bflag & (1u<<OCT_DIR_BACK)) && (k == 4)) {

        output[IDX(i,3,k)] =  (      dev_var_in[u_offset +IDX(i,k,6)]
                                    - 3.0*dev_var_in[u_offset +IDX(i,k,5)]
                                    + 3.0*dev_var_in[u_offset + IDX(i,k,4)]
                                    -     dev_var_in[u_offset + IDX(i,k,3)]
                                )/59.0/48.0*64*dz;

        output[IDX(i,j,4)] =  (     dev_var_in[u_offset + IDX(i,j,7)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,j,6)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,j,5)]
                                    - 10.0*dev_var_in[u_offset + IDX(i,j,4)]
                                    +  3.0*dev_var_in[u_offset + IDX(i,j,3)]
                                    )/43.0/48.0*64*dz;

        output[IDX(i,j,5)] =  (     dev_var_in[u_offset + IDX(i,j,8)]
                                    -  6.0*dev_var_in[u_offset + IDX(i,j,7)]
                                    + 15.0*dev_var_in[u_offset + IDX(i,j,6)]
                                    - 19.0*dev_var_in[u_offset + IDX(i,j,5)]
                                    + 12.0*dev_var_in[u_offset + IDX(i,j,4)]
                                    -  3.0*dev_var_in[u_offset + IDX(i,j,3)]
                                    )/49.0/48.0*64*dz;
        }

    if ((bflag & (1u<<OCT_DIR_FRONT)) && (k == 5)) {
        
        const int ke = host_sz_z - 3;
        output[IDX(i,j,ke-3)] = (    dev_var_in[u_offset + IDX(i,j,ke-6)]
                                        -  6.0*dev_var_in[u_offset + IDX(i,j,ke-5)]
                                        + 15.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
                                        - 19.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
                                        + 12.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
                                        -  3.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
                                        )/49.0/48.0*64*dz;
            
            output[IDX(i,j,ke-2)] = (   dev_var_in[u_offset + IDX(i,j,ke-5)]
                                        -  6.0*dev_var_in[u_offset + IDX(i,j,ke-4)]
                                        + 12.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
                                        - 10.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
                                        +  3.0*dev_var_in[u_offset + IDX(i,j,ke-1)]
                                        )/43.0/48.0*64*dz;
        
    
            output[IDX(i,j,ke-1)] = (   dev_var_in[u_offset + IDX(i,j,ke-4)]
                                        -  3.0*dev_var_in[u_offset + IDX(i,j,ke-3)]
                                        +  3.0*dev_var_in[u_offset + IDX(i,j,ke-2)]
                                        -      dev_var_in[u_offset + IDX(i,j,ke-1)]
                                        )/59.0/48.0*64*dz;
    }
        
}
