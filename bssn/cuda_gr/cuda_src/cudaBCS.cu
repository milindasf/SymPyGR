#include "hip/hip_runtime.h"
/**
 * Created on: Sep 21, 2018
 * 		Author: Akila, Eranga, Eminda, Ruwan
 **/
 
#include "cudaBCS.cuh"

__global__ void cacl_bssn_bcs_x(double * dev_var_out, double * dev_var_in, int u_offset, double * dxf, double * dyf, double * dzf, double pmin_x, double pmin_y, double pmin_z, double pmax_x, double pmax_y, double pmax_z, const double f_falloff, const double f_asymptotic, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) 
{
    int j = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int k = 3 + threadIdx.y + blockIdx.y * blockDim.y;
    int nx = host_sz_x;
    int ny = host_sz_y;
    int nz = host_sz_z;

    if(j >= ny-3 || k >= nz-3) return;

    double inv_r;
    double hx = (pmax_x - pmin_x) / (nx - 1);
    double hy = (pmax_y - pmin_y) / (ny - 1);
    double hz = (pmax_z - pmin_z) / (nz - 1);
    double x, y, z;
    int pp;

    if (bflag & (1u<<OCT_DIR_LEFT)) {
        
        x = pmin_x + 3*hx;
        z = pmin_z + k*hz;
        y = pmin_y + j*hy;
        pp = IDX(3,j,k);
        inv_r = 1.0 / sqrt(x*x + y*y + z*z);

        dev_var_out[u_offset + pp] = -  inv_r * (
                        x * dxf[pp]
                        + y * dyf[pp]
                        + z * dzf[pp]
                        + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
        }
    
    if (bflag & (1u<<OCT_DIR_RIGHT)) {
        x = pmin_x + (nx - 3)*hx;
        z = pmin_z + k*hz;
        y = pmin_y + j*hy;
        pp = IDX((nx - 3),j,k);
        inv_r = 1.0 / sqrt(x*x + y*y + z*z);

        dev_var_out[u_offset + pp] = -  inv_r * (
                        x * dxf[pp]
                    + y * dyf[pp]
                    + z * dzf[pp]
                    + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
    }
}

__global__ void cacl_bssn_bcs_y(double * dev_var_out, double * dev_var_in, int u_offset, double * dxf, double * dyf, double * dzf, double pmin_x, double pmin_y, double pmin_z, double pmax_x, double pmax_y, double pmax_z, const double f_falloff, const double f_asymptotic, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) 
{
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int k = 3 + threadIdx.y + blockIdx.y * blockDim.y;
    int nx = host_sz_x;
    int ny = host_sz_y;
    int nz = host_sz_z;

    if(i >= nx-3 || k >= nz-3) return;

    double inv_r;
    double hx = (pmax_x - pmin_x) / (nx - 1);
    double hy = (pmax_y - pmin_y) / (ny - 1);
    double hz = (pmax_z - pmin_z) / (nz - 1);
    double x, y, z;
    int pp;

    if (bflag & (1u<<OCT_DIR_DOWN)) {
        
        y = pmin_y + 3*hy;
        z = pmin_z + k*hz;
        x = pmin_x + i*hx;
        pp = IDX(i,3,k);
        inv_r = 1.0 / sqrt(x*x + y*y + z*z);

        dev_var_out[u_offset + pp] = -  inv_r * (
                        x * dxf[pp]
                        + y * dyf[pp]
                        + z * dzf[pp]
                        + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
        
        }
    
    if (bflag & (1u<<OCT_DIR_UP)) {
        x = pmin_x + i*hx;
        z = pmin_z + k*hz;
        y = pmin_y + (ny-3)*hy;
        pp = IDX(i,(ny - 3),k);
        inv_r = 1.0 / sqrt(x*x + y*y + z*z);

        dev_var_out[u_offset + pp] = -  inv_r * (
                        x * dxf[pp]
                    + y * dyf[pp]
                    + z * dzf[pp]
                    + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
        
    }
}

__global__ void cacl_bssn_bcs_z(double * dev_var_out, double * dev_var_in, int u_offset, double * dxf, double * dyf, double * dzf, double pmin_x, double pmin_y, double pmin_z, double pmax_x, double pmax_y, double pmax_z, const double f_falloff, const double f_asymptotic, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag) 
{
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
    int nx = host_sz_x;
    int ny = host_sz_y;
    int nz = host_sz_z;

    if(i >= nx-3 || j >= ny-3) return;

    double inv_r;
    double hx = (pmax_x - pmin_x) / (nx - 1);
    double hy = (pmax_y - pmin_y) / (ny - 1);
    double hz = (pmax_z - pmin_z) / (nz - 1);
    double x, y, z;
    int pp;

    if (bflag & (1u<<OCT_DIR_BACK)) {
        
        y = pmin_y + j*hy;
        z = pmin_z + 3*hz;
        x = pmin_x + i*hx;
        pp = IDX(i,j,3);
        inv_r = 1.0 / sqrt(x*x + y*y + z*z);

        dev_var_out[u_offset + pp] = -  inv_r * (
                        x * dxf[pp]
                        + y * dyf[pp]
                        + z * dzf[pp]
                        + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
    
    }
    
    if (bflag & (1u<<OCT_DIR_FRONT)) {
        x = pmin_x + i*hx;
        z = pmin_z + (nz-3)*hz;
        y = pmin_y + j*hy;
        pp = IDX(i,j,3);
        inv_r = 1.0 / sqrt(x*x + y*y + z*z);

        dev_var_out[u_offset + pp] = -  inv_r * (
                            x * dxf[pp]
                        + y * dyf[pp]
                        + z * dzf[pp]
                        + f_falloff * (   dev_var_in[u_offset + pp] - f_asymptotic ) );
        
    }
}


void bssn_bcs(double * dev_var_out, double * dev_var_in, int u_offset, double * dxf, double * dyf, double * dzf, const double * pmin, const double * pmax, const double f_falloff, const double f_asymptotic, const unsigned int * host_sz, int bflag, hipStream_t stream) 
{
    const unsigned int nx = host_sz[0];
    const unsigned int ny = host_sz[1];
    const unsigned int nz = host_sz[2];

    const int ie = nx - 3;//x direction
    const int je = ny - 3;//y direction
    const int ke = nz - 3;//z direction

    double pmin_x = pmin[0];
    double pmin_y = pmin[1];
    double pmin_z = pmin[2];

    double pmax_x = pmax[0];
    double pmax_y = pmax[1];
    double pmax_z = pmax[2];

    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];

    int maximumIterations = (je>ke) ? je: ke;
    
    int requiredBlocks = (9 + maximumIterations) / 10;
    
    int threads_y = (requiredBlocks-1+je) / requiredBlocks;
    int threads_z = (requiredBlocks-1+ke) / requiredBlocks;
    
    cacl_bssn_bcs_x <<< dim3(threads_y,threads_z), dim3(threads_y,threads_z), 0, stream >>> (
        dev_var_out, dev_var_in, 
        u_offset, dxf, dyf, dzf, 
        pmin_x, pmin_y, pmin_z, pmax_x, pmax_y, pmax_z, 
        f_falloff, f_asymptotic, 
        host_sz_x, host_sz_y, host_sz_z, 
        bflag );
    
    CHECK_ERROR(hipGetLastError(), "cacl_bssn_bcs_x Kernel launch failed");
        
    maximumIterations = (ke>ie) ? ke : ie ;
    requiredBlocks = (9 + maximumIterations)/10;
    int threads_x = (requiredBlocks-1+ie) / requiredBlocks;
    threads_z = (requiredBlocks-1+ke) / requiredBlocks;
    cacl_bssn_bcs_y <<< dim3(threads_x,threads_z), dim3(threads_x,threads_z), 0, stream >>> (
        dev_var_out, dev_var_in, 
        u_offset, dxf, dyf, dzf, 
        pmin_x, pmin_y, pmin_z, pmax_x, pmax_y, pmax_z, 
        f_falloff, f_asymptotic, 
        host_sz_x, host_sz_y, host_sz_z, 
        bflag );

    CHECK_ERROR(hipGetLastError(), "cacl_bssn_bcs_y Kernel launch failed");

    maximumIterations = (je>ie) ? je : ie ;
    requiredBlocks = (9 + maximumIterations)/10;
    threads_x = (requiredBlocks-1+ie) / requiredBlocks;
    threads_y = (requiredBlocks-1+je) / requiredBlocks;
    cacl_bssn_bcs_z <<< dim3(threads_x,threads_y), dim3(threads_x,threads_y), 0, stream >>> (
        dev_var_out, dev_var_in, 
        u_offset, dxf, dyf, dzf, 
        pmin_x, pmin_y, pmin_z, pmax_x, pmax_y, pmax_z, 
        f_falloff, f_asymptotic, 
        host_sz_x, host_sz_y, host_sz_z, 
        bflag );

    CHECK_ERROR(hipGetLastError(), "cacl_bssn_bcs_z Kernel launch failed");
}