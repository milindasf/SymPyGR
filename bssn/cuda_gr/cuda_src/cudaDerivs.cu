#include "hip/hip_runtime.h"
/**
 * Created on: Sep 21, 2018
 * 		Author: Akila, Eranga, Eminda, Ruwan
 **/
 
#include "cudaDerivs.cuh"

__global__ void calc_derivs1(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    #include "calc_deriv_calls_1.cuh"
}

__global__ void calc_derivs2(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    #include "calc_deriv_calls_2.cuh"
}

__global__ void calc_derivs1_bflag(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    #include "calc_deriv_calls_1_bflag.cuh"
}

__global__ void calc_derivs2_bflag(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    #include "calc_deriv_calls_2_bflag.cuh"
}

void calc_deriv_kernel_wrapper(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int * host_sz, int bflag, hipStream_t stream,
    #include "para_derivs_offsets.h"
){
    const int ib = 1;
    const int jb = 1;
    const int kb = 1;
    const int ie = host_sz[0] - 1;
    const int je = host_sz[1] - 1;
    const int ke = host_sz[2] - 1;
    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];

    int number_of_threads_required;
    int number_of_blocks;

    if (bflag!=0){
        number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
        number_of_blocks=ceil(1.0*number_of_threads_required/64);
        calc_derivs1_bflag <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    
        number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
        number_of_blocks=ceil(1.0*number_of_threads_required/64);
        calc_derivs2_bflag <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    }else{
        number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
        number_of_blocks=ceil(1.0*number_of_threads_required/64);
        calc_derivs1 <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    
        number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
        number_of_blocks=ceil(1.0*number_of_threads_required/64);
        calc_derivs2 <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    }
    CHECK_ERROR(hipGetLastError(), "deriv Kernel launch failed");
}



__global__ void calc_ko_derivs(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    #include "calc_ko_deriv_calls.cuh"
}


__global__ void calc_ko_derivs_bflag(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    #include "calc_ko_deriv_calls_bflag.cuh"
}

void calc_ko_deriv_kernel_wrapper(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int * host_sz, int bflag, hipStream_t stream,
    #include "para_derivs_offsets.h"
    )
{
    const int ib = 1;
    const int jb = 1;
    const int kb = 1;
    const int ie = host_sz[0] - 1;
    const int je = host_sz[1] - 1;
    const int ke = host_sz[2] - 1;
    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];

    int number_of_threads_required;
    int number_of_blocks;

    number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
    number_of_blocks=ceil(1.0*number_of_threads_required/64);

    if (bflag!=0){
        calc_ko_derivs_bflag <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    }else{
        calc_ko_derivs <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    }
    
    CHECK_ERROR(hipGetLastError(), "ko deriv Kernel launch failed");
}
