#include "hip/hip_runtime.h"
/**
 * Created on: Sep 21, 2018
 * 		Author: Akila, Eranga, Eminda, Ruwan
 **/
 
#include "cudaDerivs.cuh"

__global__ void calc_derivs1(int tile_size, double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    __shared__ double shared_var_in[4096];

    int tile_x = blockIdx.x%tile_size;
    int tile_y = blockIdx.x/tile_size%tile_size;
    int tile_z = blockIdx.x/tile_size/tile_size;

    int x_offset = tile_x*10;
    int y_offset = tile_y*10;
    int z_offset = tile_z*10;

    int nx = host_sz_x; 
    int ny = host_sz_y;

    #include "calc_deriv_calls_1.cuh"
}

__global__ void calc_derivs2(int tile_size, double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    __shared__ double shared_var_in[4096];

    int tile_x = blockIdx.x%tile_size;
    int tile_y = blockIdx.x/tile_size%tile_size;
    int tile_z = blockIdx.x/tile_size/tile_size;

    int x_offset = tile_x*10;
    int y_offset = tile_y*10;
    int z_offset = tile_z*10;

    int nx = host_sz_x; 
    int ny = host_sz_y;

    #include "calc_deriv_calls_2.cuh"
}

__global__ void calc_derivs1_bflag(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    #include "calc_deriv_calls_1_bflag.cuh"
}

__global__ void calc_derivs2_bflag(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    #include "calc_deriv_calls_2_bflag.cuh"
}

void calc_deriv_kernel_wrapper(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int * host_sz, int bflag, hipStream_t stream,
    #include "para_derivs_offsets.h"
){
    const int ib = 1;
    const int jb = 1;
    const int kb = 1;
    const int ie = host_sz[0] - 1;
    const int je = host_sz[1] - 1;
    const int ke = host_sz[2] - 1;
    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];

    int number_of_threads_required;
    int number_of_blocks;

    int x = ceil((ie-ib)/10.0);
    int y = ceil((je-jb)/10.0);
    int z = ceil((ke-kb)/10.0);

    if (bflag!=0){
        number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
        number_of_blocks=ceil(1.0*number_of_threads_required/64);
        calc_derivs1_bflag <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    
        number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
        number_of_blocks=ceil(1.0*number_of_threads_required/64);
        calc_derivs2_bflag <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    }else{
        number_of_blocks = x*y*z;
        calc_derivs1 <<< number_of_blocks, 1000, 0, stream>>> (x, dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );

        calc_derivs2 <<< number_of_blocks, 1000, 0, stream>>> (x, dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    }
    CHECK_ERROR(hipGetLastError(), "deriv Kernel launch failed");
}



__global__ void calc_ko_derivs(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    #include "calc_ko_deriv_calls.cuh"
}


__global__ void calc_ko_derivs_bflag(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, int bflag,
    #include "para_derivs_offsets.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    #include "calc_ko_deriv_calls_bflag.cuh"
}

void calc_ko_deriv_kernel_wrapper(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int * host_sz, int bflag, hipStream_t stream,
    #include "para_derivs_offsets.h"
    )
{
    const int ib = 1;
    const int jb = 1;
    const int kb = 1;
    const int ie = host_sz[0] - 1;
    const int je = host_sz[1] - 1;
    const int ke = host_sz[2] - 1;
    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];

    int number_of_threads_required;
    int number_of_blocks;

    number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
    number_of_blocks=ceil(1.0*number_of_threads_required/64);

    if (bflag!=0){
        calc_ko_derivs_bflag <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    }else{
        calc_ko_derivs <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "args_derivs_offsets.h"
        );
    }
    
    CHECK_ERROR(hipGetLastError(), "ko deriv Kernel launch failed");
}
