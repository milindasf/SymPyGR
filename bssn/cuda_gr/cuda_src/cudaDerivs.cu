#include "hip/hip_runtime.h"
/**
 * Created on: Sep 21, 2018
 * 		Author: Akila, Eranga, Eminda, Ruwan
 **/
 
#include "cudaDerivs.cuh"

__global__ void calc_derivs1(
    double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, 
    int bflag,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    calc_deriv42_x(tid, grad_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_B0, dev_var_in, B0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_B0, dev_var_in, B0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_B0, dev_var_in, B0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_B1, dev_var_in, B1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_B1, dev_var_in, B1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_B1, dev_var_in, B1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_B2, dev_var_in, B2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_B2, dev_var_in, B2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_B2, dev_var_in, B2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_Gt0, dev_var_in, Gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_Gt0, dev_var_in, Gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_Gt0, dev_var_in, Gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_Gt1, dev_var_in, Gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_Gt1, dev_var_in, Gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_Gt1, dev_var_in, Gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_Gt2, dev_var_in, Gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_Gt2, dev_var_in, Gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_Gt2, dev_var_in, Gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_K, dev_var_in, KInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_K, dev_var_in, KInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_K, dev_var_in, KInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx(tid, grad2_0_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy(tid, grad2_1_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz(tid, grad2_2_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At0, dev_var_in, At0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At0, dev_var_in, At0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At0, dev_var_in, At0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At1, dev_var_in, At1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At1, dev_var_in, At1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At1, dev_var_in, At1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At2, dev_var_in, At2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At2, dev_var_in, At2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At2, dev_var_in, At2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At3, dev_var_in, At3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At3, dev_var_in, At3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At3, dev_var_in, At3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At4, dev_var_in, At4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At4, dev_var_in, At4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At4, dev_var_in, At4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x(tid, grad_0_At5, dev_var_in, At5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad_1_At5, dev_var_in, At5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad_2_At5, dev_var_in, At5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
}

__global__ void calc_derivs2(
    double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, 
    int bflag,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    calc_deriv42_y(tid, grad2_0_1_gt0, grad_0_gt0, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt0, grad_0_gt0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt0, grad_1_gt0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_gt1, grad_0_gt1, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt1, grad_0_gt1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt1, grad_1_gt1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_gt2, grad_0_gt2, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt2, grad_0_gt2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt2, grad_1_gt2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_gt3, grad_0_gt3, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt3, grad_0_gt3, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt3, grad_1_gt3, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_gt4, grad_0_gt4, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt4, grad_0_gt4, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt4, grad_1_gt4, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_gt5, grad_0_gt5, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_gt5, grad_0_gt5, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_gt5, grad_1_gt5, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_chi, grad_0_chi, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_chi, grad_0_chi, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_chi, grad_1_chi, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_alpha, grad_0_alpha, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_alpha, grad_0_alpha, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_alpha, grad_1_alpha, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_beta0, grad_0_beta0, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_beta0, grad_0_beta0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_beta0, grad_1_beta0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_beta1, grad_0_beta1, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_beta1, grad_0_beta1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_beta1, grad_1_beta1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y(tid, grad2_0_1_beta2, grad_0_beta2, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_0_2_beta2, grad_0_beta2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z(tid, grad2_1_2_beta2, grad_1_beta2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt0, dev_var_in, gt0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt0, dev_var_in, gt0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt1, dev_var_in, gt1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt1, dev_var_in, gt1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt2, dev_var_in, gt2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt2, dev_var_in, gt2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt3, dev_var_in, gt3Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt3, dev_var_in, gt3Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt4, dev_var_in, gt4Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt4, dev_var_in, gt4Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_gt5, dev_var_in, gt5Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_gt5, dev_var_in, gt5Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);


    calc_deriv42_adv_x(tid, agrad_0_At0, dev_var_in, At0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At0, dev_var_in, At0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_At1, dev_var_in, At1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At1, dev_var_in, At1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_At2, dev_var_in, At2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At2, dev_var_in, At2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_At3, dev_var_in, At3Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At3, dev_var_in, At3Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_At4, dev_var_in, At4Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At4, dev_var_in, At4Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_At5, dev_var_in, At5Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_At5, dev_var_in, At5Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x(tid, agrad_0_alpha, dev_var_in, alphaInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_alpha, dev_var_in, alphaInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_beta0, dev_var_in, beta0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_beta1, dev_var_in, beta1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_beta2, dev_var_in, beta2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_chi, dev_var_in, chiInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_Gt0, dev_var_in, Gt0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_Gt1, dev_var_in, Gt1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_Gt2, dev_var_in, Gt2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_K, dev_var_in, KInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_B0, dev_var_in, B0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_B1, dev_var_in, B1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x(tid, agrad_0_B2, dev_var_in, B2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_beta0, dev_var_in, beta0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_beta1, dev_var_in, beta1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_beta2, dev_var_in, beta2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_chi, dev_var_in, chiInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_Gt0, dev_var_in, Gt0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_Gt1, dev_var_in, Gt1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_Gt2, dev_var_in, Gt2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_K, dev_var_in, KInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_B0, dev_var_in, B0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_B1, dev_var_in, B1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y(tid, agrad_1_B2, dev_var_in, B2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt0, dev_var_in, gt0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt1, dev_var_in, gt1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt2, dev_var_in, gt2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt3, dev_var_in, gt3Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt4, dev_var_in, gt4Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_gt5, dev_var_in, gt5Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At0, dev_var_in, At0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At1, dev_var_in, At1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At2, dev_var_in, At2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At3, dev_var_in, At3Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At4, dev_var_in, At4Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_At5, dev_var_in, At5Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_alpha, dev_var_in, alphaInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_beta0, dev_var_in, beta0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_beta1, dev_var_in, beta1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_beta2, dev_var_in, beta2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_chi, dev_var_in, chiInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_Gt0, dev_var_in, Gt0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_Gt1, dev_var_in, Gt1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_Gt2, dev_var_in, Gt2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_K, dev_var_in, KInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_B0, dev_var_in, B0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_B1, dev_var_in, B1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z(tid, agrad_2_B2, dev_var_in, B2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
}

__global__ void calc_derivs1_bflag(
    double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, 
    int bflag,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    calc_deriv42_x_bflag(tid, grad_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_B0, dev_var_in, B0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_B0, dev_var_in, B0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_B0, dev_var_in, B0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_B1, dev_var_in, B1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_B1, dev_var_in, B1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_B1, dev_var_in, B1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_B2, dev_var_in, B2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_B2, dev_var_in, B2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_B2, dev_var_in, B2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_Gt0, dev_var_in, Gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_Gt0, dev_var_in, Gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_Gt0, dev_var_in, Gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_Gt1, dev_var_in, Gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_Gt1, dev_var_in, Gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_Gt1, dev_var_in, Gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_Gt2, dev_var_in, Gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_Gt2, dev_var_in, Gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_Gt2, dev_var_in, Gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_K, dev_var_in, KInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_K, dev_var_in, KInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_K, dev_var_in, KInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_xx_bflag(tid, grad2_0_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_yy_bflag(tid, grad2_1_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_zz_bflag(tid, grad2_2_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_At0, dev_var_in, At0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_At0, dev_var_in, At0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_At0, dev_var_in, At0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_At1, dev_var_in, At1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_At1, dev_var_in, At1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_At1, dev_var_in, At1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_At2, dev_var_in, At2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_At2, dev_var_in, At2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_At2, dev_var_in, At2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_At3, dev_var_in, At3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_At3, dev_var_in, At3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_At3, dev_var_in, At3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_At4, dev_var_in, At4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_At4, dev_var_in, At4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_At4, dev_var_in, At4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_x_bflag(tid, grad_0_At5, dev_var_in, At5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad_1_At5, dev_var_in, At5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad_2_At5, dev_var_in, At5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
}

__global__ void calc_derivs2_bflag(
    double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, 
    int bflag,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    calc_deriv42_y_bflag(tid, grad2_0_1_gt0, grad_0_gt0, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_gt0, grad_0_gt0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_gt0, grad_1_gt0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad2_0_1_gt1, grad_0_gt1, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_gt1, grad_0_gt1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_gt1, grad_1_gt1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad2_0_1_gt2, grad_0_gt2, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_gt2, grad_0_gt2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_gt2, grad_1_gt2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad2_0_1_gt3, grad_0_gt3, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_gt3, grad_0_gt3, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_gt3, grad_1_gt3, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad2_0_1_gt4, grad_0_gt4, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_gt4, grad_0_gt4, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_gt4, grad_1_gt4, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad2_0_1_gt5, grad_0_gt5, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_gt5, grad_0_gt5, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_gt5, grad_1_gt5, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad2_0_1_chi, grad_0_chi, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_chi, grad_0_chi, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_chi, grad_1_chi, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad2_0_1_alpha, grad_0_alpha, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_alpha, grad_0_alpha, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_alpha, grad_1_alpha, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad2_0_1_beta0, grad_0_beta0, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_beta0, grad_0_beta0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_beta0, grad_1_beta0, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad2_0_1_beta1, grad_0_beta1, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_beta1, grad_0_beta1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_beta1, grad_1_beta1, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_y_bflag(tid, grad2_0_1_beta2, grad_0_beta2, 0, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_0_2_beta2, grad_0_beta2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_z_bflag(tid, grad2_1_2_beta2, grad_1_beta2, 0, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_gt0, dev_var_in, gt0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_gt0, dev_var_in, gt0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_gt1, dev_var_in, gt1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_gt1, dev_var_in, gt1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_gt2, dev_var_in, gt2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_gt2, dev_var_in, gt2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_gt3, dev_var_in, gt3Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_gt3, dev_var_in, gt3Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_gt4, dev_var_in, gt4Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_gt4, dev_var_in, gt4Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_gt5, dev_var_in, gt5Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_gt5, dev_var_in, gt5Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);


    calc_deriv42_adv_x_bflag(tid, agrad_0_At0, dev_var_in, At0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_At0, dev_var_in, At0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_At1, dev_var_in, At1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_At1, dev_var_in, At1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_At2, dev_var_in, At2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_At2, dev_var_in, At2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_At3, dev_var_in, At3Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_At3, dev_var_in, At3Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_At4, dev_var_in, At4Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_At4, dev_var_in, At4Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_At5, dev_var_in, At5Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_At5, dev_var_in, At5Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_deriv42_adv_x_bflag(tid, agrad_0_alpha, dev_var_in, alphaInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_alpha, dev_var_in, alphaInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_beta0, dev_var_in, beta0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_beta1, dev_var_in, beta1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_beta2, dev_var_in, beta2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_chi, dev_var_in, chiInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_Gt0, dev_var_in, Gt0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_Gt1, dev_var_in, Gt1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_Gt2, dev_var_in, Gt2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_K, dev_var_in, KInt, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_B0, dev_var_in, B0Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_B1, dev_var_in, B1Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_x_bflag(tid, agrad_0_B2, dev_var_in, B2Int, hx, beta0Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_beta0, dev_var_in, beta0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_beta1, dev_var_in, beta1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_beta2, dev_var_in, beta2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_chi, dev_var_in, chiInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_Gt0, dev_var_in, Gt0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_Gt1, dev_var_in, Gt1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_Gt2, dev_var_in, Gt2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_K, dev_var_in, KInt, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_B0, dev_var_in, B0Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_B1, dev_var_in, B1Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_y_bflag(tid, agrad_1_B2, dev_var_in, B2Int, hy, beta1Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_gt0, dev_var_in, gt0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_gt1, dev_var_in, gt1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_gt2, dev_var_in, gt2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_gt3, dev_var_in, gt3Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_gt4, dev_var_in, gt4Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_gt5, dev_var_in, gt5Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_At0, dev_var_in, At0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_At1, dev_var_in, At1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_At2, dev_var_in, At2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_At3, dev_var_in, At3Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_At4, dev_var_in, At4Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_At5, dev_var_in, At5Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_alpha, dev_var_in, alphaInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_beta0, dev_var_in, beta0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_beta1, dev_var_in, beta1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_beta2, dev_var_in, beta2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_chi, dev_var_in, chiInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_Gt0, dev_var_in, Gt0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_Gt1, dev_var_in, Gt1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_Gt2, dev_var_in, Gt2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_K, dev_var_in, KInt, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_B0, dev_var_in, B0Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_B1, dev_var_in, B1Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_deriv42_adv_z_bflag(tid, agrad_2_B2, dev_var_in, B2Int, hz, beta2Int, host_sz_x, host_sz_y, host_sz_z, bflag);
}

void calc_deriv_wrapper(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int * host_sz, int bflag, hipStream_t stream,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    const int ib = 1;
    const int jb = 1;
    const int kb = 1;
    const int ie = host_sz[0] - 1;
    const int je = host_sz[1] - 1;
    const int ke = host_sz[2] - 1;
    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];

    int number_of_threads_required;
    int number_of_blocks;

    if (bflag!=0){
        number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
        number_of_blocks=ceil(1.0*number_of_threads_required/64);
        calc_derivs1_bflag <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "list_of_offset_args.h"
            ,
            #include "list_of_args.h"
        );
    
        number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
        number_of_blocks=ceil(1.0*number_of_threads_required/64);
        calc_derivs2_bflag <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "list_of_offset_args.h"
            ,
            #include "list_of_args.h"
        );
    }else{
        number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
        number_of_blocks=ceil(1.0*number_of_threads_required/64);
        calc_derivs1 <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "list_of_offset_args.h"
            ,
            #include "list_of_args.h"
        );
    
        number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
        number_of_blocks=ceil(1.0*number_of_threads_required/64);
        calc_derivs2 <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "list_of_offset_args.h"
            ,
            #include "list_of_args.h"
        );
    }
    CHECK_ERROR(hipGetLastError(), "deriv Kernel launch failed");
}



__global__ void calc_ko_derivs(
    double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, 
    int bflag,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    calc_ko_deriv42_x(tid, grad_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At0, dev_var_in, At0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At0, dev_var_in, At0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At0, dev_var_in, At0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At1, dev_var_in, At1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At1, dev_var_in, At1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At1, dev_var_in, At1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At2, dev_var_in, At2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At2, dev_var_in, At2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At2, dev_var_in, At2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At3, dev_var_in, At3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At3, dev_var_in, At3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At3, dev_var_in, At3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At4, dev_var_in, At4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At4, dev_var_in, At4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At4, dev_var_in, At4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_At5, dev_var_in, At5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_At5, dev_var_in, At5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_At5, dev_var_in, At5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_Gt0, dev_var_in, Gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_Gt0, dev_var_in, Gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_Gt0, dev_var_in, Gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_Gt1, dev_var_in, Gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_Gt1, dev_var_in, Gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_Gt1, dev_var_in, Gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_Gt2, dev_var_in, Gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_Gt2, dev_var_in, Gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_Gt2, dev_var_in, Gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_K, dev_var_in, KInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_K, dev_var_in, KInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_K, dev_var_in, KInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_B0, dev_var_in, B0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_B0, dev_var_in, B0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_B0, dev_var_in, B0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_B1, dev_var_in, B1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_B1, dev_var_in, B1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_B1, dev_var_in, B1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x(tid, grad_0_B2, dev_var_in, B2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y(tid, grad_1_B2, dev_var_in, B2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z(tid, grad_2_B2, dev_var_in, B2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);


}


__global__ void calc_ko_derivs_bflag(
    double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, 
    const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z, 
    int bflag,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
){
    int tid = blockIdx.x*64 + threadIdx.x;

    calc_ko_deriv42_x_bflag(tid, grad_0_gt0, dev_var_in, gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_gt0, dev_var_in, gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_gt0, dev_var_in, gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_gt1, dev_var_in, gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_gt1, dev_var_in, gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_gt1, dev_var_in, gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_gt2, dev_var_in, gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_gt2, dev_var_in, gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_gt2, dev_var_in, gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_gt3, dev_var_in, gt3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_gt3, dev_var_in, gt3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_gt3, dev_var_in, gt3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_gt4, dev_var_in, gt4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_gt4, dev_var_in, gt4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_gt4, dev_var_in, gt4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_gt5, dev_var_in, gt5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_gt5, dev_var_in, gt5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_gt5, dev_var_in, gt5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_At0, dev_var_in, At0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_At0, dev_var_in, At0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_At0, dev_var_in, At0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_At1, dev_var_in, At1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_At1, dev_var_in, At1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_At1, dev_var_in, At1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_At2, dev_var_in, At2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_At2, dev_var_in, At2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_At2, dev_var_in, At2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_At3, dev_var_in, At3Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_At3, dev_var_in, At3Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_At3, dev_var_in, At3Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_At4, dev_var_in, At4Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_At4, dev_var_in, At4Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_At4, dev_var_in, At4Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_At5, dev_var_in, At5Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_At5, dev_var_in, At5Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_At5, dev_var_in, At5Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_alpha, dev_var_in, alphaInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_alpha, dev_var_in, alphaInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_alpha, dev_var_in, alphaInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_beta0, dev_var_in, beta0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_beta0, dev_var_in, beta0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_beta0, dev_var_in, beta0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_beta1, dev_var_in, beta1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_beta1, dev_var_in, beta1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_beta1, dev_var_in, beta1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_beta2, dev_var_in, beta2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_beta2, dev_var_in, beta2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_beta2, dev_var_in, beta2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_chi, dev_var_in, chiInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_chi, dev_var_in, chiInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_chi, dev_var_in, chiInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_Gt0, dev_var_in, Gt0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_Gt0, dev_var_in, Gt0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_Gt0, dev_var_in, Gt0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_Gt1, dev_var_in, Gt1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_Gt1, dev_var_in, Gt1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_Gt1, dev_var_in, Gt1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_Gt2, dev_var_in, Gt2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_Gt2, dev_var_in, Gt2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_Gt2, dev_var_in, Gt2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_K, dev_var_in, KInt, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_K, dev_var_in, KInt, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_K, dev_var_in, KInt, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_B0, dev_var_in, B0Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_B0, dev_var_in, B0Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_B0, dev_var_in, B0Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_B1, dev_var_in, B1Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_B1, dev_var_in, B1Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_B1, dev_var_in, B1Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);

    calc_ko_deriv42_x_bflag(tid, grad_0_B2, dev_var_in, B2Int, hx, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_y_bflag(tid, grad_1_B2, dev_var_in, B2Int, hy, host_sz_x, host_sz_y, host_sz_z, bflag);
    calc_ko_deriv42_z_bflag(tid, grad_2_B2, dev_var_in, B2Int, hz, host_sz_x, host_sz_y, host_sz_z, bflag);


}

void calc_ko_deriv_wrapper(double * dev_var_out, double * dev_var_in, double hx, double hy, double hz, const unsigned int * host_sz, int bflag, hipStream_t stream,
    #include "list_of_offset_para.h"
    ,
    #include "list_of_para.h"
    )
{
    const int ib = 1;
    const int jb = 1;
    const int kb = 1;
    const int ie = host_sz[0] - 1;
    const int je = host_sz[1] - 1;
    const int ke = host_sz[2] - 1;
    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];

    int number_of_threads_required;
    int number_of_blocks;

    number_of_threads_required=ceil((ie-ib)*(je-jb)*(ke-kb));
    number_of_blocks=ceil(1.0*number_of_threads_required/64);

    if (bflag!=0){
        calc_ko_derivs_bflag <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "list_of_offset_args.h"
            ,
            #include "list_of_args.h"
        );
    }else{
        calc_ko_derivs <<< number_of_blocks, 64, 0, stream>>> (dev_var_out, dev_var_in, hx, hy, hz, host_sz_x, host_sz_y, host_sz_z, bflag,
            #include "list_of_offset_args.h"
            ,
            #include "list_of_args.h"
        );
    }
    
    CHECK_ERROR(hipGetLastError(), "ko deriv Kernel launch failed");
}
