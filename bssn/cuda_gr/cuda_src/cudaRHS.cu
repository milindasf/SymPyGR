/**
 * Created on: Sep 21, 2018
 * 		Author: Akila, Eranga, Eminda, Ruwan
 **/
 
#include "cudaRHS.cuh"

enum VAR_CU {U_ALPHA=0,U_CHI,U_K,U_GT0,U_GT1,U_GT2,U_BETA0,U_BETA1,U_BETA2,U_B0,U_B1,U_B2,U_SYMGT0,U_SYMGT1,U_SYMGT2,U_SYMGT3,U_SYMGT4,U_SYMGT5,U_SYMAT0,U_SYMAT1,U_SYMAT2,U_SYMAT3,U_SYMAT4,U_SYMAT5};

void cuda_bssnrhs(double * dev_var_out, double * dev_var_in, const unsigned int unzip_dof, 
const double * pmin, const double * pmax, const unsigned int * sz, 
const unsigned int& bflag, hipStream_t stream,
#include "list_of_para.h"
)
{ 
    CHECK_ERROR(hipMemsetAsync(dev_var_out, 0, 24*unzip_dof*sizeof(double), stream), "output array cleaning call"); // Clean output array

    int alphaInt = (VAR_CU::U_ALPHA) * unzip_dof;
    int chiInt = (VAR_CU::U_CHI) * unzip_dof;
    int KInt = (VAR_CU::U_K) * unzip_dof;
    int gt0Int = (VAR_CU::U_SYMGT0) * unzip_dof;
    int gt1Int = (VAR_CU::U_SYMGT1) * unzip_dof;
    int gt2Int =  (VAR_CU::U_SYMGT2) * unzip_dof;
    int gt3Int = (VAR_CU::U_SYMGT3) * unzip_dof;
    int gt4Int = (VAR_CU::U_SYMGT4) * unzip_dof;
    int gt5Int = (VAR_CU::U_SYMGT5) * unzip_dof;
    int beta0Int = (VAR_CU::U_BETA0) * unzip_dof;
    int beta1Int = (VAR_CU::U_BETA1) * unzip_dof;
    int beta2Int = (VAR_CU::U_BETA2) * unzip_dof;
    int At0Int = (VAR_CU::U_SYMAT0) * unzip_dof;
    int At1Int = (VAR_CU::U_SYMAT1) * unzip_dof;
    int At2Int = (VAR_CU::U_SYMAT2) * unzip_dof;
    int At3Int = (VAR_CU::U_SYMAT3) * unzip_dof;
    int At4Int = (VAR_CU::U_SYMAT4) * unzip_dof;
    int At5Int = (VAR_CU::U_SYMAT5) * unzip_dof;
    int Gt0Int = (VAR_CU::U_GT0) * unzip_dof;
    int Gt1Int = (VAR_CU::U_GT1) * unzip_dof;
    int Gt2Int = (VAR_CU::U_GT2) * unzip_dof;
    int B0Int = (VAR_CU::U_B0) * unzip_dof;
    int B1Int = (VAR_CU::U_B1) * unzip_dof;
    int B2Int = (VAR_CU::U_B2) * unzip_dof;

    double hx = (pmax[0] - pmin[0]) / (sz[0] - 1);
    double hy = (pmax[1] - pmin[1]) / (sz[1] - 1);
    double hz = (pmax[2] - pmin[2]) / (sz[2] - 1);

    calc_deriv_wrapper(dev_var_out, dev_var_in, hx, hy, hz, sz, bflag, stream,
        #include "list_of_offset_args.h"
        ,
        #include "list_of_args.h"
    );

    calc_bssn_eqns(dev_var_in, dev_var_out, sz, pmin, hz, hy, hx, stream,
    #include "list_of_offset_args.h"
    ,
    #include "list_of_args.h"
    );

    if (bflag!=0) {
        bssn_bcs(dev_var_out, dev_var_in, alphaInt, grad_0_alpha, grad_1_alpha, grad_2_alpha,
            pmin, pmax, 1.0, 1.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, chiInt, grad_0_chi, grad_1_chi, grad_2_chi,
            pmin, pmax, 1.0, 1.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, KInt, grad_0_K, grad_1_K, grad_2_K,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);

        bssn_bcs(dev_var_out, dev_var_in, beta0Int, grad_0_beta0, grad_1_beta0, grad_2_beta0,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, beta1Int, grad_0_beta1, grad_1_beta1, grad_2_beta1,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, beta2Int, grad_0_beta2, grad_1_beta2, grad_2_beta2,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);

        bssn_bcs(dev_var_out, dev_var_in, Gt0Int, grad_0_Gt0, grad_1_Gt0, grad_2_Gt0,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, Gt1Int, grad_0_Gt1, grad_1_Gt1, grad_2_Gt1,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, Gt2Int, grad_0_Gt2, grad_1_Gt2, grad_2_Gt2,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);

        bssn_bcs(dev_var_out, dev_var_in, B0Int, grad_0_B0, grad_1_B0, grad_2_B0,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, B1Int, grad_0_B1, grad_1_B1, grad_2_B1,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, B2Int, grad_0_B2, grad_1_B2, grad_2_B2,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);

        bssn_bcs(dev_var_out, dev_var_in, At0Int, grad_0_At0, grad_1_At0, grad_2_At0,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, At1Int, grad_0_At1, grad_1_At1, grad_2_At1,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, At2Int, grad_0_At2, grad_1_At2, grad_2_At2,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, At3Int, grad_0_At3, grad_1_At3, grad_2_At3,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, At4Int, grad_0_At4, grad_1_At4, grad_2_At4,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, At5Int, grad_0_At5, grad_1_At5, grad_2_At5,
            pmin, pmax, 2.0, 0.0, sz, bflag, stream); 

        bssn_bcs(dev_var_out, dev_var_in, gt0Int, grad_0_gt0, grad_1_gt0, grad_2_gt0,
            pmin, pmax, 1.0, 1.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, gt1Int, grad_0_gt1, grad_1_gt1, grad_2_gt1,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, gt2Int, grad_0_gt2, grad_1_gt2, grad_2_gt2,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, gt3Int, grad_0_gt3, grad_1_gt3, grad_2_gt3,
            pmin, pmax, 1.0, 1.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, gt4Int, grad_0_gt4, grad_1_gt4, grad_2_gt4,
            pmin, pmax, 1.0, 0.0, sz, bflag, stream);
        bssn_bcs(dev_var_out, dev_var_in, gt5Int, grad_0_gt5, grad_1_gt5, grad_2_gt5,
            pmin, pmax, 1.0, 1.0, sz, bflag, stream); 
    }

    calc_ko_deriv_wrapper(dev_var_out, dev_var_in, hx, hy, hz, sz, bflag, stream,
        #include "list_of_offset_args.h"
        ,
        #include "list_of_args.h"
    );

    get_output(dev_var_out, sz, stream,
        #include "list_of_offset_args.h"
        ,
        #include "list_of_args.h"
    );
    return;
}
