#include "hip/hip_runtime.h"
/**
 * Created on: Sep 21, 2018
 * 		Author: Akila, Eranga, Eminda, Ruwan
 **/
 
#include "getOutput.cuh"

__global__ void calc_get_output (double * dev_var_out, const unsigned int host_sz_x, const unsigned int host_sz_y, const unsigned int host_sz_z,
    #include "para_derivs_offsets.h"
) 
{
    int thread_id = blockIdx.x*1024 + threadIdx.x;

    int i = thread_id%(host_sz_x-6) + 3;
    int j = ((thread_id/(host_sz_x-6))%(host_sz_y-6)) + 3;
    int k = (thread_id/(host_sz_y-6)/(host_sz_x-6)) + 3;

    int nx = host_sz_x;
    int ny = host_sz_y;

    if(i >= nx-3 || j >= ny-3 || k >= host_sz_z-3) return;

    const  double sigma = 1e-4;
    int pp = i + nx*(j + ny*k);

    dev_var_out[alphaInt + pp] += sigma * (grad_0_alpha[pp] + grad_1_alpha[pp] + grad_2_alpha[pp]);
    dev_var_out[beta0Int + pp] += sigma * (grad_0_beta0[pp] + grad_1_beta0[pp] + grad_2_beta0[pp]);
    dev_var_out[beta1Int + pp] += sigma * (grad_0_beta1[pp] + grad_1_beta1[pp] + grad_2_beta1[pp]);
    dev_var_out[beta2Int + pp] += sigma * (grad_0_beta2[pp] + grad_1_beta2[pp] + grad_2_beta2[pp]);

    dev_var_out[gt0Int + pp] += sigma * (grad_0_gt0[pp] + grad_1_gt0[pp] + grad_2_gt0[pp]);
    dev_var_out[gt1Int + pp] += sigma * (grad_0_gt1[pp] + grad_1_gt1[pp] + grad_2_gt1[pp]);
    dev_var_out[gt2Int + pp] += sigma * (grad_0_gt2[pp] + grad_1_gt2[pp] + grad_2_gt2[pp]);
    dev_var_out[gt3Int + pp] += sigma * (grad_0_gt3[pp] + grad_1_gt3[pp] + grad_2_gt3[pp]);
    dev_var_out[gt4Int + pp] += sigma * (grad_0_gt4[pp] + grad_1_gt4[pp] + grad_2_gt4[pp]);
    dev_var_out[gt5Int + pp] += sigma * (grad_0_gt5[pp] + grad_1_gt5[pp] + grad_2_gt5[pp]);

    dev_var_out[chiInt + pp]  += sigma * (grad_0_chi[pp] + grad_1_chi[pp] + grad_2_chi[pp]);

    dev_var_out[At0Int + pp] += sigma * (grad_0_At0[pp] + grad_1_At0[pp] + grad_2_At0[pp]);
    dev_var_out[At1Int + pp] += sigma * (grad_0_At1[pp] + grad_1_At1[pp] + grad_2_At1[pp]);
    dev_var_out[At2Int + pp] += sigma * (grad_0_At2[pp] + grad_1_At2[pp] + grad_2_At2[pp]);
    dev_var_out[At3Int + pp] += sigma * (grad_0_At3[pp] + grad_1_At3[pp] + grad_2_At3[pp]);
    dev_var_out[At4Int + pp] += sigma * (grad_0_At4[pp] + grad_1_At4[pp] + grad_2_At4[pp]);
    dev_var_out[At5Int + pp] += sigma * (grad_0_At5[pp] + grad_1_At5[pp] + grad_2_At5[pp]);

    dev_var_out[KInt + pp] += sigma * (grad_0_K[pp] + grad_1_K[pp] + grad_2_K[pp]);
    
    dev_var_out[Gt0Int + pp] += sigma * (grad_0_Gt0[pp] + grad_1_Gt0[pp] + grad_2_Gt0[pp]);
    dev_var_out[Gt1Int + pp] += sigma * (grad_0_Gt1[pp] + grad_1_Gt1[pp] + grad_2_Gt1[pp]);
    dev_var_out[Gt2Int + pp] += sigma * (grad_0_Gt2[pp] + grad_1_Gt2[pp] + grad_2_Gt2[pp]);

    dev_var_out[B0Int + pp] += sigma * (grad_0_B0[pp] + grad_1_B0[pp] + grad_2_B0[pp]);
    dev_var_out[B1Int + pp] += sigma * (grad_0_B1[pp] + grad_1_B1[pp] + grad_2_B1[pp]);
    dev_var_out[B2Int + pp] += sigma * (grad_0_B2[pp] + grad_1_B2[pp] + grad_2_B2[pp]);
}

void get_output_kernel_wrapper(double * dev_var_out, const unsigned int * host_sz, hipStream_t stream,
    #include "para_derivs_offsets.h"
) 
{
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction

    const unsigned int host_sz_x = host_sz[0];
    const unsigned int host_sz_y = host_sz[1];
    const unsigned int host_sz_z = host_sz[2];

    int total_points = ceil(1.0*ie*je*ke);
    int blocks = ceil(1.0*total_points/1024);

    calc_get_output <<< blocks, 1024, 0, stream >>> (dev_var_out, 
                host_sz_x, host_sz_y, host_sz_z,
                #include "args_derivs_offsets.h"
                );
    
    CHECK_ERROR(hipGetLastError(), "kernal_get_output Kernel launch failed");
}